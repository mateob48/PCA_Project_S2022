//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// new_mvnpdf_initialize.cu
//
// Code generation for function 'new_mvnpdf_initialize'
//

// Include files
#include "new_mvnpdf_initialize.h"
#include "_coder_new_mvnpdf_mex.h"
#include "new_mvnpdf_data.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"

// Function Definitions
void new_mvnpdf_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(
      emlrtRootTLSGlobal,
      (const char_T *)"EMLRT:runTime:MexFunctionNeedsLicense",
      (const char_T *)"distrib_computing_toolbox", 2);
  cublasEnsureInitialization(HIPBLAS_POINTER_MODE_HOST);
  cusolverEnsureInitialization();
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (new_mvnpdf_initialize.cu)
