//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// new_mvnpdf_terminate.cu
//
// Code generation for function 'new_mvnpdf_terminate'
//

// Include files
#include "new_mvnpdf_terminate.h"
#include "_coder_new_mvnpdf_mex.h"
#include "new_mvnpdf_data.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"

// Function Definitions
void new_mvnpdf_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  cusolverDestroyWorkspace();
  cusolverEnsureDestruction();
}

void new_mvnpdf_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  cublasEnsureDestruction();
}

// End of code generation (new_mvnpdf_terminate.cu)
