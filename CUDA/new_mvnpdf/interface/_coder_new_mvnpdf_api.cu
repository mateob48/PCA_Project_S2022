//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_new_mvnpdf_api.cu
//
// Code generation for function '_coder_new_mvnpdf_api'
//

// Include files
#include "_coder_new_mvnpdf_api.h"
#include "new_mvnpdf.h"
#include "new_mvnpdf_data.h"
#include "new_mvnpdf_types.h"
#include "rt_nonfinite.h"

// Function Declarations
static real_T (*b_emlrt_marshallIn(const mxArray *Mu,
                                   const char_T *identifier))[58];

static real_T (*b_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[58];

static real_T (*c_emlrt_marshallIn(const mxArray *Sigma,
                                   const char_T *identifier))[3364];

static real_T (*c_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[3364];

static real_T (*d_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[580000];

static real_T (*e_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[58];

static real_T (*emlrt_marshallIn(const mxArray *X,
                                 const char_T *identifier))[580000];

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[580000];

static const mxArray *emlrt_marshallOut(const real_T u[10000]);

static real_T (*f_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[3364];

// Function Definitions
static real_T (*b_emlrt_marshallIn(const mxArray *Mu,
                                   const char_T *identifier))[58]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[58];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(Mu), &thisId);
  emlrtDestroyArray(&Mu);
  return y;
}

static real_T (*b_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[58]
{
  real_T(*y)[58];
  y = e_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T (*c_emlrt_marshallIn(const mxArray *Sigma,
                                   const char_T *identifier))[3364]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[3364];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = c_emlrt_marshallIn(emlrtAlias(Sigma), &thisId);
  emlrtDestroyArray(&Sigma);
  return y;
}

static real_T (*c_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[3364]
{
  real_T(*y)[3364];
  y = f_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T (*d_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[580000]
{
  static const int32_T dims[2]{10000, 58};
  real_T(*ret)[580000];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  ret = (real_T(*)[580000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*e_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[58]
{
  static const int32_T dims[2]{1, 58};
  real_T(*ret)[58];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  ret = (real_T(*)[58])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*emlrt_marshallIn(const mxArray *X,
                                 const char_T *identifier))[580000]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[580000];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = emlrt_marshallIn(emlrtAlias(X), &thisId);
  emlrtDestroyArray(&X);
  return y;
}

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[580000]
{
  real_T(*y)[580000];
  y = d_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static const mxArray *emlrt_marshallOut(const real_T u[10000])
{
  static const int32_T iv[1]{0};
  static const int32_T iv1[1]{10000};
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateNumericArray(1, (const void *)&iv[0], mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m, (void *)&u[0]);
  emlrtSetDimensions((mxArray *)m, &iv1[0], 1);
  emlrtAssign(&y, m);
  return y;
}

static real_T (*f_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[3364]
{
  static const int32_T dims[2]{58, 58};
  real_T(*ret)[3364];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  ret = (real_T(*)[3364])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

void new_mvnpdf_api(new_mvnpdfStackData *SD, const mxArray *const prhs[3],
                    const mxArray *plhs[1])
{
  real_T(*X)[580000];
  real_T(*y)[10000];
  real_T(*Sigma)[3364];
  real_T(*Mu)[58];
  y = (real_T(*)[10000])mxMalloc(sizeof(real_T[10000]));
  // Marshall function inputs
  X = emlrt_marshallIn(emlrtAlias(prhs[0]), "X");
  Mu = b_emlrt_marshallIn(emlrtAlias(prhs[1]), "Mu");
  Sigma = c_emlrt_marshallIn(emlrtAlias(prhs[2]), "Sigma");
  // Invoke the target function
  new_mvnpdf(SD, *X, *Mu, *Sigma, *y);
  // Marshall function outputs
  plhs[0] = emlrt_marshallOut(*y);
}

// End of code generation (_coder_new_mvnpdf_api.cu)
