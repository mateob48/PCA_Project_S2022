//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_new_mvnpdf_mex.cu
//
// Code generation for function '_coder_new_mvnpdf_mex'
//

// Include files
#include "_coder_new_mvnpdf_mex.h"
#include "_coder_new_mvnpdf_api.h"
#include "new_mvnpdf_data.h"
#include "new_mvnpdf_initialize.h"
#include "new_mvnpdf_terminate.h"
#include "new_mvnpdf_types.h"
#include "rt_nonfinite.h"
#include <stdexcept>

void emlrtExceptionBridge();
void emlrtExceptionBridge()
{
  throw std::runtime_error("");
}
// Function Definitions
void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs,
                 const mxArray *prhs[])
{
  new_mvnpdfStackData *new_mvnpdfStackDataGlobal{nullptr};
  new_mvnpdfStackDataGlobal =
      static_cast<new_mvnpdfStackData *>(new new_mvnpdfStackData);
  mexAtExit(&new_mvnpdf_atexit);
  // Module initialization.
  new_mvnpdf_initialize();
  try {
    emlrtShouldCleanupOnError((emlrtCTX *)emlrtRootTLSGlobal, false);
    // Dispatch the entry-point.
    unsafe_new_mvnpdf_mexFunction(new_mvnpdfStackDataGlobal, nlhs, plhs, nrhs,
                                  prhs);
    // Module termination.
    new_mvnpdf_terminate();
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
  delete new_mvnpdfStackDataGlobal;
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLSR2022a(&emlrtRootTLSGlobal, &emlrtContextGlobal, nullptr, 1,
                           (void *)&emlrtExceptionBridge,
                           (const char_T *)"windows-1252", true);
  return emlrtRootTLSGlobal;
}

void unsafe_new_mvnpdf_mexFunction(new_mvnpdfStackData *SD, int32_T nlhs,
                                   mxArray *plhs[1], int32_T nrhs,
                                   const mxArray *prhs[3])
{
  const mxArray *outputs[1];
  // Check for proper number of arguments.
  if (nrhs != 3) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 3, 4, 10, "new_mvnpdf");
  }
  if (nlhs > 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 10,
                        "new_mvnpdf");
  }
  // Call the function.
  new_mvnpdf_api(SD, prhs, outputs);
  // Copy over outputs to the caller.
  emlrtReturnArrays(1, &plhs[0], &outputs[0]);
}

// End of code generation (_coder_new_mvnpdf_mex.cu)
