#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// new_mvnpdf.cu
//
// Code generation for function 'new_mvnpdf'
//

// Include files
#include "new_mvnpdf.h"
#include "new_mvnpdf_data.h"
#include "new_mvnpdf_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "lapacke.h"
#include "hip/hip_math_constants.h"
#include <cmath>
#include <cstddef>
#include <math.h>

// Variable Definitions
static emlrtMCInfo emlrtMCI{
    27,      // lineNo
    5,       // colNo
    "error", // fName
    "C:\\Program "
    "Files\\MATLAB\\R2022a\\toolbox\\eml\\lib\\matlab\\lang\\error.m" // pName
};

// Function Declarations
static __device__ real_T atomicOpreal_T(real_T *address, real_T value);

static void b_error(const mxArray *m, const mxArray *m1, emlrtMCInfo *location);

static __global__ void new_mvnpdf_kernel1(const real_T Mu[58],
                                          const real_T X[580000],
                                          real_T X0[580000]);

static __global__ void new_mvnpdf_kernel10(const real_T T_data[3364],
                                           int32_T T_size, real_T A_data[3364]);

static __global__ void new_mvnpdf_kernel11(const real_T T_data[3364],
                                           const int32_T T_size_dim0,
                                           const int32_T T_size,
                                           int32_T b_T_size,
                                           real_T A_data[3364]);

static __global__ void new_mvnpdf_kernel12(real_T X0[580000], real_T B[580000]);

static __global__ void
new_mvnpdf_kernel13(int32_T *i2, ptrdiff_t jpvt_t_data[58], int32_T ipiv_t[58]);

static __global__ void new_mvnpdf_kernel14(const ptrdiff_t jpvt_t_data[58],
                                           int32_T *i2, int32_T ipiv_t[58]);

static __global__ void new_mvnpdf_kernel15(int32_T *i2, real_T A_data[3364]);

static __global__ void new_mvnpdf_kernel16(int32_T *i2, real_T y[58]);

static __global__ void new_mvnpdf_kernel17(int32_T *i2, int32_T ipiv_t[58]);

static __global__ void new_mvnpdf_kernel18(int16_T iv2_idx_0,
                                           real_T Y_data[580000]);

static __global__ void new_mvnpdf_kernel19(real_T B[580000]);

static __global__ void new_mvnpdf_kernel2(const real_T Sigma[3364],
                                          real_T y[58]);

static __global__ void new_mvnpdf_kernel20(const int32_T Y_size_dim0,
                                           const int32_T *rankA,
                                           real_T A_data[3364],
                                           real_T B[580000], int32_T ipiv_t[58],
                                           real_T Y_data[580000]);

static __global__ void new_mvnpdf_kernel21(const real_T Y_data[580000],
                                           const int32_T Y_size_dim0,
                                           int32_T Y_size,
                                           real_T xRinv_data[580000]);

static __global__ void new_mvnpdf_kernel22(int32_T ipiv_t[58],
                                           int32_T ipiv_data[58]);

static __global__ void new_mvnpdf_kernel23(int32_T Y_size, real_T A_data[3364]);

static __global__ void new_mvnpdf_kernel24(int32_T ipiv_data[58]);

static __global__ void new_mvnpdf_kernel25(real_T X0[580000],
                                           real_T xRinv_data[580000]);

static __global__ void new_mvnpdf_kernel26(const real_T T_data[3364],
                                           const int32_T T_size_dim0,
                                           int32_T *i2, real_T y[58]);

static __global__ void new_mvnpdf_kernel27(int32_T *i2, real_T y[58]);

static __global__ void new_mvnpdf_kernel28(real_T y[58], real_T *tol);

static __global__ void new_mvnpdf_kernel29(int32_T *info_t, real_T y[58],
                                           real_T *tol);

static __global__ void new_mvnpdf_kernel3(const real_T Sigma[3364],
                                          real_T x[3364]);

static __global__ void new_mvnpdf_kernel30(const real_T xRinv_data[580000],
                                           int32_T *i2, real_T y_data[580000]);

static __global__ void new_mvnpdf_kernel31(const real_T y_data[580000],
                                           real_T y[10000]);

static __global__ void new_mvnpdf_kernel32(const real_T y_data[580000],
                                           const int32_T *i2, real_T y[10000]);

static __global__ void new_mvnpdf_kernel33(real_T y[10000]);

static __global__ void new_mvnpdf_kernel34(real_T *tol, real_T y[10000]);

static __global__ void new_mvnpdf_kernel4(real_T x[3364], real_T y[3364]);

static __global__ void new_mvnpdf_kernel5(boolean_T y[58]);

static __global__ void new_mvnpdf_kernel6(const real_T y[3364],
                                          const int32_T initAuxVar, real_T *tol,
                                          boolean_T b_y[58]);

static __global__ void new_mvnpdf_kernel7(const real_T Sigma[3364],
                                          real_T x[3364]);

static __global__ void new_mvnpdf_kernel8(real_T x[3364]);

static __global__ void new_mvnpdf_kernel9(const int32_T T_size_dim0,
                                          const int32_T *rankA, int32_T *i2,
                                          real_T x[3364], real_T T_data[3364]);

static __device__ real_T shflDown2(real_T in1, uint32_T offset, uint32_T mask);

static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
                                              uint32_T mask);

static __device__ real_T workGroupReduction(real_T val, uint32_T mask,
                                            uint32_T numActiveWarps);

// Function Definitions
static __device__ real_T atomicOpreal_T(real_T *address, real_T value)
{
  unsigned long long int old;
  unsigned long long int *address_as_up;
  address_as_up = (unsigned long long int *)address;
  old = *address_as_up;
  unsigned long long int assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_up, old,
                    __double_as_longlong(value + __longlong_as_double(old)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

static void b_error(const mxArray *m, const mxArray *m1, emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  pArrays[0] = m;
  pArrays[1] = m1;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, nullptr, 2, &pArrays[0],
                        (const char_T *)"error", true, location);
}

static __global__ __launch_bounds__(512, 1) void new_mvnpdf_kernel1(
    const real_T Mu[58], const real_T X[580000], real_T X0[580000])
{
  uint64_T threadId;
  int32_T i14;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i14 = static_cast<int32_T>(threadId % 10000ULL);
  k = static_cast<int32_T>((threadId - static_cast<uint64_T>(i14)) / 10000ULL);
  if ((static_cast<int32_T>(k < 58)) && (static_cast<int32_T>(i14 < 10000))) {
    X0[i14 + 10000 * k] = X[i14 + 10000 * k] - Mu[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void new_mvnpdf_kernel10(
    const real_T T_data[3364], int32_T T_size, real_T A_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(T_size);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    A_data[i] = T_data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void new_mvnpdf_kernel11(
    const real_T T_data[3364], const int32_T T_size_dim0, const int32_T T_size,
    int32_T b_T_size, real_T A_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(T_size) + 1ULL) *
                (static_cast<uint64_T>(b_T_size) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T i14;
    i14 = static_cast<int32_T>(idx % (static_cast<uint64_T>(T_size) + 1ULL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(i14)) /
                             (static_cast<uint64_T>(T_size) + 1ULL));
    A_data[i14 + 58 * i] = T_data[i + T_size_dim0 * i14];
  }
}

static __global__
    __launch_bounds__(512, 1) void new_mvnpdf_kernel12(real_T X0[580000],
                                                       real_T B[580000])
{
  uint64_T threadId;
  int32_T i;
  int32_T i14;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i14 = static_cast<int32_T>(threadId % 58ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(i14)) / 58ULL);
  if ((static_cast<int32_T>(i < 10000)) && (static_cast<int32_T>(i14 < 58))) {
    B[i14 + 58 * i] = X0[i + 10000 * i14];
  }
}

static __global__ __launch_bounds__(1024, 1) void new_mvnpdf_kernel13(
    int32_T *i2, ptrdiff_t jpvt_t_data[58], int32_T ipiv_t[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*i2 - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    ipiv_t[i] = 0;
    jpvt_t_data[i] = (ptrdiff_t)0;
  }
}

static __global__ __launch_bounds__(1024, 1) void new_mvnpdf_kernel14(
    const ptrdiff_t jpvt_t_data[58], int32_T *i2, int32_T ipiv_t[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*i2 - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    ipiv_t[k] = (int32_T)jpvt_t_data[k];
  }
}

static __global__
    __launch_bounds__(1024, 1) void new_mvnpdf_kernel15(int32_T *i2,
                                                        real_T A_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 58ULL * (static_cast<uint64_T>(*i2 - 1) + 1ULL) - 1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % 58ULL);
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) / 58ULL);
    A_data[j * 58 + i] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void new_mvnpdf_kernel16(int32_T *i2,
                                                                real_T y[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*i2 - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y[k] = HIP_NAN;
  }
}

static __global__
    __launch_bounds__(1024, 1) void new_mvnpdf_kernel17(int32_T *i2,
                                                        int32_T ipiv_t[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*i2 - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    ipiv_t[k] = k + 1;
  }
}

static __global__
    __launch_bounds__(1024, 1) void new_mvnpdf_kernel18(int16_T iv2_idx_0,
                                                        real_T Y_data[580000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(static_cast<int32_T>(iv2_idx_0) * 10000 - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    Y_data[i] = 0.0;
  }
}

static __global__
    __launch_bounds__(512, 1) void new_mvnpdf_kernel19(real_T B[580000])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 580000) {
    B[i] = HIP_NAN;
  }
}

static __global__
    __launch_bounds__(64, 1) void new_mvnpdf_kernel2(const real_T Sigma[3364],
                                                     real_T y[58])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 58) {
    //  Assume identity covariance, data are already standardized
    //  Special case: if Sigma is supplied, then use it to try to interpret
    //  X and Mu as row vectors if they were both column vectors.
    // Check that sigma is the right size
    //  Make sure Sigma is a valid covariance matrix
    // CHOLCOV  Cholesky-like decomposition for covariance matrix.
    //    T = CHOLCOV(SIGMA) computes T such that SIGMA = T'*T.  SIGMA must be
    //    square, symmetric, and positive semi-definite.  If SIGMA is positive
    //    definite, then T is the square, upper triangular Cholesky factor.
    //
    //    If SIGMA is not positive definite, T is computed from an eigenvalue
    //    decomposition of SIGMA.  T is not necessarily triangular or square in
    //    this case.  Any eigenvectors whose corresponding eigenvalue is close
    //    to zero (within a small tolerance) are omitted.  If any remaining
    //    eigenvalues are negative, T is empty.
    //
    //    [T,P] = CHOLCOV(SIGMA) returns the number of negative eigenvalues of
    //    SIGMA, and T is empty if P>0.  If P==0, SIGMA is positive
    //    semi-definite.
    //
    //    If SIGMA is not square and symmetric, P is NaN and T is empty.
    //
    //    [T,P] = CHOLCOV(SIGMA,0) returns P==0 if SIGMA is positive definite,
    //    and T is the Cholesky factor.  If SIGMA is not positive definite, P is
    //    a positive integer and T is empty.  [...] = CHOLCOV(SIGMA,1) is
    //    equivalent to [...] = CHOLCOV(SIGMA).
    //
    //    Example:
    //    Factor a rank-deficient covariance matrix C.
    //        C = [2 1 1 2;1 2 1 2;1 1 2 2;2 2 2 3]
    //        T = cholcov(C)
    //        C2 = T'*T
    //    Generate data with this covariance (aside from random variation).
    //        C3 = cov(randn(10000,3)*T)
    //
    //    See also CHOL.
    //    Copyright 1993-2009 The MathWorks, Inc.
    //  Test for square, symmetric
    y[k] = fabs(Sigma[k + 58 * k]);
  }
}

static __global__ __launch_bounds__(512, 1) void new_mvnpdf_kernel20(
    const int32_T Y_size_dim0, const int32_T *rankA, real_T A_data[3364],
    real_T B[580000], int32_T ipiv_t[58], real_T Y_data[580000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 10000) {
    int32_T i;
    for (i = 0; i < *rankA; i++) {
      Y_data[(ipiv_t[i] + Y_size_dim0 * k) - 1] = B[i + 58 * k];
    }
    for (int32_T j{0}; j < *rankA; j++) {
      int32_T b_j;
      int32_T i14;
      b_j = *rankA - j;
      i = ipiv_t[b_j - 1];
      Y_data[(i + Y_size_dim0 * k) - 1] /= A_data[(b_j + 58 * (b_j - 1)) - 1];
      i14 = b_j - 1;
      for (i = 0; i < i14; i++) {
        Y_data[(ipiv_t[i] + Y_size_dim0 * k) - 1] -=
            Y_data[(ipiv_t[b_j - 1] + Y_size_dim0 * k) - 1] *
            A_data[i + 58 * (b_j - 1)];
      }
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void new_mvnpdf_kernel21(
    const real_T Y_data[580000], const int32_T Y_size_dim0, int32_T Y_size,
    real_T xRinv_data[580000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 10000ULL * (static_cast<uint64_T>(Y_size) + 1ULL) - 1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T i14;
    i14 = static_cast<int32_T>(idx % 10000ULL);
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(i14)) / 10000ULL);
    xRinv_data[i14 + 10000 * i] = Y_data[i + Y_size_dim0 * i14];
  }
}

static __global__
    __launch_bounds__(64, 1) void new_mvnpdf_kernel22(int32_T ipiv_t[58],
                                                      int32_T ipiv_data[58])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 58) {
    ipiv_data[k] = ipiv_t[k];
  }
}

static __global__
    __launch_bounds__(1024, 1) void new_mvnpdf_kernel23(int32_T Y_size,
                                                        real_T A_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(Y_size);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    A_data[i] = HIP_NAN;
  }
}

static __global__
    __launch_bounds__(64, 1) void new_mvnpdf_kernel24(int32_T ipiv_data[58])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 58) {
    ipiv_data[k] = k + 1;
  }
}

static __global__ __launch_bounds__(512, 1) void new_mvnpdf_kernel25(
    real_T X0[580000], real_T xRinv_data[580000])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 580000) {
    xRinv_data[i] = X0[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void new_mvnpdf_kernel26(
    const real_T T_data[3364], const int32_T T_size_dim0, int32_T *i2,
    real_T y[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*i2 - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y[k] = T_data[k + T_size_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void new_mvnpdf_kernel27(int32_T *i2,
                                                                real_T y[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*i2);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y[k] = log(y[k]);
  }
}

static __global__
    __launch_bounds__(32, 1) void new_mvnpdf_kernel28(real_T y[58], real_T *tol)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *tol = y[0];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void new_mvnpdf_kernel29(int32_T *info_t,
                                                                real_T y[58],
                                                                real_T *tol)
{
  int64_T loopEnd;
  real_T tmpRed0;
  uint32_T blockStride;
  uint32_T m;
  uint32_T thBlkId;
  uint32_T threadId;
  uint32_T threadStride;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  loopEnd = (static_cast<int64_T>(*info_t - 2) + 1LL) - 1LL;
  if (static_cast<uint32_T>(mwGetBlockIndex()) ==
      static_cast<uint32_T>(static_cast<int64_T>(*info_t - 2) + 1LL) /
          blockStride) {
    m = (static_cast<int64_T>(*info_t - 2) + 1LL) %
        static_cast<int64_T>(blockStride);
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (static_cast<int64_T>(threadId) <= loopEnd) {
    tmpRed0 = y[static_cast<int32_T>(threadId) + 1];
  }
  m = __ballot_sync(MAX_uint32_T, static_cast<int64_T>(threadId) <= loopEnd);
  for (uint32_T idx{threadId + threadStride};
       idx <= static_cast<uint32_T>(loopEnd); idx += threadStride) {
    tmpRed0 += y[static_cast<int32_T>(idx) + 1];
  }
  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if ((static_cast<int32_T>(static_cast<int64_T>(threadId) <= loopEnd)) &&
      (static_cast<int32_T>(thBlkId == 0U))) {
    atomicOpreal_T(&tol[0], tmpRed0);
  }
}

static __global__
    __launch_bounds__(512, 1) void new_mvnpdf_kernel3(const real_T Sigma[3364],
                                                      real_T x[3364])
{
  uint64_T threadId;
  int32_T i;
  int32_T i14;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i14 = static_cast<int32_T>(threadId % 58ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(i14)) / 58ULL);
  if ((static_cast<int32_T>(i < 58)) && (static_cast<int32_T>(i14 < 58))) {
    x[i14 + 58 * i] = Sigma[i14 + 58 * i] - Sigma[i + 58 * i14];
  }
}

static __global__ __launch_bounds__(1024, 1) void new_mvnpdf_kernel30(
    const real_T xRinv_data[580000], int32_T *i2, real_T y_data[580000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*i2);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    real_T d1;
    int32_T k;
    k = static_cast<int32_T>(idx);
    d1 = xRinv_data[k];
    y_data[k] = d1 * d1;
  }
}

static __global__ __launch_bounds__(512, 1) void new_mvnpdf_kernel31(
    const real_T y_data[580000], real_T y[10000])
{
  uint64_T threadId;
  int32_T j;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int32_T>(threadId);
  if (j < 10000) {
    y[j] = y_data[j];
  }
}

static __global__ __launch_bounds__(512, 1) void new_mvnpdf_kernel32(
    const real_T y_data[580000], const int32_T *i2, real_T y[10000])
{
  uint64_T threadId;
  int32_T j;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int32_T>(threadId);
  if (j < 10000) {
    y[j] += y_data[*i2 + j];
  }
}

static __global__ __launch_bounds__(512,
                                    1) void new_mvnpdf_kernel33(real_T y[10000])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 10000) {
    y[i] = 0.0;
  }
}

static __global__ __launch_bounds__(512,
                                    1) void new_mvnpdf_kernel34(real_T *tol,
                                                                real_T y[10000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 10000) {
    y[k] = exp((-0.5 * y[k] - *tol) - 53.298434925871014);
  }
}

static __global__ __launch_bounds__(512,
                                    1) void new_mvnpdf_kernel4(real_T x[3364],
                                                               real_T y[3364])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 3364) {
    y[k] = fabs(x[k]);
  }
}

static __global__ __launch_bounds__(64,
                                    1) void new_mvnpdf_kernel5(boolean_T y[58])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 58) {
    y[i] = true;
  }
}

static __global__ __launch_bounds__(64, 1) void new_mvnpdf_kernel6(
    const real_T y[3364], const int32_T initAuxVar, real_T *tol,
    boolean_T b_y[58])
{
  uint64_T threadId;
  int32_T vlen;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  vlen = static_cast<int32_T>(threadId);
  if (vlen < 58) {
    int32_T i2;
    int32_T info_t;
    boolean_T exitg1;
    i2 = initAuxVar + vlen * 58;
    info_t = i2 + 57;
    exitg1 = false;
    while ((!static_cast<int32_T>(exitg1)) &&
           (static_cast<int32_T>(i2 <= info_t))) {
      if (!static_cast<int32_T>(y[i2 - 1] < *tol)) {
        b_y[vlen] = false;
        exitg1 = true;
      } else {
        i2++;
      }
    }
  }
}

static __global__
    __launch_bounds__(512, 1) void new_mvnpdf_kernel7(const real_T Sigma[3364],
                                                      real_T x[3364])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 3364) {
    x[i] = Sigma[i];
  }
}

static __global__ __launch_bounds__(512,
                                    1) void new_mvnpdf_kernel8(real_T x[3364])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 3364) {
    x[i] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void new_mvnpdf_kernel9(
    const int32_T T_size_dim0, const int32_T *rankA, int32_T *i2,
    real_T x[3364], real_T T_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(*rankA) + 1ULL) *
                (static_cast<uint64_T>(*i2) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T i14;
    i14 = static_cast<int32_T>(idx % (static_cast<uint64_T>(*rankA) + 1ULL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(i14)) /
                             (static_cast<uint64_T>(*rankA) + 1ULL));
    T_data[i14 + T_size_dim0 * i] = x[i14 + 58 * i];
  }
}

static __device__ real_T shflDown2(real_T in1, uint32_T offset, uint32_T mask)
{
  int2 tmp;
  tmp.x = __shfl_down_sync(mask, ((int2 *)&in1)->x, offset);
  tmp.y = __shfl_down_sync(mask, ((int2 *)&in1)->y, offset);
  return *(real_T *)&tmp;
}

static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
                                              uint32_T mask)
{
  uint32_T activeSize;
  uint32_T offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    real_T other;
    other = shflDown2(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }
    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }
  return val;
}

static __device__ real_T workGroupReduction(real_T val, uint32_T mask,
                                            uint32_T numActiveWarps)
{
  __shared__ real_T shared[32];
  uint32_T lane;
  uint32_T thBlkId;
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  lane = thBlkId % warpSize;
  thBlkId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[thBlkId] = val;
  }
  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (thBlkId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }
  return val;
}

void new_mvnpdf(new_mvnpdfStackData *SD, const real_T X[580000],
                const real_T Mu[58], const real_T Sigma[3364], real_T y[10000])
{
  static const int32_T iv[2]{1, 26};
  static const int32_T iv1[2]{1, 46};
  static const char_T varargin_2[46]{
      'S', 'I', 'G', 'M', 'A', ' ', 'm', 'u', 's', 't', ' ', 'b',
      'e', ' ', 's', 'y', 'm', 'm', 'e', 't', 'r', 'i', 'c', ' ',
      'a', 'n', 'd', ' ', 'p', 'o', 's', 'i', 't', 'i', 'v', 'e',
      ' ', 'd', 'e', 'f', 'i', 'n', 'i', 't', 'e', '.'};
  static const char_T varargin_1[26]{
      's', 't', 'a', 't', 's', ':', 'm', 'v', 'n', 'p', 'd', 'f', ':',
      'B', 'a', 'd', 'C', 'o', 'v', 'a', 'r', 'i', 'a', 'n', 'c', 'e'};
  ptrdiff_t jpvt_t_data[58];
  ptrdiff_t(*gpu_jpvt_t_data)[58];
  dim3 block;
  dim3 grid;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *m;
  real_T(*gpu_B)[580000];
  real_T(*gpu_X)[580000];
  real_T(*gpu_X0)[580000];
  real_T(*gpu_Y_data)[580000];
  real_T(*gpu_xRinv_data)[580000];
  real_T(*gpu_y_data)[580000];
  real_T(*d_gpu_y)[10000];
  real_T A_data[3364];
  real_T x[3364];
  real_T(*b_gpu_y)[3364];
  real_T(*gpu_A_data)[3364];
  real_T(*gpu_Sigma)[3364];
  real_T(*gpu_T_data)[3364];
  real_T(*gpu_x)[3364];
  real_T b_y[58];
  real_T(*gpu_Mu)[58];
  real_T(*gpu_y)[58];
  real_T tol;
  real_T *gpu_tol;
  int32_T ipiv_data[58];
  int32_T(*gpu_ipiv_data)[58];
  int32_T(*gpu_ipiv_t)[58];
  int32_T T_size[2];
  int32_T i2;
  int32_T info_t;
  int32_T rankA;
  int32_T vlen;
  int32_T *gpu_i2;
  int32_T *gpu_info_t;
  int32_T *gpu_rankA;
  boolean_T c_y[58];
  boolean_T(*c_gpu_y)[58];
  boolean_T A_data_dirtyOnGpu;
  boolean_T exitg1;
  boolean_T jpvt_t_data_dirtyOnGpu;
  boolean_T p;
  boolean_T tol_dirtyOnCpu;
  boolean_T validLaunchParams;
  boolean_T y_dirtyOnGpu;
  hipMalloc(&gpu_Y_data, 4640000ULL);
  hipMalloc(&d_gpu_y, 80000ULL);
  hipMalloc(&gpu_y_data, 4640000ULL);
  hipMalloc(&gpu_xRinv_data, 4640000ULL);
  hipMalloc(&gpu_jpvt_t_data, 464ULL);
  hipMalloc(&gpu_B, 4640000ULL);
  hipMalloc(&gpu_ipiv_data, 232ULL);
  hipMalloc(&gpu_ipiv_t, 232ULL);
  hipMalloc(&gpu_i2, 4ULL);
  hipMalloc(&gpu_rankA, 4ULL);
  hipMalloc(&gpu_A_data, 26912ULL);
  hipMalloc(&gpu_T_data, 26912ULL);
  hipMalloc(&gpu_info_t, 4ULL);
  hipMalloc(&gpu_tol, 8ULL);
  hipMalloc(&c_gpu_y, 58ULL);
  hipMalloc(&b_gpu_y, 26912ULL);
  hipMalloc(&gpu_x, 26912ULL);
  hipMalloc(&gpu_y, 464ULL);
  hipMalloc(&gpu_Sigma, 26912ULL);
  hipMalloc(&gpu_X0, 4640000ULL);
  hipMalloc(&gpu_X, 4640000ULL);
  hipMalloc(&gpu_Mu, 464ULL);
  jpvt_t_data_dirtyOnGpu = false;
  A_data_dirtyOnGpu = false;
  // MVNPDF Multivariate normal probability density function (pdf).
  //    Y = MVNPDF(X) returns the probability density of the multivariate normal
  //    distribution with zero mean and identity covariance matrix, evaluated at
  //    each row of X.  Rows of the N-by-D matrix X correspond to observations
  //    or points, and columns correspond to variables or coordinates.  Y is an
  //    N-by-1 vector.
  //
  //    Y = MVNPDF(X,MU) returns the density of the multivariate normal
  //    distribution with mean MU and identity covariance matrix, evaluated
  //    at each row of X.  MU is a 1-by-D vector, or an N-by-D matrix, in which
  //    case the density is evaluated for each row of X with the corresponding
  //    row of MU.  MU can also be a scalar value, which MVNPDF replicates to
  //    match the size of X.
  //
  //    Y = MVNPDF(X,MU,SIGMA) returns the density of the multivariate normal
  //    distribution with mean MU and covariance SIGMA, evaluated at each row
  //    of X.  SIGMA is a D-by-D matrix, or an D-by-D-by-N array, in which case
  //    the density is evaluated for each row of X with the corresponding page
  //    of SIGMA, i.e., MVNPDF computes Y(I) using X(I,:) and SIGMA(:,:,I).
  //    If the covariance matrix is diagonal, containing variances along the
  //    diagonal and zero covariances off the diagonal, SIGMA may also be
  //    specified as a 1-by-D matrix or a 1-by-D-by-N array, containing
  //    just the diagonal. Pass in the empty matrix for MU to use its default
  //    value when you want to only specify SIGMA.
  //
  //    If X is a 1-by-D vector, MVNPDF replicates it to match the leading
  //    dimension of MU or the trailing dimension of SIGMA.
  //
  //    Example:
  //
  //       mu = [1 -1]; Sigma = [.9 .4; .4 .3];
  //       [X1,X2] = meshgrid(linspace(-1,3,25)', linspace(-3,1,25)');
  //       X = [X1(:) X2(:)];
  //       p = mvnpdf(X, mu, Sigma);
  //       surf(X1,X2,reshape(p,25,25));
  //
  //    See also MVTPDF, MVNCDF, MVNRND, NORMPDF.
  //    Copyright 1993-2008 The MathWorks, Inc.
  //    $Revision: 1.2.4.8 $  $Date: 2008/02/29 13:12:08 $
  //  Get size of data.  Column vectors provisionally interpreted as multiple
  //  scalar data. Assume zero mean, data are already centered mean is a single
  //  row, rep it out to match data
  hipMemcpy(*gpu_Mu, Mu, 464ULL, hipMemcpyHostToDevice);
  hipMemcpy(*gpu_X, X, 4640000ULL, hipMemcpyHostToDevice);
  new_mvnpdf_kernel1<<<dim3(1133U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_Mu, *gpu_X, *gpu_X0);
  //  Assume identity covariance, data are already standardized
  //  Special case: if Sigma is supplied, then use it to try to interpret
  //  X and Mu as row vectors if they were both column vectors.
  // Check that sigma is the right size
  //  Make sure Sigma is a valid covariance matrix
  // CHOLCOV  Cholesky-like decomposition for covariance matrix.
  //    T = CHOLCOV(SIGMA) computes T such that SIGMA = T'*T.  SIGMA must be
  //    square, symmetric, and positive semi-definite.  If SIGMA is positive
  //    definite, then T is the square, upper triangular Cholesky factor.
  //
  //    If SIGMA is not positive definite, T is computed from an eigenvalue
  //    decomposition of SIGMA.  T is not necessarily triangular or square in
  //    this case.  Any eigenvectors whose corresponding eigenvalue is close to
  //    zero (within a small tolerance) are omitted.  If any remaining
  //    eigenvalues are negative, T is empty.
  //
  //    [T,P] = CHOLCOV(SIGMA) returns the number of negative eigenvalues of
  //    SIGMA, and T is empty if P>0.  If P==0, SIGMA is positive semi-definite.
  //
  //    If SIGMA is not square and symmetric, P is NaN and T is empty.
  //
  //    [T,P] = CHOLCOV(SIGMA,0) returns P==0 if SIGMA is positive definite, and
  //    T is the Cholesky factor.  If SIGMA is not positive definite, P is a
  //    positive integer and T is empty.  [...] = CHOLCOV(SIGMA,1) is equivalent
  //    to [...] = CHOLCOV(SIGMA).
  //
  //    Example:
  //    Factor a rank-deficient covariance matrix C.
  //        C = [2 1 1 2;1 2 1 2;1 1 2 2;2 2 2 3]
  //        T = cholcov(C)
  //        C2 = T'*T
  //    Generate data with this covariance (aside from random variation).
  //        C3 = cov(randn(10000,3)*T)
  //
  //    See also CHOL.
  //    Copyright 1993-2009 The MathWorks, Inc.
  //  Test for square, symmetric
  hipMemcpy(*gpu_Sigma, Sigma, 26912ULL, hipMemcpyHostToDevice);
  new_mvnpdf_kernel2<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*gpu_Sigma,
                                                              *gpu_y);
  hipMemcpy(b_y, *gpu_y, 464ULL, hipMemcpyDeviceToHost);
  tol = b_y[0];
  for (vlen = 0; vlen < 57; vlen++) {
    real_T d;
    d = b_y[vlen + 1];
    if (std::isnan(d)) {
      p = false;
    } else if (std::isnan(tol)) {
      p = true;
    } else {
      p = (tol < d);
    }
    if (p) {
      tol = d;
    }
  }
  if ((!std::isinf(tol)) && (!std::isnan(tol))) {
    if (tol <= 2.2250738585072014E-308) {
      tol = 4.94065645841247E-324;
    } else {
      frexp(tol, &i2);
      tol = std::ldexp(1.0, i2 - 53);
    }
  } else {
    tol = rtNaN;
  }
  tol *= 10.0;
  new_mvnpdf_kernel3<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_Sigma,
                                                               *gpu_x);
  new_mvnpdf_kernel4<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_x,
                                                               *b_gpu_y);
  new_mvnpdf_kernel5<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*c_gpu_y);
  hipMemcpy(gpu_tol, &tol, 8ULL, hipMemcpyHostToDevice);
  new_mvnpdf_kernel6<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
      *b_gpu_y, 1, gpu_tol, *c_gpu_y);
  y_dirtyOnGpu = true;
  p = true;
  rankA = 0;
  exitg1 = false;
  while ((!exitg1) && (rankA < 58)) {
    if (y_dirtyOnGpu) {
      hipMemcpy(c_y, *c_gpu_y, 58ULL, hipMemcpyDeviceToHost);
    }
    y_dirtyOnGpu = false;
    if (!c_y[rankA]) {
      p = false;
      exitg1 = true;
    } else {
      rankA++;
    }
  }
  if (p) {
    new_mvnpdf_kernel7<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_Sigma,
                                                                 *gpu_x);
    hipsolverDnDpotrf_bufferSize(
        getCuSolverGlobalHandle(), HIPBLAS_FILL_MODE_UPPER, 58,
        (double *)&(*gpu_x)[0], 58, getCuSolverWorkspaceReq());
    setCuSolverWorkspaceTypeSize(8);
    cusolverInitWorkspace();
    hipsolverDnDpotrf(getCuSolverGlobalHandle(), HIPBLAS_FILL_MODE_UPPER, 58,
                     (double *)&(*gpu_x)[0], 58,
                     static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                     *getCuSolverWorkspaceReq(), gpu_info_t);
    y_dirtyOnGpu = false;
    p = true;
    hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
    if (info_t < 0) {
      new_mvnpdf_kernel8<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_x);
    }
    if (info_t == 0) {
      i2 = 56;
    } else {
      i2 = info_t - 3;
    }
    for (rankA = 0; rankA <= i2; rankA++) {
      for (vlen = 0; vlen <= i2 - rankA; vlen++) {
        if (p) {
          hipMemcpy(x, *gpu_x, 26912ULL, hipMemcpyDeviceToHost);
        }
        x[((rankA + vlen) + 58 * rankA) + 1] = 0.0;
        p = false;
        y_dirtyOnGpu = true;
      }
    }
    if (i2 + 2 < 1) {
      rankA = -1;
      i2 = -1;
    } else {
      rankA = i2 + 1;
      i2++;
    }
    T_size[0] = rankA + 1;
    T_size[1] = i2 + 1;
    validLaunchParams =
        mwGetLaunchParameters1D(static_cast<real_T>((rankA + 1LL) * (i2 + 1LL)),
                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(gpu_rankA, &rankA, 4ULL, hipMemcpyHostToDevice);
      hipMemcpy(gpu_i2, &i2, 4ULL, hipMemcpyHostToDevice);
      if (y_dirtyOnGpu) {
        hipMemcpy(*gpu_x, x, 26912ULL, hipMemcpyHostToDevice);
      }
      new_mvnpdf_kernel9<<<grid, block>>>(rankA + 1, gpu_rankA, gpu_i2, *gpu_x,
                                          *gpu_T_data);
    }
    tol = info_t;
    tol_dirtyOnCpu = true;
    if (info_t > 0) {
      //  Test for positive definiteness
      T_size[0] = 0;
      T_size[1] = 0;
    }
  } else {
    T_size[0] = 0;
    T_size[1] = 0;
    tol = rtNaN;
    tol_dirtyOnCpu = true;
  }
  if (tol != 0.0) {
    d_y = nullptr;
    m = emlrtCreateCharArray(2, &iv[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 26, m, &varargin_1[0]);
    emlrtAssign(&d_y, m);
    e_y = nullptr;
    m = emlrtCreateCharArray(2, &iv1[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 46, m, &varargin_2[0]);
    emlrtAssign(&e_y, m);
    b_error(d_y, e_y, &emlrtMCI);
  }
  //  Create array of standardized data, and compute log(sqrt(det(Sigma)))
  i2 = T_size[0] * T_size[1] - 1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(i2 + 1LL),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    new_mvnpdf_kernel10<<<grid, block>>>(*gpu_T_data, i2, *gpu_A_data);
    A_data_dirtyOnGpu = true;
  }
  if (T_size[0] == 0) {
    vlen = 0;
  } else if (T_size[0] == 58) {
    boolean_T i2_dirtyOnCpu;
    hipsolverDnDgetrf_bufferSize(getCuSolverGlobalHandle(), 58, 58,
                                (double *)&(*gpu_A_data)[0], 58,
                                getCuSolverWorkspaceReq());
    setCuSolverWorkspaceTypeSize(8);
    cusolverInitWorkspace();
    hipsolverDnDgetrf(getCuSolverGlobalHandle(), 58, 58,
                     (double *)&(*gpu_A_data)[0], 58,
                     static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                     &(*gpu_ipiv_t)[0], gpu_info_t);
    hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
    if (info_t < 0) {
      i2 = 58 * T_size[1] - 1;
      validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(i2 + 1LL),
                                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        new_mvnpdf_kernel23<<<grid, block>>>(i2, *gpu_A_data);
      }
      new_mvnpdf_kernel24<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
          *gpu_ipiv_data);
      p = true;
    } else {
      new_mvnpdf_kernel22<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
          *gpu_ipiv_t, *gpu_ipiv_data);
      p = true;
    }
    tol = 1.0;
    hipblasDtrsm(getCublasGlobalHandle(), HIPBLAS_SIDE_RIGHT,
                HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                10000, 58, (double *)&tol, (double *)&(*gpu_A_data)[0], 58,
                (double *)&(*gpu_X0)[0], 10000);
    tol = 1.0;
    hipblasDtrsm(getCublasGlobalHandle(), HIPBLAS_SIDE_RIGHT,
                HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, 10000,
                58, (double *)&tol, (double *)&(*gpu_A_data)[0], 58,
                (double *)&(*gpu_X0)[0], 10000);
    y_dirtyOnGpu = false;
    i2_dirtyOnCpu = true;
    for (rankA = 0; rankA < 57; rankA++) {
      if (p) {
        hipMemcpy(ipiv_data, *gpu_ipiv_data, 232ULL, hipMemcpyDeviceToHost);
      }
      p = false;
      i2 = ipiv_data[56 - rankA];
      if (i2 != 57 - rankA) {
        for (vlen = 0; vlen < 10000; vlen++) {
          if (i2_dirtyOnCpu) {
            hipMemcpy(SD->f0.X0, *gpu_X0, 4640000ULL, hipMemcpyDeviceToHost);
          }
          tol = SD->f0.X0[vlen + 10000 * (56 - rankA)];
          SD->f0.X0[vlen + 10000 * (56 - rankA)] =
              SD->f0.X0[vlen + 10000 * (i2 - 1)];
          SD->f0.X0[vlen + 10000 * (i2 - 1)] = tol;
          i2_dirtyOnCpu = false;
          y_dirtyOnGpu = true;
        }
      }
    }
    vlen = 58;
    if (y_dirtyOnGpu) {
      hipMemcpy(*gpu_X0, SD->f0.X0, 4640000ULL, hipMemcpyHostToDevice);
    }
    new_mvnpdf_kernel25<<<dim3(1133U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_X0, *gpu_xRinv_data);
  } else {
    ptrdiff_t b_info_t;
    boolean_T i2_dirtyOnCpu;
    i2 = T_size[0];
    i2_dirtyOnCpu = true;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>(((T_size[1] - 1) + 1LL) * ((T_size[0] - 1) + 1LL)),
        &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      new_mvnpdf_kernel11<<<grid, block>>>(
          *gpu_T_data, T_size[0], T_size[1] - 1, T_size[0] - 1, *gpu_A_data);
      A_data_dirtyOnGpu = true;
    }
    new_mvnpdf_kernel12<<<dim3(1133U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_X0,
                                                                     *gpu_B);
    validLaunchParams =
        mwGetLaunchParameters1D(static_cast<real_T>((T_size[0] - 1) + 1LL),
                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(gpu_i2, &T_size[0], 4ULL, hipMemcpyHostToDevice);
      i2_dirtyOnCpu = false;
      new_mvnpdf_kernel13<<<grid, block>>>(gpu_i2, *gpu_jpvt_t_data,
                                           *gpu_ipiv_t);
      jpvt_t_data_dirtyOnGpu = true;
    }
    if (A_data_dirtyOnGpu) {
      hipMemcpy(A_data, *gpu_A_data, 26912ULL, hipMemcpyDeviceToHost);
    }
    if (jpvt_t_data_dirtyOnGpu) {
      hipMemcpy(jpvt_t_data, *gpu_jpvt_t_data, 464ULL, hipMemcpyDeviceToHost);
    }
    b_info_t =
        LAPACKE_dgeqp3(102, (ptrdiff_t)58, (ptrdiff_t)T_size[0], &A_data[0],
                       (ptrdiff_t)58, &jpvt_t_data[0], &b_y[0]);
    p = true;
    A_data_dirtyOnGpu = false;
    y_dirtyOnGpu = true;
    if ((int32_T)b_info_t != 0) {
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>(58LL * ((T_size[0] - 1) + 1LL)), &grid, &block,
          1024U, 65535U);
      if (validLaunchParams) {
        if (i2_dirtyOnCpu) {
          hipMemcpy(gpu_i2, &T_size[0], 4ULL, hipMemcpyHostToDevice);
        }
        i2_dirtyOnCpu = false;
        hipMemcpy(*gpu_A_data, A_data, 26912ULL, hipMemcpyHostToDevice);
        new_mvnpdf_kernel15<<<grid, block>>>(gpu_i2, *gpu_A_data);
        y_dirtyOnGpu = false;
        A_data_dirtyOnGpu = true;
      }
      validLaunchParams =
          mwGetLaunchParameters1D(static_cast<real_T>((T_size[0] - 1) + 1LL),
                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        if (i2_dirtyOnCpu) {
          hipMemcpy(gpu_i2, &T_size[0], 4ULL, hipMemcpyHostToDevice);
        }
        i2_dirtyOnCpu = false;
        hipMemcpy(*gpu_y, b_y, 464ULL, hipMemcpyHostToDevice);
        new_mvnpdf_kernel16<<<grid, block>>>(gpu_i2, *gpu_y);
        p = false;
      }
      validLaunchParams =
          mwGetLaunchParameters1D(static_cast<real_T>((T_size[0] - 1) + 1LL),
                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        if (i2_dirtyOnCpu) {
          hipMemcpy(gpu_i2, &T_size[0], 4ULL, hipMemcpyHostToDevice);
        }
        new_mvnpdf_kernel17<<<grid, block>>>(gpu_i2, *gpu_ipiv_t);
      }
    } else {
      validLaunchParams =
          mwGetLaunchParameters1D(static_cast<real_T>((T_size[0] - 1) + 1LL),
                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        hipMemcpy(*gpu_jpvt_t_data, jpvt_t_data, 464ULL,
                   hipMemcpyHostToDevice);
        if (i2_dirtyOnCpu) {
          hipMemcpy(gpu_i2, &T_size[0], 4ULL, hipMemcpyHostToDevice);
        }
        new_mvnpdf_kernel14<<<grid, block>>>(*gpu_jpvt_t_data, gpu_i2,
                                             *gpu_ipiv_t);
      }
    }
    rankA = 0;
    if (A_data_dirtyOnGpu) {
      hipMemcpy(A_data, *gpu_A_data, 26912ULL, hipMemcpyDeviceToHost);
    }
    tol = 1.2878587085651816E-13 * std::abs(A_data[0]);
    while ((rankA < i2) && (!(std::abs(A_data[rankA + 58 * rankA]) <= tol))) {
      rankA++;
    }
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((static_cast<int16_T>(T_size[0]) * 10000 - 1) +
                            1LL),
        &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      new_mvnpdf_kernel18<<<grid, block>>>(static_cast<int16_T>(T_size[0]),
                                           *gpu_Y_data);
    }
    info_t = 0;
    if (y_dirtyOnGpu) {
      hipMemcpy(*gpu_A_data, A_data, 26912ULL, hipMemcpyHostToDevice);
    }
    if (p) {
      hipMemcpy(*gpu_y, b_y, 464ULL, hipMemcpyHostToDevice);
    }
    hipsolverDnDormqr_bufferSize(
        getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, 58, 10000,
        T_size[0], (double *)&(*gpu_A_data)[0], 58, (double *)&(*gpu_y)[0],
        (double *)&(*gpu_B)[0], 58, getCuSolverWorkspaceReq());
    setCuSolverWorkspaceTypeSize(8);
    cusolverInitWorkspace();
    hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
    hipsolverDnDormqr(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                     58, 10000, T_size[0], (double *)&(*gpu_A_data)[0], 58,
                     (double *)&(*gpu_y)[0], (double *)&(*gpu_B)[0], 58,
                     static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                     *getCuSolverWorkspaceReq(), gpu_info_t);
    hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
    if (info_t < 0) {
      new_mvnpdf_kernel19<<<dim3(1133U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_B);
    }
    hipMemcpy(gpu_rankA, &rankA, 4ULL, hipMemcpyHostToDevice);
    new_mvnpdf_kernel20<<<dim3(20U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        T_size[0], gpu_rankA, *gpu_A_data, *gpu_B, *gpu_ipiv_t, *gpu_Y_data);
    vlen = T_size[0];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>(10000LL * ((T_size[0] - 1) + 1LL)), &grid, &block,
        1024U, 65535U);
    if (validLaunchParams) {
      new_mvnpdf_kernel21<<<grid, block>>>(*gpu_Y_data, T_size[0],
                                           T_size[0] - 1, *gpu_xRinv_data);
    }
  }
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>((T_size[0] - 1) + 1LL), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(gpu_i2, &T_size[0], 4ULL, hipMemcpyHostToDevice);
    new_mvnpdf_kernel26<<<grid, block>>>(*gpu_T_data, T_size[0], gpu_i2,
                                         *gpu_y);
  }
  i2 = T_size[0] - 1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>((T_size[0] - 1) + 1LL), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(gpu_i2, &i2, 4ULL, hipMemcpyHostToDevice);
    new_mvnpdf_kernel27<<<grid, block>>>(gpu_i2, *gpu_y);
  }
  if (T_size[0] == 0) {
    tol = 0.0;
  } else {
    hipMemcpy(gpu_tol, &tol, 8ULL, hipMemcpyHostToDevice);
    new_mvnpdf_kernel28<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_y,
                                                                 gpu_tol);
    tol_dirtyOnCpu = false;
    validLaunchParams =
        mwGetLaunchParameters(static_cast<real_T>((T_size[0] - 2) + 1LL), &grid,
                              &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(gpu_info_t, &T_size[0], 4ULL, hipMemcpyHostToDevice);
      new_mvnpdf_kernel29<<<grid, block>>>(gpu_info_t, *gpu_y, gpu_tol);
    }
  }
  //  The quadratic form is the inner products of the standardized data
  i2 = 10000 * vlen - 1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(i2 + 1LL),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(gpu_i2, &i2, 4ULL, hipMemcpyHostToDevice);
    new_mvnpdf_kernel30<<<grid, block>>>(*gpu_xRinv_data, gpu_i2, *gpu_y_data);
  }
  if (vlen == 0) {
    new_mvnpdf_kernel33<<<dim3(20U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*d_gpu_y);
  } else {
    new_mvnpdf_kernel31<<<dim3(20U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_y_data,
                                                                   *d_gpu_y);
    for (rankA = 0; rankA <= vlen - 2; rankA++) {
      i2 = (rankA + 1) * 10000;
      hipMemcpy(gpu_i2, &i2, 4ULL, hipMemcpyHostToDevice);
      new_mvnpdf_kernel32<<<dim3(20U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          *gpu_y_data, gpu_i2, *d_gpu_y);
    }
  }
  if (tol_dirtyOnCpu) {
    hipMemcpy(gpu_tol, &tol, 8ULL, hipMemcpyHostToDevice);
  }
  new_mvnpdf_kernel34<<<dim3(20U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_tol,
                                                                 *d_gpu_y);
  hipMemcpy(y, *d_gpu_y, 80000ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_Mu);
  hipFree(*gpu_X);
  hipFree(*gpu_X0);
  hipFree(*gpu_Sigma);
  hipFree(*gpu_y);
  hipFree(*gpu_x);
  hipFree(*b_gpu_y);
  hipFree(*c_gpu_y);
  hipFree(gpu_tol);
  hipFree(gpu_info_t);
  hipFree(*gpu_T_data);
  hipFree(*gpu_A_data);
  hipFree(gpu_rankA);
  hipFree(gpu_i2);
  hipFree(*gpu_ipiv_t);
  hipFree(*gpu_ipiv_data);
  hipFree(*gpu_B);
  hipFree(*gpu_jpvt_t_data);
  hipFree(*gpu_xRinv_data);
  hipFree(*gpu_y_data);
  hipFree(*d_gpu_y);
  hipFree(*gpu_Y_data);
}

// End of code generation (new_mvnpdf.cu)
