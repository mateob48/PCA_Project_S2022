#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// betaCalc.cu
//
// Code generation for function 'betaCalc'
//

// Include files
#include "betaCalc.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"

// Function Declarations
static __device__ real_T atomicOpreal_T(real_T *address, real_T value);

static __global__ void betaCalc_kernel1(const real_T beta1[40000],
                                        const int32_T i1, const real_T B[40000],
                                        const int32_T i, const real_T trans[16],
                                        const int32_T b_i, real_T x[4]);

static __global__ void betaCalc_kernel2(const real_T x[4], real_T *y);

static __global__ void betaCalc_kernel3(const real_T scale1[10000],
                                        const real_T *y, const int32_T t,
                                        const int32_T i, real_T beta1[40000]);

static __global__ void betaCalc_kernel4(const real_T beta1[40000],
                                        real_T beta[40000]);

static __device__ real_T shflDown2(real_T in1, uint32_T offset, uint32_T mask);

static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
                                              uint32_T mask);

static __device__ real_T workGroupReduction(real_T val, uint32_T mask,
                                            uint32_T numActiveWarps);

// Function Definitions
static __device__ real_T atomicOpreal_T(real_T *address, real_T value)
{
  unsigned long long int old;
  unsigned long long int *address_as_up;
  address_as_up = (unsigned long long int *)address;
  old = *address_as_up;
  unsigned long long int assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_up, old,
                    __double_as_longlong(value + __longlong_as_double(old)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

static __global__ __launch_bounds__(32, 1) void betaCalc_kernel1(
    const real_T beta1[40000], const int32_T i1, const real_T B[40000],
    const int32_T i, const real_T trans[16], const int32_T b_i, real_T x[4])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 4) {
    x[k] = trans[b_i + (k << 2)] * B[i + 10000 * k] * beta1[i1 + 10000 * k];
  }
}

static __global__
    __launch_bounds__(32, 1) void betaCalc_kernel2(const real_T x[4], real_T *y)
{
  real_T tmpRed0;
  uint32_T blockStride;
  uint32_T mask;
  uint32_T thBlkId;
  uint32_T threadId;
  uint32_T threadStride;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  if (static_cast<uint32_T>(mwGetBlockIndex()) == 4U / blockStride) {
    int32_T m;
    m = static_cast<int32_T>(4U % blockStride);
    if (static_cast<uint32_T>(m) > 0U) {
      blockStride = static_cast<uint32_T>(m);
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 3U) {
    tmpRed0 = x[static_cast<int32_T>(threadId)];
  }
  mask = __ballot_sync(MAX_uint32_T, threadId <= 3U);
  for (uint32_T idx{threadId + threadStride}; idx <= 3U; idx += threadStride) {
    tmpRed0 += x[static_cast<int32_T>(idx)];
  }
  tmpRed0 = workGroupReduction(tmpRed0, mask, blockStride);
  if ((static_cast<int32_T>(threadId <= 3U)) &&
      (static_cast<int32_T>(thBlkId == 0U))) {
    atomicOpreal_T(&y[0], tmpRed0);
  }
}

static __global__ __launch_bounds__(32, 1) void betaCalc_kernel3(
    const real_T scale1[10000], const real_T *y, const int32_T t,
    const int32_T i, real_T beta1[40000])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    beta1[(10000 * i - t) + 9998] = *y / scale1[9998 - t];
  }
}

static __global__
    __launch_bounds__(512, 1) void betaCalc_kernel4(const real_T beta1[40000],
                                                    real_T beta[40000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 40000) {
    beta[k] = beta1[k];
  }
}

static __device__ real_T shflDown2(real_T in1, uint32_T offset, uint32_T mask)
{
  int2 tmp;
  tmp.x = __shfl_down_sync(mask, ((int2 *)&in1)->x, offset);
  tmp.y = __shfl_down_sync(mask, ((int2 *)&in1)->y, offset);
  return *(real_T *)&tmp;
}

static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
                                              uint32_T mask)
{
  uint32_T activeSize;
  uint32_T offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    real_T other;
    other = shflDown2(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }
    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }
  return val;
}

static __device__ real_T workGroupReduction(real_T val, uint32_T mask,
                                            uint32_T numActiveWarps)
{
  __shared__ real_T shared[32];
  uint32_T lane;
  uint32_T thBlkId;
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  lane = thBlkId % warpSize;
  thBlkId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[thBlkId] = val;
  }
  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (thBlkId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }
  return val;
}

void betaCalc(real_T beta1[40000], const real_T scale1[10000],
              const real_T trans[16], const real_T B[40000], real_T N,
              real_T beta[40000])
{
  real_T(*gpu_B)[40000];
  real_T(*gpu_beta)[40000];
  real_T(*gpu_beta1)[40000];
  real_T(*gpu_scale1)[10000];
  real_T(*gpu_trans)[16];
  real_T(*gpu_x)[4];
  real_T y;
  real_T *gpu_y;
  int32_T i;
  boolean_T B_dirtyOnCpu;
  boolean_T beta1_dirtyOnCpu;
  boolean_T scale1_dirtyOnCpu;
  boolean_T trans_dirtyOnCpu;
  hipMalloc(&gpu_scale1, 80000ULL);
  hipMalloc(&gpu_y, 8ULL);
  hipMalloc(&gpu_x, 32ULL);
  hipMalloc(&gpu_trans, 128ULL);
  hipMalloc(&gpu_B, 320000ULL);
  hipMalloc(&gpu_beta, 320000ULL);
  hipMalloc(&gpu_beta1, 320000ULL);
  scale1_dirtyOnCpu = true;
  trans_dirtyOnCpu = true;
  B_dirtyOnCpu = true;
  beta1_dirtyOnCpu = true;
  i = static_cast<int32_T>(N);
  for (int32_T t{0}; t < 9999; t++) {
    for (int32_T b_i{0}; b_i < i; b_i++) {
      if (beta1_dirtyOnCpu) {
        hipMemcpy(*gpu_beta1, beta1, 320000ULL, hipMemcpyHostToDevice);
      }
      if (B_dirtyOnCpu) {
        hipMemcpy(*gpu_B, B, 320000ULL, hipMemcpyHostToDevice);
      }
      B_dirtyOnCpu = false;
      if (trans_dirtyOnCpu) {
        hipMemcpy(*gpu_trans, trans, 128ULL, hipMemcpyHostToDevice);
      }
      trans_dirtyOnCpu = false;
      betaCalc_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          *gpu_beta1, 9999 - t, *gpu_B, 9999 - t, *gpu_trans, b_i, *gpu_x);
      y = 0.0;
      hipMemcpy(gpu_y, &y, 8ULL, hipMemcpyHostToDevice);
      betaCalc_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_x, gpu_y);
      if (scale1_dirtyOnCpu) {
        hipMemcpy(*gpu_scale1, scale1, 80000ULL, hipMemcpyHostToDevice);
      }
      scale1_dirtyOnCpu = false;
      betaCalc_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          *gpu_scale1, gpu_y, t, b_i, *gpu_beta1);
      beta1_dirtyOnCpu = false;
    }
  }
  if (beta1_dirtyOnCpu) {
    hipMemcpy(*gpu_beta1, beta1, 320000ULL, hipMemcpyHostToDevice);
  }
  betaCalc_kernel4<<<dim3(79U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_beta1,
                                                              *gpu_beta);
  hipMemcpy(beta, *gpu_beta, 320000ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_beta1);
  hipFree(*gpu_beta);
  hipFree(*gpu_B);
  hipFree(*gpu_trans);
  hipFree(*gpu_x);
  hipFree(gpu_y);
  hipFree(*gpu_scale1);
}

// End of code generation (betaCalc.cu)
