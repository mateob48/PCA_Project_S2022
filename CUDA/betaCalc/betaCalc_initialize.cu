//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// betaCalc_initialize.cu
//
// Code generation for function 'betaCalc_initialize'
//

// Include files
#include "betaCalc_initialize.h"
#include "_coder_betaCalc_mex.h"
#include "betaCalc_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void betaCalc_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(
      emlrtRootTLSGlobal,
      (const char_T *)"EMLRT:runTime:MexFunctionNeedsLicense",
      (const char_T *)"distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (betaCalc_initialize.cu)
