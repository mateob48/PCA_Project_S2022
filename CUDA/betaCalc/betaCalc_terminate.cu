//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// betaCalc_terminate.cu
//
// Code generation for function 'betaCalc_terminate'
//

// Include files
#include "betaCalc_terminate.h"
#include "_coder_betaCalc_mex.h"
#include "betaCalc_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void betaCalc_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void betaCalc_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (betaCalc_terminate.cu)
