//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_betaCalc_api.cu
//
// Code generation for function '_coder_betaCalc_api'
//

// Include files
#include "_coder_betaCalc_api.h"
#include "betaCalc.h"
#include "betaCalc_data.h"
#include "rt_nonfinite.h"

// Function Declarations
static real_T (*b_emlrt_marshallIn(const mxArray *scale1,
                                   const char_T *identifier))[10000];

static real_T (*b_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[10000];

static real_T (*c_emlrt_marshallIn(const mxArray *trans,
                                   const char_T *identifier))[16];

static real_T (*c_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[16];

static real_T d_emlrt_marshallIn(const mxArray *N, const char_T *identifier);

static real_T d_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId);

static real_T (*e_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[40000];

static real_T (*emlrt_marshallIn(const mxArray *beta1,
                                 const char_T *identifier))[40000];

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[40000];

static const mxArray *emlrt_marshallOut(const real_T u[40000]);

static real_T (*f_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[10000];

static real_T (*g_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[16];

static real_T h_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId);

// Function Definitions
static real_T (*b_emlrt_marshallIn(const mxArray *scale1,
                                   const char_T *identifier))[10000]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[10000];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(scale1), &thisId);
  emlrtDestroyArray(&scale1);
  return y;
}

static real_T (*b_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[10000]
{
  real_T(*y)[10000];
  y = f_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T (*c_emlrt_marshallIn(const mxArray *trans,
                                   const char_T *identifier))[16]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[16];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = c_emlrt_marshallIn(emlrtAlias(trans), &thisId);
  emlrtDestroyArray(&trans);
  return y;
}

static real_T (*c_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[16]
{
  real_T(*y)[16];
  y = g_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T d_emlrt_marshallIn(const mxArray *N, const char_T *identifier)
{
  emlrtMsgIdentifier thisId;
  real_T y;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = d_emlrt_marshallIn(emlrtAlias(N), &thisId);
  emlrtDestroyArray(&N);
  return y;
}

static real_T d_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId)
{
  real_T y;
  y = h_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T (*e_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[40000]
{
  static const int32_T dims[2]{10000, 4};
  real_T(*ret)[40000];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  ret = (real_T(*)[40000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*emlrt_marshallIn(const mxArray *beta1,
                                 const char_T *identifier))[40000]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[40000];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = emlrt_marshallIn(emlrtAlias(beta1), &thisId);
  emlrtDestroyArray(&beta1);
  return y;
}

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[40000]
{
  real_T(*y)[40000];
  y = e_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static const mxArray *emlrt_marshallOut(const real_T u[40000])
{
  static const int32_T iv[2]{0, 0};
  static const int32_T iv1[2]{10000, 4};
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateNumericArray(2, (const void *)&iv[0], mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m, (void *)&u[0]);
  emlrtSetDimensions((mxArray *)m, &iv1[0], 2);
  emlrtAssign(&y, m);
  return y;
}

static real_T (*f_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[10000]
{
  static const int32_T dims[2]{1, 10000};
  real_T(*ret)[10000];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  ret = (real_T(*)[10000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*g_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[16]
{
  static const int32_T dims[2]{4, 4};
  real_T(*ret)[16];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  ret = (real_T(*)[16])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T h_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 0U, (void *)&dims);
  ret = *static_cast<real_T *>(emlrtMxGetData(src));
  emlrtDestroyArray(&src);
  return ret;
}

void betaCalc_api(const mxArray *const prhs[5], const mxArray *plhs[1])
{
  const mxArray *prhs_copy_idx_0;
  real_T(*B)[40000];
  real_T(*beta)[40000];
  real_T(*beta1)[40000];
  real_T(*scale1)[10000];
  real_T(*trans)[16];
  real_T N;
  beta = (real_T(*)[40000])mxMalloc(sizeof(real_T[40000]));
  prhs_copy_idx_0 = emlrtProtectR2012b(prhs[0], 0, false, -1);
  // Marshall function inputs
  beta1 = emlrt_marshallIn(emlrtAlias(prhs_copy_idx_0), "beta1");
  scale1 = b_emlrt_marshallIn(emlrtAlias(prhs[1]), "scale1");
  trans = c_emlrt_marshallIn(emlrtAlias(prhs[2]), "trans");
  B = emlrt_marshallIn(emlrtAlias(prhs[3]), "B");
  N = d_emlrt_marshallIn(emlrtAliasP(prhs[4]), "N");
  // Invoke the target function
  betaCalc(*beta1, *scale1, *trans, *B, N, *beta);
  // Marshall function outputs
  plhs[0] = emlrt_marshallOut(*beta);
}

// End of code generation (_coder_betaCalc_api.cu)
