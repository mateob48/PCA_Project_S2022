//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_betaCalc_mex.cu
//
// Code generation for function '_coder_betaCalc_mex'
//

// Include files
#include "_coder_betaCalc_mex.h"
#include "_coder_betaCalc_api.h"
#include "betaCalc_data.h"
#include "betaCalc_initialize.h"
#include "betaCalc_terminate.h"
#include "rt_nonfinite.h"
#include <stdexcept>

void emlrtExceptionBridge();
void emlrtExceptionBridge()
{
  throw std::runtime_error("");
}
// Function Definitions
void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs,
                 const mxArray *prhs[])
{
  mexAtExit(&betaCalc_atexit);
  // Module initialization.
  betaCalc_initialize();
  try {
    emlrtShouldCleanupOnError((emlrtCTX *)emlrtRootTLSGlobal, false);
    // Dispatch the entry-point.
    unsafe_betaCalc_mexFunction(nlhs, plhs, nrhs, prhs);
    // Module termination.
    betaCalc_terminate();
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLSR2022a(&emlrtRootTLSGlobal, &emlrtContextGlobal, nullptr, 1,
                           (void *)&emlrtExceptionBridge,
                           (const char_T *)"windows-1252", true);
  return emlrtRootTLSGlobal;
}

void unsafe_betaCalc_mexFunction(int32_T nlhs, mxArray *plhs[1], int32_T nrhs,
                                 const mxArray *prhs[5])
{
  const mxArray *outputs[1];
  // Check for proper number of arguments.
  if (nrhs != 5) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 5, 4, 8, "betaCalc");
  }
  if (nlhs > 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 8,
                        "betaCalc");
  }
  // Call the function.
  betaCalc_api(prhs, outputs);
  // Copy over outputs to the caller.
  emlrtReturnArrays(1, &plhs[0], &outputs[0]);
}

// End of code generation (_coder_betaCalc_mex.cu)
