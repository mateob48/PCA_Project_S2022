//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// expectation_terminate.cu
//
// Code generation for function 'expectation_terminate'
//

// Include files
#include "expectation_terminate.h"
#include "_coder_expectation_mex.h"
#include "expectation_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void expectation_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void expectation_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (expectation_terminate.cu)
