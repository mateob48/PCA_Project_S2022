//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_expectation_mex.cu
//
// Code generation for function '_coder_expectation_mex'
//

// Include files
#include "_coder_expectation_mex.h"
#include "_coder_expectation_api.h"
#include "expectation_data.h"
#include "expectation_initialize.h"
#include "expectation_terminate.h"
#include "rt_nonfinite.h"
#include <stdexcept>

void emlrtExceptionBridge();
void emlrtExceptionBridge()
{
  throw std::runtime_error("");
}
// Function Definitions
void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs,
                 const mxArray *prhs[])
{
  mexAtExit(&expectation_atexit);
  // Module initialization.
  expectation_initialize();
  try {
    emlrtShouldCleanupOnError((emlrtCTX *)emlrtRootTLSGlobal, false);
    // Dispatch the entry-point.
    unsafe_expectation_mexFunction(nlhs, plhs, nrhs, prhs);
    // Module termination.
    expectation_terminate();
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLSR2022a(&emlrtRootTLSGlobal, &emlrtContextGlobal, nullptr, 1,
                           (void *)&emlrtExceptionBridge,
                           (const char_T *)"windows-1252", true);
  return emlrtRootTLSGlobal;
}

void unsafe_expectation_mexFunction(int32_T nlhs, mxArray *plhs[2],
                                    int32_T nrhs, const mxArray *prhs[6])
{
  const mxArray *outputs[2];
  int32_T b_nlhs;
  // Check for proper number of arguments.
  if (nrhs != 6) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 6, 4, 11, "expectation");
  }
  if (nlhs > 2) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 11,
                        "expectation");
  }
  // Call the function.
  expectation_api(prhs, nlhs, outputs);
  // Copy over outputs to the caller.
  if (nlhs < 1) {
    b_nlhs = 1;
  } else {
    b_nlhs = nlhs;
  }
  emlrtReturnArrays(b_nlhs, &plhs[0], &outputs[0]);
}

// End of code generation (_coder_expectation_mex.cu)
