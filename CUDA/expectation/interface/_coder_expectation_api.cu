//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_expectation_api.cu
//
// Code generation for function '_coder_expectation_api'
//

// Include files
#include "_coder_expectation_api.h"
#include "expectation.h"
#include "expectation_data.h"
#include "expectation_emxutil.h"
#include "expectation_types.h"
#include "rt_nonfinite.h"

// Variable Definitions
static emlrtRTEInfo f_emlrtRTEI{
    1,                        // lineNo
    1,                        // colNo
    "_coder_expectation_api", // fName
    ""                        // pName
};

// Function Declarations
static real_T (*b_emlrt_marshallIn(const mxArray *trans,
                                   const char_T *identifier))[16];

static real_T (*b_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[16];

static const mxArray *b_emlrt_marshallOut(const emxArray_real_T *u);

static real_T c_emlrt_marshallIn(const mxArray *T, const char_T *identifier);

static real_T c_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId);

static real_T (*d_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[40000];

static real_T (*e_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[16];

static real_T (*emlrt_marshallIn(const mxArray *alpha,
                                 const char_T *identifier))[40000];

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[40000];

static const mxArray *emlrt_marshallOut(const emxArray_real_T *u);

static real_T f_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId);

// Function Definitions
static real_T (*b_emlrt_marshallIn(const mxArray *trans,
                                   const char_T *identifier))[16]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[16];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(trans), &thisId);
  emlrtDestroyArray(&trans);
  return y;
}

static real_T (*b_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[16]
{
  real_T(*y)[16];
  y = e_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static const mxArray *b_emlrt_marshallOut(const emxArray_real_T *u)
{
  static const int32_T iv[2]{0, 0};
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateNumericArray(2, (const void *)&iv[0], mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m, &u->data[0]);
  emlrtSetDimensions((mxArray *)m, &u->size[0], 2);
  emlrtAssign(&y, m);
  return y;
}

static real_T c_emlrt_marshallIn(const mxArray *T, const char_T *identifier)
{
  emlrtMsgIdentifier thisId;
  real_T y;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = c_emlrt_marshallIn(emlrtAlias(T), &thisId);
  emlrtDestroyArray(&T);
  return y;
}

static real_T c_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId)
{
  real_T y;
  y = f_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T (*d_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[40000]
{
  static const int32_T dims[2]{10000, 4};
  real_T(*ret)[40000];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  ret = (real_T(*)[40000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*e_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[16]
{
  static const int32_T dims[2]{4, 4};
  real_T(*ret)[16];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  ret = (real_T(*)[16])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*emlrt_marshallIn(const mxArray *alpha,
                                 const char_T *identifier))[40000]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[40000];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = emlrt_marshallIn(emlrtAlias(alpha), &thisId);
  emlrtDestroyArray(&alpha);
  return y;
}

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[40000]
{
  real_T(*y)[40000];
  y = d_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static const mxArray *emlrt_marshallOut(const emxArray_real_T *u)
{
  static const int32_T iv[3]{0, 0, 0};
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateNumericArray(3, (const void *)&iv[0], mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m, &u->data[0]);
  emlrtSetDimensions((mxArray *)m, &u->size[0], 3);
  emlrtAssign(&y, m);
  return y;
}

static real_T f_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 0U, (void *)&dims);
  ret = *static_cast<real_T *>(emlrtMxGetData(src));
  emlrtDestroyArray(&src);
  return ret;
}

void expectation_api(const mxArray *const prhs[6], int32_T nlhs,
                     const mxArray *plhs[2])
{
  emxArray_real_T *Gamma;
  emxArray_real_T *Xi;
  real_T(*B)[40000];
  real_T(*alpha)[40000];
  real_T(*beta)[40000];
  real_T(*trans)[16];
  real_T N;
  real_T T;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&Xi, 3, &f_emlrtRTEI, true);
  emxInit_real_T(&Gamma, 2, &f_emlrtRTEI, true);
  // Marshall function inputs
  alpha = emlrt_marshallIn(emlrtAlias(prhs[0]), "alpha");
  trans = b_emlrt_marshallIn(emlrtAlias(prhs[1]), "trans");
  B = emlrt_marshallIn(emlrtAlias(prhs[2]), "B");
  beta = emlrt_marshallIn(emlrtAlias(prhs[3]), "beta");
  T = c_emlrt_marshallIn(emlrtAliasP(prhs[4]), "T");
  N = c_emlrt_marshallIn(emlrtAliasP(prhs[5]), "N");
  // Invoke the target function
  expectation(*alpha, *trans, *B, *beta, T, N, Xi, Gamma);
  // Marshall function outputs
  Xi->canFreeData = false;
  plhs[0] = emlrt_marshallOut(Xi);
  emxFree_real_T(&Xi);
  if (nlhs > 1) {
    Gamma->canFreeData = false;
    plhs[1] = b_emlrt_marshallOut(Gamma);
  }
  emxFree_real_T(&Gamma);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

// End of code generation (_coder_expectation_api.cu)
