//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// expectation_initialize.cu
//
// Code generation for function 'expectation_initialize'
//

// Include files
#include "expectation_initialize.h"
#include "_coder_expectation_mex.h"
#include "expectation_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void expectation_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(
      emlrtRootTLSGlobal,
      (const char_T *)"EMLRT:runTime:MexFunctionNeedsLicense",
      (const char_T *)"distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (expectation_initialize.cu)
