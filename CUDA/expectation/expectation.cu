#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// expectation.cu
//
// Code generation for function 'expectation'
//

// Include files
#include "expectation.h"
#include "expectation_data.h"
#include "expectation_emxutil.h"
#include "expectation_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cstring>

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    3,             // lineNo
    5,             // colNo
    "expectation", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\expectation.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    13,            // lineNo
    5,             // colNo
    "expectation", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\expectation.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    170,                     // lineNo
    24,                      // colNo
    "combineVectorElements", // fName
    "C:\\Program "
    "Files\\MATLAB\\R2022a\\toolbox\\eml\\lib\\matlab\\datafun\\private\\combin"
    "eVectorElements.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    10,            // lineNo
    9,             // colNo
    "expectation", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\expectation.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    11,            // lineNo
    19,            // colNo
    "expectation", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\expectation.m" // pName
};

// Function Declarations
static __global__ void expectation_kernel1(real_T T, real_T N,
                                           emxArray_real_T Xi);

static __global__ void expectation_kernel2(
    const real_T beta[40000], const real_T B[40000], const real_T trans[16],
    const real_T alpha[40000], const int32_T t, const real_T N, int32_T xoffset,
    const int32_T Xi_dim0, const int32_T Xi_dim1, emxArray_real_T Xi);

static __global__ void
expectation_kernel3(const emxArray_real_T Xi, const int32_T t, const int32_T k,
                    int32_T vstride, const int32_T a_dim1,
                    const int32_T Xi_dim0, const int32_T Xi_dim1,
                    emxArray_real_T a);

static __global__ void expectation_kernel4(const emxArray_real_T Xi,
                                           const int32_T t, real_T *y);

static __global__ void
expectation_kernel5(const real_T *y, const emxArray_real_T Xi, const int32_T t,
                    const int32_T k, int32_T vstride, const int32_T Xi_dim1,
                    const int32_T Xi_dim0, const int32_T b_Xi_dim1,
                    emxArray_real_T b_Xi);

static __global__ void expectation_kernel6(const emxArray_real_T Xi,
                                           const int32_T t, const int32_T b_Xi,
                                           int32_T c_Xi, const int32_T Xi_dim0,
                                           const int32_T Xi_dim1,
                                           const int32_T b_Xi_dim1,
                                           emxArray_real_T d_Xi);

static __global__ void expectation_kernel7(const emxArray_real_T Xi,
                                           int32_T vstride,
                                           emxArray_real_T Gamma);

static __global__ void expectation_kernel8(const emxArray_real_T Xi,
                                           const int32_T xoffset,
                                           int32_T vstride,
                                           emxArray_real_T Gamma);

static __global__ void expectation_kernel9(int32_T sz, emxArray_real_T Gamma);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

// Function Definitions
static __global__
    __launch_bounds__(1024, 1) void expectation_kernel1(real_T T, real_T N,
                                                        emxArray_real_T Xi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(static_cast<int32_T>(T - 1.0) *
                                      static_cast<int32_T>(N) *
                                      static_cast<int32_T>(N) -
                                  1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    Xi.data[j] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void expectation_kernel2(
    const real_T beta[40000], const real_T B[40000], const real_T trans[16],
    const real_T alpha[40000], const int32_T t, const real_T N, int32_T xoffset,
    const int32_T Xi_dim0, const int32_T Xi_dim1, emxArray_real_T Xi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(static_cast<int32_T>(N) - 1) + 1ULL) *
                (static_cast<uint64_T>(xoffset) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T j;
    j = static_cast<int32_T>(
        idx % (static_cast<uint64_T>(static_cast<int32_T>(N) - 1) + 1ULL));
    i = static_cast<int32_T>(
        (idx - static_cast<uint64_T>(j)) /
        (static_cast<uint64_T>(static_cast<int32_T>(N) - 1) + 1ULL));
    Xi.data[(t + Xi_dim0 * i) + Xi_dim0 * Xi_dim1 * j] =
        alpha[t + 10000 * i] * trans[i + (j << 2)] * B[(t + 10000 * j) + 1] *
        beta[(t + 10000 * j) + 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void expectation_kernel3(
    const emxArray_real_T Xi, const int32_T t, const int32_T k, int32_T vstride,
    const int32_T a_dim1, const int32_T Xi_dim0, const int32_T Xi_dim1,
    emxArray_real_T a)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(k) + 1ULL) *
                (static_cast<uint64_T>(vstride) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(k) + 1ULL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(k) + 1ULL));
    a.data[i + a_dim1 * j] = Xi.data[(t + Xi_dim0 * i) + Xi_dim0 * Xi_dim1 * j];
  }
}

static __global__ __launch_bounds__(32, 1) void expectation_kernel4(
    const emxArray_real_T Xi, const int32_T t, real_T *y)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *y = Xi.data[t];
  }
}

static __global__ __launch_bounds__(1024, 1) void expectation_kernel5(
    const real_T *y, const emxArray_real_T Xi, const int32_T t, const int32_T k,
    int32_T vstride, const int32_T Xi_dim1, const int32_T Xi_dim0,
    const int32_T b_Xi_dim1, emxArray_real_T b_Xi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(k) + 1ULL) *
                (static_cast<uint64_T>(vstride) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(k) + 1ULL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(k) + 1ULL));
    b_Xi.data[i + Xi_dim1 * j] =
        Xi.data[(t + Xi_dim0 * i) + Xi_dim0 * b_Xi_dim1 * j] / *y;
  }
}

static __global__ __launch_bounds__(1024, 1) void expectation_kernel6(
    const emxArray_real_T Xi, const int32_T t, const int32_T b_Xi, int32_T c_Xi,
    const int32_T Xi_dim0, const int32_T Xi_dim1, const int32_T b_Xi_dim1,
    emxArray_real_T d_Xi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_Xi) + 1ULL) *
                (static_cast<uint64_T>(c_Xi) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_Xi) + 1ULL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b_Xi) + 1ULL));
    d_Xi.data[(t + Xi_dim0 * i) + Xi_dim0 * Xi_dim1 * j] =
        Xi.data[i + b_Xi_dim1 * j];
  }
}

static __global__ __launch_bounds__(1024, 1) void expectation_kernel7(
    const emxArray_real_T Xi, int32_T vstride, emxArray_real_T Gamma)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(vstride);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    Gamma.data[j] = Xi.data[j];
  }
}

static __global__ __launch_bounds__(1024, 1) void expectation_kernel8(
    const emxArray_real_T Xi, const int32_T xoffset, int32_T vstride,
    emxArray_real_T Gamma)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(vstride);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    Gamma.data[j] += Xi.data[xoffset + j];
  }
}

static __global__
    __launch_bounds__(1024, 1) void expectation_kernel9(int32_T sz,
                                                        emxArray_real_T Gamma)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(sz);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    Gamma.data[j] = 0.0;
  }
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  real_T *newData;
  if (gpu->data == 0) {
    newData = 0ULL;
    hipMalloc(&newData, cpu->allocatedSize * sizeof(real_T));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0ULL;
      hipMalloc(&newData, cpu->allocatedSize * sizeof(real_T));
      hipMemcpy(newData, gpu->data, actualSizeGpu * sizeof(real_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        hipFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    hipFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      hipFree(gpu->data);
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(real_T));
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(real_T),
             hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(real_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real_T));
}

void expectation(const real_T alpha[40000], const real_T trans[16],
                 const real_T B[40000], const real_T beta[40000], real_T T,
                 real_T N, emxArray_real_T *Xi, emxArray_real_T *Gamma)
{
  dim3 block;
  dim3 grid;
  emxArray_real_T b_gpu_Xi;
  emxArray_real_T gpu_Gamma;
  emxArray_real_T gpu_Xi;
  emxArray_real_T gpu_a;
  emxArray_real_T *a;
  emxArray_real_T *b_Xi;
  real_T(*gpu_B)[40000];
  real_T(*gpu_alpha)[40000];
  real_T(*gpu_beta)[40000];
  real_T(*gpu_trans)[16];
  real_T y;
  real_T *gpu_y;
  int32_T b_Xi_dim1;
  int32_T i;
  int32_T k;
  int32_T vstride;
  int32_T xoffset;
  boolean_T B_dirtyOnCpu;
  boolean_T Gamma_dirtyOnCpu;
  boolean_T Gamma_dirtyOnGpu;
  boolean_T Xi_dirtyOnCpu;
  boolean_T Xi_dirtyOnGpu;
  boolean_T a_dirtyOnGpu;
  boolean_T alpha_dirtyOnCpu;
  boolean_T beta_dirtyOnCpu;
  boolean_T trans_dirtyOnCpu;
  boolean_T validLaunchParams;
  boolean_T y_dirtyOnCpu;
  gpuEmxReset_real_T(&b_gpu_Xi);
  hipMalloc(&gpu_y, 8ULL);
  gpuEmxReset_real_T(&gpu_a);
  gpuEmxReset_real_T(&gpu_Gamma);
  hipMalloc(&gpu_alpha, 320000ULL);
  hipMalloc(&gpu_trans, 128ULL);
  hipMalloc(&gpu_B, 320000ULL);
  hipMalloc(&gpu_beta, 320000ULL);
  gpuEmxReset_real_T(&gpu_Xi);
  a_dirtyOnGpu = false;
  y_dirtyOnCpu = false;
  Gamma_dirtyOnGpu = false;
  Gamma_dirtyOnCpu = true;
  alpha_dirtyOnCpu = true;
  trans_dirtyOnCpu = true;
  B_dirtyOnCpu = true;
  beta_dirtyOnCpu = true;
  Xi_dirtyOnGpu = false;
  Xi_dirtyOnCpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  k = Xi->size[0] * Xi->size[1] * Xi->size[2];
  Xi->size[0] = static_cast<int32_T>(T - 1.0);
  Xi->size[1] = static_cast<int32_T>(N);
  Xi->size[2] = static_cast<int32_T>(N);
  emxEnsureCapacity_real_T(Xi, k, &emlrtRTEI);
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>(
          (static_cast<int32_T>(T - 1.0) * static_cast<int32_T>(N) *
               static_cast<int32_T>(N) -
           1) +
          1LL),
      &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_Xi, Xi);
    expectation_kernel1<<<grid, block>>>(T, N, gpu_Xi);
    Xi_dirtyOnCpu = false;
    Xi_dirtyOnGpu = true;
  }
  i = static_cast<int32_T>(T - 1.0);
  emxInit_real_T(&a, 3, &d_emlrtRTEI, true);
  emxInit_real_T(&b_Xi, 3, &e_emlrtRTEI, true);
  for (int32_T t{0}; t < i; t++) {
    int32_T Xi_dim0;
    int32_T Xi_dim1;
    Xi_dim0 = Xi->size[0];
    Xi_dim1 = Xi->size[1];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>(((static_cast<int32_T>(N) - 1) + 1LL) *
                            ((static_cast<int32_T>(N) - 1) + 1LL)),
        &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (beta_dirtyOnCpu) {
        hipMemcpy(*gpu_beta, beta, 320000ULL, hipMemcpyHostToDevice);
      }
      beta_dirtyOnCpu = false;
      if (B_dirtyOnCpu) {
        hipMemcpy(*gpu_B, B, 320000ULL, hipMemcpyHostToDevice);
      }
      B_dirtyOnCpu = false;
      if (trans_dirtyOnCpu) {
        hipMemcpy(*gpu_trans, trans, 128ULL, hipMemcpyHostToDevice);
      }
      trans_dirtyOnCpu = false;
      if (alpha_dirtyOnCpu) {
        hipMemcpy(*gpu_alpha, alpha, 320000ULL, hipMemcpyHostToDevice);
      }
      alpha_dirtyOnCpu = false;
      if (Xi_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Xi, Xi);
      }
      expectation_kernel2<<<grid, block>>>(
          *gpu_beta, *gpu_B, *gpu_trans, *gpu_alpha, t, N,
          static_cast<int32_T>(N) - 1, Xi_dim0, Xi_dim1, gpu_Xi);
      Xi_dirtyOnCpu = false;
      Xi_dirtyOnGpu = true;
    }
    k = Xi->size[1] - 1;
    vstride = Xi->size[2] - 1;
    xoffset = a->size[0] * a->size[1] * a->size[2];
    a->size[0] = 1;
    a->size[1] = Xi->size[1];
    a->size[2] = Xi->size[2];
    emxEnsureCapacity_real_T(a, xoffset, &d_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(a, &gpu_a);
    xoffset = a->size[1];
    Xi_dim0 = Xi->size[0];
    Xi_dim1 = Xi->size[1];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((k + 1LL) * (vstride + 1LL)), &grid, &block, 1024U,
        65535U);
    if (validLaunchParams) {
      if (Xi_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Xi, Xi);
      }
      Xi_dirtyOnCpu = false;
      expectation_kernel3<<<grid, block>>>(gpu_Xi, t, k, vstride, xoffset,
                                           Xi_dim0, Xi_dim1, gpu_a);
      a_dirtyOnGpu = true;
    }
    xoffset = Xi->size[1] * Xi->size[2];
    if (Xi->size[1] * Xi->size[2] == 0) {
      y = 0.0;
      y_dirtyOnCpu = true;
    } else {
      if (Xi_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Xi, Xi);
      }
      Xi_dirtyOnCpu = false;
      if (y_dirtyOnCpu) {
        hipMemcpy(gpu_y, &y, 8ULL, hipMemcpyHostToDevice);
      }
      expectation_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_Xi, t,
                                                                   gpu_y);
      y_dirtyOnCpu = false;
      validLaunchParams = true;
      for (k = 0; k <= xoffset - 2; k++) {
        if (validLaunchParams) {
          hipMemcpy(&y, gpu_y, 8ULL, hipMemcpyDeviceToHost);
        }
        if (a_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(a, &gpu_a);
        }
        a_dirtyOnGpu = false;
        y += a->data[k + 1];
        validLaunchParams = false;
        y_dirtyOnCpu = true;
      }
    }
    k = Xi->size[1] - 1;
    vstride = Xi->size[2] - 1;
    xoffset = b_Xi->size[0] * b_Xi->size[1] * b_Xi->size[2];
    b_Xi->size[0] = 1;
    b_Xi->size[1] = Xi->size[1];
    b_Xi->size[2] = Xi->size[2];
    emxEnsureCapacity_real_T(b_Xi, xoffset, &e_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(b_Xi, &b_gpu_Xi);
    Xi_dim1 = b_Xi->size[1];
    Xi_dim0 = Xi->size[0];
    b_Xi_dim1 = Xi->size[1];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((k + 1LL) * (vstride + 1LL)), &grid, &block, 1024U,
        65535U);
    if (validLaunchParams) {
      if (y_dirtyOnCpu) {
        hipMemcpy(gpu_y, &y, 8ULL, hipMemcpyHostToDevice);
      }
      y_dirtyOnCpu = false;
      if (Xi_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Xi, Xi);
      }
      Xi_dirtyOnCpu = false;
      expectation_kernel5<<<grid, block>>>(
          gpu_y, gpu_Xi, t, k, vstride, Xi_dim1, Xi_dim0, b_Xi_dim1, b_gpu_Xi);
    }
    xoffset = b_Xi->size[2] - 1;
    vstride = b_Xi->size[1] - 1;
    Xi_dim0 = Xi->size[0];
    Xi_dim1 = Xi->size[1];
    b_Xi_dim1 = b_Xi->size[1];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((vstride + 1LL) * (xoffset + 1LL)), &grid, &block,
        1024U, 65535U);
    if (validLaunchParams) {
      if (Xi_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Xi, Xi);
      }
      expectation_kernel6<<<grid, block>>>(b_gpu_Xi, t, vstride, xoffset,
                                           Xi_dim0, Xi_dim1, b_Xi_dim1, gpu_Xi);
      Xi_dirtyOnCpu = false;
      Xi_dirtyOnGpu = true;
    }
  }
  emxFree_real_T(&b_Xi);
  emxFree_real_T(&a);
  xoffset = Xi->size[2];
  if ((Xi->size[0] == 0) || (Xi->size[1] == 0) || (Xi->size[2] == 0)) {
    uint32_T sz[3];
    for (b_Xi_dim1 = 0; b_Xi_dim1 < 3; b_Xi_dim1++) {
      sz[b_Xi_dim1] = static_cast<uint32_T>(Xi->size[b_Xi_dim1]);
    }
    k = Gamma->size[0] * Gamma->size[1];
    Gamma->size[0] = static_cast<int32_T>(sz[0]);
    Gamma->size[1] = static_cast<int32_T>(sz[1]);
    emxEnsureCapacity_real_T(Gamma, k, &b_emlrtRTEI);
    xoffset = static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>(xoffset + 1LL), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_Gamma, Gamma);
      expectation_kernel9<<<grid, block>>>(xoffset, gpu_Gamma);
      Gamma_dirtyOnGpu = true;
    }
  } else {
    uint32_T sz[3];
    vstride = Xi->size[0] * Xi->size[1] - 1;
    for (b_Xi_dim1 = 0; b_Xi_dim1 < 3; b_Xi_dim1++) {
      sz[b_Xi_dim1] = static_cast<uint32_T>(Xi->size[b_Xi_dim1]);
    }
    k = Gamma->size[0] * Gamma->size[1];
    Gamma->size[0] = static_cast<int32_T>(sz[0]);
    Gamma->size[1] = static_cast<int32_T>(sz[1]);
    emxEnsureCapacity_real_T(Gamma, k, &c_emlrtRTEI);
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>(vstride + 1LL), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (Xi_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Xi, Xi);
      }
      Xi_dirtyOnCpu = false;
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_Gamma, Gamma);
      expectation_kernel7<<<grid, block>>>(gpu_Xi, vstride, gpu_Gamma);
      Gamma_dirtyOnCpu = false;
      Gamma_dirtyOnGpu = true;
    }
    for (k = 0; k <= xoffset - 2; k++) {
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>(vstride + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        if (Xi_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_Xi, Xi);
        }
        Xi_dirtyOnCpu = false;
        if (Gamma_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_Gamma, Gamma);
        }
        expectation_kernel8<<<grid, block>>>(gpu_Xi, (k + 1) * (vstride + 1),
                                             vstride, gpu_Gamma);
        Gamma_dirtyOnCpu = false;
        Gamma_dirtyOnGpu = true;
      }
    }
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (Xi_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(Xi, &gpu_Xi);
  }
  if (Gamma_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(Gamma, &gpu_Gamma);
  }
  gpuEmxFree_real_T(&gpu_Xi);
  hipFree(*gpu_beta);
  hipFree(*gpu_B);
  hipFree(*gpu_trans);
  hipFree(*gpu_alpha);
  gpuEmxFree_real_T(&gpu_Gamma);
  gpuEmxFree_real_T(&gpu_a);
  hipFree(gpu_y);
  gpuEmxFree_real_T(&b_gpu_Xi);
}

// End of code generation (expectation.cu)
