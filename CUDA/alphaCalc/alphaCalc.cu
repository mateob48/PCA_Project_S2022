#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// alphaCalc.cu
//
// Code generation for function 'alphaCalc'
//

// Include files
#include "alphaCalc.h"
#include "alphaCalc_data.h"
#include "alphaCalc_emxutil.h"
#include "alphaCalc_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cstring>

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    11,          // lineNo
    5,           // colNo
    "alphaCalc", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\alphaCalc.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    7,           // lineNo
    23,          // colNo
    "alphaCalc", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\alphaCalc.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    5,           // lineNo
    29,          // colNo
    "alphaCalc", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\alphaCalc.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    12,          // lineNo
    5,           // colNo
    "alphaCalc", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\alphaCalc.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    8,           // lineNo
    21,          // colNo
    "alphaCalc", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\alphaCalc.m" // pName
};

// Function Declarations
static __global__ void
alphaCalc_kernel1(const int32_T i, const emxArray_real_T trans,
                  const emxArray_real_T alpha1, const int32_T t, int32_T b_i,
                  const int32_T alpha1_dim0, const int32_T trans_dim0,
                  emxArray_real_T x);

static __global__ void alphaCalc_kernel2(const emxArray_real_T alpha1,
                                         const int32_T vlen, int32_T i,
                                         const int32_T alpha1_dim0,
                                         emxArray_real_T x);

static __global__ void alphaCalc_kernel3(const real_T y,
                                         const emxArray_real_T alpha1,
                                         const int32_T vlen, int32_T i,
                                         const int32_T alpha1_dim0,
                                         emxArray_real_T b_alpha1);

static __global__ void alphaCalc_kernel4(const emxArray_real_T alpha1,
                                         const int32_T i1, int32_T b_alpha1,
                                         const int32_T alpha1_dim0,
                                         emxArray_real_T c_alpha1);

static __global__ void alphaCalc_kernel5(const emxArray_real_T alpha1,
                                         int32_T b_alpha1,
                                         emxArray_real_T alpha);

static __global__ void alphaCalc_kernel6(const emxArray_real_T scale1,
                                         int32_T b_scale1,
                                         emxArray_real_T scale);

static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             int32_T in3, const emxArray_real_T *in4,
                             int32_T in5);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void alphaCalc_kernel1(
    const int32_T i, const emxArray_real_T trans, const emxArray_real_T alpha1,
    const int32_T t, int32_T b_i, const int32_T alpha1_dim0,
    const int32_T trans_dim0, emxArray_real_T x)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_i);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    x.data[i1] =
        alpha1.data[t + alpha1_dim0 * i1] * trans.data[i1 + trans_dim0 * i];
  }
}

static __global__ __launch_bounds__(1024, 1) void alphaCalc_kernel2(
    const emxArray_real_T alpha1, const int32_T vlen, int32_T i,
    const int32_T alpha1_dim0, emxArray_real_T x)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(i);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    x.data[i1] = alpha1.data[vlen + alpha1_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void alphaCalc_kernel3(
    const real_T y, const emxArray_real_T alpha1, const int32_T vlen, int32_T i,
    const int32_T alpha1_dim0, emxArray_real_T b_alpha1)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(i);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_alpha1.data[i1] = alpha1.data[vlen + alpha1_dim0 * i1] / y;
  }
}

static __global__ __launch_bounds__(1024, 1) void alphaCalc_kernel4(
    const emxArray_real_T alpha1, const int32_T i1, int32_T b_alpha1,
    const int32_T alpha1_dim0, emxArray_real_T c_alpha1)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_alpha1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_i1;
    b_i1 = static_cast<int32_T>(idx);
    c_alpha1.data[i1 + alpha1_dim0 * b_i1] = alpha1.data[b_i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void alphaCalc_kernel5(
    const emxArray_real_T alpha1, int32_T b_alpha1, emxArray_real_T alpha)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_alpha1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    alpha.data[i1] = alpha1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void alphaCalc_kernel6(
    const emxArray_real_T scale1, int32_T b_scale1, emxArray_real_T scale)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_scale1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    scale.data[i1] = scale1.data[i1];
  }
}

static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             int32_T in3, const emxArray_real_T *in4,
                             int32_T in5)
{
  int32_T i;
  int32_T i1;
  int32_T i2;
  int32_T stride_0_1;
  int32_T stride_1_1;
  i = in2->size[1];
  i1 = in4->size[0];
  i2 = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  if (i1 == 1) {
    in1->size[1] = i;
  } else {
    in1->size[1] = i1;
  }
  emxEnsureCapacity_real_T(in1, i2, &c_emlrtRTEI);
  stride_0_1 = (i != 1);
  stride_1_1 = (i1 != 1);
  if (i1 == 1) {
    i1 = i;
  }
  for (i2 = 0; i2 < i1; i2++) {
    in1->data[i2] = in2->data[in3 + in2->size[0] * (i2 * stride_0_1)] *
                    in4->data[i2 * stride_1_1 + in4->size[0] * in5];
  }
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  real_T *newData;
  if (gpu->data == 0) {
    newData = 0ULL;
    hipMalloc(&newData, cpu->allocatedSize * sizeof(real_T));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0ULL;
      hipMalloc(&newData, cpu->allocatedSize * sizeof(real_T));
      hipMemcpy(newData, gpu->data, actualSizeGpu * sizeof(real_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        hipFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    hipFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      hipFree(gpu->data);
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(real_T));
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(real_T),
             hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(real_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real_T));
}

void alphaCalc(emxArray_real_T *alpha1, emxArray_real_T *scale1,
               const emxArray_real_T *trans, const emxArray_real_T *B, real_T N,
               real_T T, emxArray_real_T *alpha, emxArray_real_T *scale)
{
  dim3 block;
  dim3 grid;
  emxArray_real_T b_gpu_alpha1;
  emxArray_real_T b_gpu_x;
  emxArray_real_T gpu_alpha;
  emxArray_real_T gpu_alpha1;
  emxArray_real_T gpu_scale;
  emxArray_real_T gpu_scale1;
  emxArray_real_T gpu_trans;
  emxArray_real_T gpu_x;
  emxArray_real_T *b_alpha1;
  emxArray_real_T *b_x;
  emxArray_real_T *x;
  int32_T c_i;
  int32_T i;
  int32_T vlen;
  boolean_T alpha1_dirtyOnCpu;
  boolean_T alpha1_dirtyOnGpu;
  boolean_T alpha_dirtyOnGpu;
  boolean_T b_x_dirtyOnGpu;
  boolean_T scale_dirtyOnGpu;
  boolean_T trans_dirtyOnCpu;
  boolean_T validLaunchParams;
  boolean_T x_dirtyOnCpu;
  boolean_T x_dirtyOnGpu;
  gpuEmxReset_real_T(&b_gpu_alpha1);
  gpuEmxReset_real_T(&gpu_scale);
  gpuEmxReset_real_T(&gpu_scale1);
  gpuEmxReset_real_T(&b_gpu_x);
  gpuEmxReset_real_T(&gpu_trans);
  gpuEmxReset_real_T(&gpu_x);
  gpuEmxReset_real_T(&gpu_alpha);
  gpuEmxReset_real_T(&gpu_alpha1);
  b_x_dirtyOnGpu = false;
  x_dirtyOnGpu = false;
  x_dirtyOnCpu = false;
  scale_dirtyOnGpu = false;
  trans_dirtyOnCpu = true;
  alpha_dirtyOnGpu = false;
  alpha1_dirtyOnGpu = false;
  alpha1_dirtyOnCpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  i = static_cast<int32_T>(T + -1.0);
  emxInit_real_T(&x, 2, &b_emlrtRTEI, true);
  emxInit_real_T(&b_x, 2, &c_emlrtRTEI, true);
  emxInit_real_T(&b_alpha1, 2, &e_emlrtRTEI, true);
  for (int32_T t{0}; t < i; t++) {
    real_T y;
    int32_T i1;
    int32_T k;
    i1 = static_cast<int32_T>(N);
    for (int32_T b_i{0}; b_i < i1; b_i++) {
      if (trans->size[0] == alpha1->size[1]) {
        c_i = alpha1->size[1] - 1;
        k = b_x->size[0] * b_x->size[1];
        b_x->size[0] = 1;
        b_x->size[1] = alpha1->size[1];
        emxEnsureCapacity_real_T(b_x, k, &c_emlrtRTEI);
        if (!x_dirtyOnCpu) {
          gpuEmxEnsureCapacity_real_T(b_x, &b_gpu_x);
        }
        k = alpha1->size[0];
        vlen = trans->size[0];
        validLaunchParams = mwGetLaunchParameters1D(
            static_cast<real_T>(c_i + 1LL), &grid, &block, 1024U, 65535U);
        if (validLaunchParams) {
          if (trans_dirtyOnCpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_trans, trans);
          }
          trans_dirtyOnCpu = false;
          if (alpha1_dirtyOnCpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha1, alpha1);
          }
          if (x_dirtyOnCpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_x, b_x);
          }
          alphaCalc_kernel1<<<grid, block>>>(b_i, gpu_trans, gpu_alpha1, t, c_i,
                                             k, vlen, b_gpu_x);
          x_dirtyOnCpu = false;
          b_x_dirtyOnGpu = true;
        }
      } else {
        if (b_x_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(b_x, &b_gpu_x);
        }
        if (alpha1_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(alpha1, &gpu_alpha1);
        }
        alpha1_dirtyOnGpu = false;
        binary_expand_op(b_x, alpha1, t, trans, b_i);
        b_x_dirtyOnGpu = false;
        x_dirtyOnCpu = true;
      }
      vlen = b_x->size[1];
      if (b_x->size[1] == 0) {
        y = 0.0;
      } else {
        if (b_x_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(b_x, &b_gpu_x);
        }
        b_x_dirtyOnGpu = false;
        y = b_x->data[0];
        for (k = 0; k <= vlen - 2; k++) {
          y += b_x->data[k + 1];
        }
      }
      if (alpha1_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(alpha1, &gpu_alpha1);
      }
      alpha1->data[(t + alpha1->size[0] * b_i) + 1] =
          y * B->data[(t + B->size[0] * b_i) + 1];
      alpha1_dirtyOnGpu = false;
      alpha1_dirtyOnCpu = true;
    }
    c_i = alpha1->size[1] - 1;
    k = x->size[0] * x->size[1];
    x->size[0] = 1;
    x->size[1] = alpha1->size[1];
    emxEnsureCapacity_real_T(x, k, &b_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(x, &gpu_x);
    k = alpha1->size[0];
    validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(c_i + 1LL),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (alpha1_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha1, alpha1);
      }
      alpha1_dirtyOnCpu = false;
      alphaCalc_kernel2<<<grid, block>>>(gpu_alpha1, t + 1, c_i, k, gpu_x);
      x_dirtyOnGpu = true;
    }
    vlen = alpha1->size[1];
    if (alpha1->size[1] == 0) {
      y = 0.0;
    } else {
      if (alpha1_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(alpha1, &gpu_alpha1);
      }
      alpha1_dirtyOnGpu = false;
      y = alpha1->data[t + 1];
      for (k = 0; k <= vlen - 2; k++) {
        if (x_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(x, &gpu_x);
        }
        x_dirtyOnGpu = false;
        y += x->data[k + 1];
      }
    }
    scale1->data[t + 1] = y;
    c_i = alpha1->size[1] - 1;
    k = b_alpha1->size[0] * b_alpha1->size[1];
    b_alpha1->size[0] = 1;
    b_alpha1->size[1] = alpha1->size[1];
    emxEnsureCapacity_real_T(b_alpha1, k, &e_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(b_alpha1, &b_gpu_alpha1);
    k = alpha1->size[0];
    validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(c_i + 1LL),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (alpha1_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha1, alpha1);
      }
      alpha1_dirtyOnCpu = false;
      alphaCalc_kernel3<<<grid, block>>>(y, gpu_alpha1, t + 1, c_i, k,
                                         b_gpu_alpha1);
    }
    vlen = b_alpha1->size[1] - 1;
    k = alpha1->size[0];
    validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(vlen + 1LL),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (alpha1_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha1, alpha1);
      }
      alphaCalc_kernel4<<<grid, block>>>(b_gpu_alpha1, t + 1, vlen, k,
                                         gpu_alpha1);
      alpha1_dirtyOnCpu = false;
      alpha1_dirtyOnGpu = true;
    }
  }
  emxFree_real_T(&b_alpha1);
  emxFree_real_T(&b_x);
  emxFree_real_T(&x);
  c_i = alpha->size[0] * alpha->size[1];
  alpha->size[0] = alpha1->size[0];
  alpha->size[1] = alpha1->size[1];
  emxEnsureCapacity_real_T(alpha, c_i, &emlrtRTEI);
  vlen = alpha1->size[0] * alpha1->size[1] - 1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(vlen + 1LL),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    if (alpha1_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha1, alpha1);
    }
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
    alphaCalc_kernel5<<<grid, block>>>(gpu_alpha1, vlen, gpu_alpha);
    alpha_dirtyOnGpu = true;
  }
  c_i = scale->size[0] * scale->size[1];
  scale->size[0] = 1;
  scale->size[1] = scale1->size[1];
  emxEnsureCapacity_real_T(scale, c_i, &d_emlrtRTEI);
  vlen = scale1->size[1] - 1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(vlen + 1LL),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_scale1, scale1);
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_scale, scale);
    alphaCalc_kernel6<<<grid, block>>>(gpu_scale1, vlen, gpu_scale);
    scale_dirtyOnGpu = true;
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (alpha1_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(alpha1, &gpu_alpha1);
  }
  if (alpha_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(alpha, &gpu_alpha);
  }
  if (scale_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(scale, &gpu_scale);
  }
  gpuEmxFree_real_T(&gpu_alpha1);
  gpuEmxFree_real_T(&gpu_alpha);
  gpuEmxFree_real_T(&gpu_x);
  gpuEmxFree_real_T(&gpu_trans);
  gpuEmxFree_real_T(&b_gpu_x);
  gpuEmxFree_real_T(&gpu_scale1);
  gpuEmxFree_real_T(&gpu_scale);
  gpuEmxFree_real_T(&b_gpu_alpha1);
}

// End of code generation (alphaCalc.cu)
