//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_alphaCalc_api.cu
//
// Code generation for function '_coder_alphaCalc_api'
//

// Include files
#include "_coder_alphaCalc_api.h"
#include "alphaCalc.h"
#include "alphaCalc_data.h"
#include "alphaCalc_emxutil.h"
#include "alphaCalc_types.h"
#include "rt_nonfinite.h"

// Variable Definitions
static emlrtRTEInfo f_emlrtRTEI{
    1,                      // lineNo
    1,                      // colNo
    "_coder_alphaCalc_api", // fName
    ""                      // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *scale1, const char_T *identifier,
                               emxArray_real_T *y);

static void b_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               emxArray_real_T *y);

static real_T b_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId);

static void c_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_real_T *ret);

static void d_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_real_T *ret);

static void emlrt_marshallIn(const mxArray *alpha1, const char_T *identifier,
                             emxArray_real_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_real_T *y);

static real_T emlrt_marshallIn(const mxArray *N, const char_T *identifier);

static real_T emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId);

static const mxArray *emlrt_marshallOut(const emxArray_real_T *u);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *scale1, const char_T *identifier,
                               emxArray_real_T *y)
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  b_emlrt_marshallIn(emlrtAlias(scale1), &thisId, y);
  emlrtDestroyArray(&scale1);
}

static void b_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               emxArray_real_T *y)
{
  d_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static real_T b_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 0U, (void *)&dims);
  ret = *static_cast<real_T *>(emlrtMxGetData(src));
  emlrtDestroyArray(&src);
  return ret;
}

static void c_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_real_T *ret)
{
  static const int32_T dims[2]{-1, -1};
  int32_T iv[2];
  int32_T i;
  const boolean_T bv[2]{true, true};
  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                            (const char_T *)"double", false, 2U,
                            (void *)&dims[0], &bv[0], &iv[0]);
  ret->allocatedSize = iv[0] * iv[1];
  i = ret->size[0] * ret->size[1];
  ret->size[0] = iv[0];
  ret->size[1] = iv[1];
  emxEnsureCapacity_real_T(ret, i, static_cast<emlrtRTEInfo *>(nullptr));
  ret->data = static_cast<real_T *>(emlrtMxGetData(src));
  ret->canFreeData = false;
  emlrtDestroyArray(&src);
}

static void d_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_real_T *ret)
{
  static const int32_T dims[2]{1, -1};
  int32_T iv[2];
  int32_T i;
  const boolean_T bv[2]{false, true};
  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                            (const char_T *)"double", false, 2U,
                            (void *)&dims[0], &bv[0], &iv[0]);
  ret->allocatedSize = iv[0] * iv[1];
  i = ret->size[0] * ret->size[1];
  ret->size[0] = iv[0];
  ret->size[1] = iv[1];
  emxEnsureCapacity_real_T(ret, i, static_cast<emlrtRTEInfo *>(nullptr));
  ret->data = static_cast<real_T *>(emlrtMxGetData(src));
  ret->canFreeData = false;
  emlrtDestroyArray(&src);
}

static void emlrt_marshallIn(const mxArray *alpha1, const char_T *identifier,
                             emxArray_real_T *y)
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(alpha1), &thisId, y);
  emlrtDestroyArray(&alpha1);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_real_T *y)
{
  c_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static real_T emlrt_marshallIn(const mxArray *N, const char_T *identifier)
{
  emlrtMsgIdentifier thisId;
  real_T y;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = emlrt_marshallIn(emlrtAlias(N), &thisId);
  emlrtDestroyArray(&N);
  return y;
}

static real_T emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId)
{
  real_T y;
  y = b_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static const mxArray *emlrt_marshallOut(const emxArray_real_T *u)
{
  static const int32_T iv[2]{0, 0};
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateNumericArray(2, (const void *)&iv[0], mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m, &u->data[0]);
  emlrtSetDimensions((mxArray *)m, &u->size[0], 2);
  emlrtAssign(&y, m);
  return y;
}

void alphaCalc_api(const mxArray *const prhs[6], int32_T nlhs,
                   const mxArray *plhs[2])
{
  emxArray_real_T *B;
  emxArray_real_T *alpha;
  emxArray_real_T *alpha1;
  emxArray_real_T *scale;
  emxArray_real_T *scale1;
  emxArray_real_T *trans;
  const mxArray *prhs_copy_idx_0;
  const mxArray *prhs_copy_idx_1;
  real_T N;
  real_T T;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&alpha1, 2, &f_emlrtRTEI, true);
  emxInit_real_T(&scale1, 2, &f_emlrtRTEI, true);
  emxInit_real_T(&trans, 2, &f_emlrtRTEI, true);
  emxInit_real_T(&B, 2, &f_emlrtRTEI, true);
  emxInit_real_T(&alpha, 2, &f_emlrtRTEI, true);
  emxInit_real_T(&scale, 2, &f_emlrtRTEI, true);
  prhs_copy_idx_0 = emlrtProtectR2012b(prhs[0], 0, false, -1);
  prhs_copy_idx_1 = emlrtProtectR2012b(prhs[1], 1, false, -1);
  // Marshall function inputs
  alpha1->canFreeData = false;
  emlrt_marshallIn(emlrtAlias(prhs_copy_idx_0), "alpha1", alpha1);
  scale1->canFreeData = false;
  b_emlrt_marshallIn(emlrtAlias(prhs_copy_idx_1), "scale1", scale1);
  trans->canFreeData = false;
  emlrt_marshallIn(emlrtAlias(prhs[2]), "trans", trans);
  B->canFreeData = false;
  emlrt_marshallIn(emlrtAlias(prhs[3]), "B", B);
  N = emlrt_marshallIn(emlrtAliasP(prhs[4]), "N");
  T = emlrt_marshallIn(emlrtAliasP(prhs[5]), "T");
  // Invoke the target function
  alphaCalc(alpha1, scale1, trans, B, N, T, alpha, scale);
  // Marshall function outputs
  alpha->canFreeData = false;
  plhs[0] = emlrt_marshallOut(alpha);
  emxFree_real_T(&alpha);
  emxFree_real_T(&B);
  emxFree_real_T(&trans);
  emxFree_real_T(&scale1);
  emxFree_real_T(&alpha1);
  if (nlhs > 1) {
    scale->canFreeData = false;
    plhs[1] = emlrt_marshallOut(scale);
  }
  emxFree_real_T(&scale);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

// End of code generation (_coder_alphaCalc_api.cu)
