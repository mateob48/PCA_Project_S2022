//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_alphaCalc_mex.cu
//
// Code generation for function '_coder_alphaCalc_mex'
//

// Include files
#include "_coder_alphaCalc_mex.h"
#include "_coder_alphaCalc_api.h"
#include "alphaCalc_data.h"
#include "alphaCalc_initialize.h"
#include "alphaCalc_terminate.h"
#include "rt_nonfinite.h"
#include <stdexcept>

void emlrtExceptionBridge();
void emlrtExceptionBridge()
{
  throw std::runtime_error("");
}
// Function Definitions
void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs,
                 const mxArray *prhs[])
{
  mexAtExit(&alphaCalc_atexit);
  // Module initialization.
  alphaCalc_initialize();
  try {
    emlrtShouldCleanupOnError((emlrtCTX *)emlrtRootTLSGlobal, false);
    // Dispatch the entry-point.
    unsafe_alphaCalc_mexFunction(nlhs, plhs, nrhs, prhs);
    // Module termination.
    alphaCalc_terminate();
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLSR2022a(&emlrtRootTLSGlobal, &emlrtContextGlobal, nullptr, 1,
                           (void *)&emlrtExceptionBridge,
                           (const char_T *)"windows-1252", true);
  return emlrtRootTLSGlobal;
}

void unsafe_alphaCalc_mexFunction(int32_T nlhs, mxArray *plhs[2], int32_T nrhs,
                                  const mxArray *prhs[6])
{
  const mxArray *outputs[2];
  int32_T b_nlhs;
  // Check for proper number of arguments.
  if (nrhs != 6) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 6, 4, 9, "alphaCalc");
  }
  if (nlhs > 2) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 9,
                        "alphaCalc");
  }
  // Call the function.
  alphaCalc_api(prhs, nlhs, outputs);
  // Copy over outputs to the caller.
  if (nlhs < 1) {
    b_nlhs = 1;
  } else {
    b_nlhs = nlhs;
  }
  emlrtReturnArrays(b_nlhs, &plhs[0], &outputs[0]);
}

// End of code generation (_coder_alphaCalc_mex.cu)
