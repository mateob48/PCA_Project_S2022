//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// alphaCalc_terminate.cu
//
// Code generation for function 'alphaCalc_terminate'
//

// Include files
#include "alphaCalc_terminate.h"
#include "_coder_alphaCalc_mex.h"
#include "alphaCalc_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void alphaCalc_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void alphaCalc_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (alphaCalc_terminate.cu)
