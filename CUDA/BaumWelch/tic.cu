//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// tic.cu
//
// Code generation for function 'tic'
//

// Include files
#include "tic.h"
#include "rt_nonfinite.h"
#include "emlrt.h"

// Function Definitions
namespace coder {
emlrtTimespec tic()
{
  emlrtTimespec tstart;
  emlrtClockGettimeMonotonic(&tstart);
  return tstart;
}

} // namespace coder

// End of code generation (tic.cu)
