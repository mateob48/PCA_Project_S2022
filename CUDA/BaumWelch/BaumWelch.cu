#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// BaumWelch.cu
//
// Code generation for function 'BaumWelch'
//

// Include files
#include "BaumWelch.h"
#include "BaumWelch_data.h"
#include "BaumWelch_emxutil.h"
#include "BaumWelch_types.h"
#include "div.h"
#include "rt_nonfinite.h"
#include "tic.h"
#include "toc.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "emlrt.h"
#include "lapacke.h"
#include "hip/hip_math_constants.h"
#include <cmath>
#include <cstddef>
#include <cstring>
#include <math.h>

// Type Definitions
struct emxArray_real_T_1000 {
  real_T data[1000];
  int32_T size[1];
};

struct cell_wrap_0 {
  emxArray_real_T_1000 f1;
};

// Variable Definitions
static emlrtMCInfo emlrtMCI{
    27,      // lineNo
    5,       // colNo
    "error", // fName
    "C:\\Program "
    "Files\\MATLAB\\R2022a\\toolbox\\eml\\lib\\matlab\\lang\\error.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    37,          // lineNo
    6,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    38,          // lineNo
    1,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    103,    // lineNo
    24,     // colNo
    "rand", // fName
    "C:\\Program "
    "Files\\MATLAB\\R2022a\\toolbox\\eml\\lib\\matlab\\randfun\\rand.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    46,          // lineNo
    1,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    50,          // lineNo
    18,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    170,                     // lineNo
    24,                      // colNo
    "combineVectorElements", // fName
    "C:\\Program "
    "Files\\MATLAB\\R2022a\\toolbox\\eml\\lib\\matlab\\datafun\\private\\combin"
    "eVectorElements.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    20,    // lineNo
    1,     // colNo
    "sum", // fName
    "C:\\Program "
    "Files\\MATLAB\\R2022a\\toolbox\\eml\\lib\\matlab\\datafun\\sum.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    50,          // lineNo
    14,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    57,          // lineNo
    9,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    1,           // lineNo
    33,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    66,          // lineNo
    5,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    70,          // lineNo
    16,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    82,          // lineNo
    5,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    75,          // lineNo
    28,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    78,          // lineNo
    20,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    92,          // lineNo
    5,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    86,          // lineNo
    27,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    102,         // lineNo
    5,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    100,         // lineNo
    19,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    107,         // lineNo
    5,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    110,         // lineNo
    40,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    57,          // lineNo
    5,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    63,          // lineNo
    7,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

static emlrtRTEInfo ab_emlrtRTEI{
    110,         // lineNo
    28,          // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

// Function Declarations
static __global__ void BaumWelch_kernel1(const real_T X[58000],
                                         int32_T idx[1000]);

static __global__ void BaumWelch_kernel10(const int32_T x_size_dim0,
                                          real_T x_data[58000], real_T x[58]);

static __global__ void BaumWelch_kernel11(const int32_T x_size_dim0,
                                          real_T x_data[58000], real_T *muj);

static __global__ void BaumWelch_kernel12(real_T *muj, real_T x[58]);

static __global__ void BaumWelch_kernel13(real_T *muj, real_T dv_data[3364]);

static __global__ void BaumWelch_kernel14(const int8_T dv_size_dim0,
                                          const int32_T i, real_T dv_data[3364],
                                          emxArray_real_T Cov);

static __global__ void
BaumWelch_kernel15(const real_T X[58000], const cell_wrap_0 idx1[4],
                   const int32_T i, const int32_T x_size_dim0,
                   const int32_T b_idx1, real_T x_data[58000]);

static __global__ void BaumWelch_kernel16(const int32_T vlen,
                                          const cell_wrap_0 idx1[4],
                                          const int32_T i, real_T x_data[58000],
                                          real_T y[58]);

static __global__ void BaumWelch_kernel17(real_T y[58]);

static __global__ void BaumWelch_kernel18(const int32_T *m, const real_T y[58],
                                          const int32_T i,
                                          const int32_T Mu_dim0,
                                          emxArray_real_T Mu);

static __global__ void BaumWelch_kernel19(int32_T kEnd, real_T *muj,
                                          emxArray_real_T Pi);

static __global__ void BaumWelch_kernel2(const int32_T idx[1000],
                                         const real_T N, int32_T i,
                                         cell_wrap_0 idx1[4]);

static __global__ void BaumWelch_kernel20(const emxArray_real_T trans,
                                          const int32_T b_trans,
                                          int32_T c_trans, const int32_T x_dim0,
                                          const int32_T trans_dim0,
                                          emxArray_real_T x);

static __global__ void BaumWelch_kernel21(const int32_T vlen,
                                          const emxArray_real_T x, int32_T *LDA,
                                          const int32_T x_dim0,
                                          emxArray_real_T y);

static __global__ void BaumWelch_kernel22(int32_T sz, emxArray_real_T y);

static __global__ void BaumWelch_kernel23(const emxArray_real_T y, int32_T b_y,
                                          emxArray_real_T c_y);

static __global__ void
BaumWelch_kernel24(const emxArray_real_T y, const emxArray_real_T trans,
                   const int32_T b_trans, int32_T c_trans,
                   const int32_T trans_dim0, const int32_T b_trans_dim0,
                   emxArray_real_T d_trans);

static __global__ void BaumWelch_kernel25(const emxArray_real_T trans,
                                          int32_T b_trans,
                                          emxArray_real_T c_trans);

static __global__ void BaumWelch_kernel26(const emxArray_real_T Cov,
                                          const int32_T i, real_T C[3364]);

static __global__ void BaumWelch_kernel27(const emxArray_real_T Mu,
                                          const int32_T i,
                                          const real_T X[58000],
                                          const int32_T Mu_dim0,
                                          real_T X0[58000]);

static __global__ void BaumWelch_kernel28(const emxArray_real_T Cov,
                                          const int32_T i, real_T x[58]);

static __global__ void BaumWelch_kernel29(const emxArray_real_T Cov,
                                          const int32_T i, real_T x[3364]);

static __global__ void BaumWelch_kernel3(real_T N, emxArray_real_T Cov);

static __global__ void BaumWelch_kernel30(const real_T x[3364], real_T y[3364]);

static __global__ void BaumWelch_kernel31(boolean_T y[58]);

static __global__ void BaumWelch_kernel32(const int32_T initAuxVar, real_T *muj,
                                          real_T y[3364], boolean_T b_y[58]);

static __global__ void BaumWelch_kernel33(real_T C[3364]);

static __global__ void BaumWelch_kernel34(const int32_T T_size_dim0,
                                          const int32_T *LDA, int32_T *m,
                                          real_T C[3364], real_T T_data[3364]);

static __global__ void BaumWelch_kernel35(const real_T T_data[3364],
                                          int32_T T_size, real_T dv_data[3364]);

static __global__ void BaumWelch_kernel36(const real_T T_data[3364],
                                          const int32_T T_size_dim0,
                                          const int32_T T_size,
                                          int32_T b_T_size,
                                          real_T dv_data[3364]);

static __global__ void BaumWelch_kernel37(real_T X0[58000], real_T B[58000]);

static __global__ void BaumWelch_kernel38(int32_T *m, int32_T ipiv_t[58]);

static __global__ void BaumWelch_kernel39(int32_T *m,
                                          ptrdiff_t jpvt_t_data[58]);

static __global__ void
BaumWelch_kernel4(const real_T X[58000], const cell_wrap_0 idx1[4],
                  const int32_T i, const int32_T x_size_dim0,
                  const int32_T b_idx1, real_T x_data[58000]);

static __global__ void BaumWelch_kernel40(const ptrdiff_t jpvt_t_data[58],
                                          int32_T *m, int32_T ipiv_t[58]);

static __global__ void BaumWelch_kernel41(int32_T *m, real_T dv_data[3364]);

static __global__ void BaumWelch_kernel42(int32_T *m, real_T x[58]);

static __global__ void BaumWelch_kernel43(int32_T *m, int32_T ipiv_t[58]);

static __global__ void BaumWelch_kernel44(int16_T iv4_idx_0,
                                          real_T Y_data[58000]);

static __global__ void BaumWelch_kernel45(real_T B[58000]);

static __global__ void BaumWelch_kernel46(const int32_T Y_size_dim0,
                                          const int32_T *LDA,
                                          real_T dv_data[3364], real_T B[58000],
                                          int32_T ipiv_t[58],
                                          real_T Y_data[58000]);

static __global__ void BaumWelch_kernel47(const real_T Y_data[58000],
                                          const int32_T Y_size_dim0,
                                          int32_T Y_size,
                                          real_T xRinv_data[58000]);

static __global__ void BaumWelch_kernel48(int32_T ipiv_t[58],
                                          int32_T ipiv_data[58]);

static __global__ void BaumWelch_kernel49(int32_T x_size, real_T dv_data[3364]);

static __global__ void BaumWelch_kernel5(real_T C[3364]);

static __global__ void BaumWelch_kernel50(int32_T ipiv_data[58]);

static __global__ void BaumWelch_kernel51(real_T X0[58000],
                                          real_T xRinv_data[58000]);

static __global__ void BaumWelch_kernel52(const real_T T_data[3364],
                                          const int32_T T_size_dim0, int32_T *m,
                                          real_T x[58]);

static __global__ void BaumWelch_kernel53(int32_T *m, real_T x[58]);

static __global__ void BaumWelch_kernel54(real_T x[58], real_T *muj);

static __global__ void BaumWelch_kernel55(int32_T vlen, real_T x[58],
                                          real_T *muj);

static __global__ void BaumWelch_kernel56(const real_T xRinv_data[58000],
                                          int32_T *m, real_T y_data[58000]);

static __global__ void BaumWelch_kernel57(const real_T y_data[58000],
                                          real_T quadform[1000]);

static __global__ void BaumWelch_kernel58(const real_T y_data[58000],
                                          const int32_T *LDA,
                                          real_T quadform[1000]);

static __global__ void BaumWelch_kernel59(real_T quadform[1000]);

static __global__ void BaumWelch_kernel6(const int32_T x_size_dim0,
                                         const int32_T *m,
                                         real_T x_data[58000]);

static __global__ void BaumWelch_kernel60(const real_T quadform[1000],
                                          const int32_T i, real_T *muj,
                                          emxArray_real_T B);

static __global__ void BaumWelch_kernel61(const emxArray_int32_T iv1,
                                          int32_T kEnd, emxArray_real_T B);

static __global__ void BaumWelch_kernel62(real_T N, emxArray_real_T alpha);

static __global__ void BaumWelch_kernel63(real_T scale[1000]);

static __global__ void BaumWelch_kernel64(const emxArray_real_T B,
                                          const emxArray_real_T Pi,
                                          int32_T b_Pi, emxArray_real_T alpha);

static __global__ void BaumWelch_kernel65(real_T *muj, real_T scale[1000]);

static __global__ void BaumWelch_kernel66(const emxArray_real_T alpha,
                                          int32_T kEnd, real_T *muj,
                                          emxArray_real_T b_alpha);

static __global__ void BaumWelch_kernel67(const emxArray_real_T alpha,
                                          int32_T b_alpha,
                                          emxArray_real_T c_alpha);

static __global__ void BaumWelch_kernel68(
    const int32_T i, const emxArray_real_T trans, const emxArray_real_T alpha,
    int32_T kEnd, const int32_T trans_dim0, int32_T *info_t, emxArray_real_T x);

static __global__ void BaumWelch_kernel69(const emxArray_real_T B,
                                          const int32_T i, real_T *muj,
                                          int32_T *info_t,
                                          emxArray_real_T alpha);

static __global__ void BaumWelch_kernel7(real_T C[3364]);

static __global__ void BaumWelch_kernel70(real_T *muj, int32_T *info_t,
                                          real_T scale[1000]);

static __global__ void BaumWelch_kernel71(const emxArray_real_T alpha,
                                          const int32_T *m, int32_T kEnd,
                                          real_T *muj, emxArray_real_T b_alpha);

static __global__ void BaumWelch_kernel72(const emxArray_real_T alpha,
                                          const int32_T *LDA, int32_T b_alpha,
                                          emxArray_real_T c_alpha);

static __global__ void BaumWelch_kernel73(real_T N, emxArray_real_T beta);

static __global__ void BaumWelch_kernel74(const real_T scale[1000],
                                          int32_T unnamed_idx_1,
                                          emxArray_real_T beta);

static __global__ void
BaumWelch_kernel75(const emxArray_real_T beta, const int32_T vlen,
                   const emxArray_real_T B, const int32_T kEnd,
                   const emxArray_real_T trans, const int32_T i, int32_T *LDA,
                   const int32_T trans_dim0, emxArray_real_T x);

static __global__ void BaumWelch_kernel76(const real_T scale[1000],
                                          const int32_T i, real_T *muj,
                                          int32_T *info_t,
                                          emxArray_real_T beta);

static __global__ void BaumWelch_kernel77(real_T N, emxArray_real_T Xi);

static __global__ void
BaumWelch_kernel78(const emxArray_real_T beta, const emxArray_real_T B,
                   const emxArray_real_T trans, const emxArray_real_T alpha,
                   const real_T N, int32_T i11, const int32_T Xi_dim1,
                   const int32_T trans_dim0, int32_T *info_t,
                   emxArray_real_T Xi);

static __global__ void
BaumWelch_kernel79(const emxArray_real_T Xi, const int32_T kEnd, int32_T vlen,
                   const int32_T Xi_dim1, const int32_T b_Xi_dim1, real_T *muj,
                   int32_T *info_t, emxArray_real_T b_Xi);

static __global__ void BaumWelch_kernel8(real_T C[3364]);

static __global__ void
BaumWelch_kernel80(const emxArray_real_T Xi, const int32_T b_Xi, int32_T c_Xi,
                   const int32_T Xi_dim1, const int32_T b_Xi_dim1,
                   int32_T *info_t, emxArray_real_T d_Xi);

static __global__ void BaumWelch_kernel81(const emxArray_real_T Xi, int32_T *m,
                                          emxArray_real_T Gamma);

static __global__ void BaumWelch_kernel82(const emxArray_real_T Xi,
                                          const int32_T *LDA, int32_T *m,
                                          emxArray_real_T Gamma);

static __global__ void BaumWelch_kernel83(int32_T sz, emxArray_real_T Gamma);

static __global__ void BaumWelch_kernel84(const emxArray_real_T Gamma,
                                          int32_T kEnd, emxArray_real_T Pi);

static __global__ void BaumWelch_kernel85(const emxArray_real_T Xi,
                                          const int32_T i, int32_T b_i,
                                          const int32_T Xi_dim1,
                                          emxArray_real_T a);

static __global__ void BaumWelch_kernel86(const emxArray_real_T a, int32_T *LDA,
                                          emxArray_real_T y);

static __global__ void BaumWelch_kernel87(const emxArray_real_T y,
                                          const int32_T i, int32_T b_y,
                                          const int32_T trans_dim0, real_T *muj,
                                          emxArray_real_T trans);

static __global__ void BaumWelch_kernel88(const emxArray_real_T Gamma,
                                          const int32_T i,
                                          const real_T X[58000],
                                          real_T x[57942]);

static __global__ void BaumWelch_kernel89(const real_T x[57942], real_T y[58]);

static __global__ void BaumWelch_kernel9(real_T C[3364], real_T dv_data[3364]);

static __global__ void BaumWelch_kernel90(const real_T y[58], const int32_T i,
                                          const int32_T Mu_dim0, real_T *muj,
                                          emxArray_real_T Mu);

static __global__ void BaumWelch_kernel91(const emxArray_real_T Mu,
                                          const int32_T i,
                                          const real_T X[58000],
                                          const int32_T Mu_dim0,
                                          real_T d[57942]);

static __global__ void BaumWelch_kernel92(const emxArray_real_T Gamma,
                                          const int32_T i, real_T d[57942],
                                          real_T A[57942]);

static __global__ void BaumWelch_kernel93(const int32_T i, real_T *muj,
                                          real_T y[3364], emxArray_real_T Cov);

static __global__ void BaumWelch_kernel94(const int32_T iwork[1000],
                                          const int32_T j, int32_T kEnd,
                                          int32_T idx[1000]);

static __device__ real_T atomicOpreal_T(real_T *address, real_T value);

static void b_error(const mxArray *m, const mxArray *m1, emlrtMCInfo *location);

static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in3,
                             const emxArray_real_T *in4);

static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             int32_T in3, const emxArray_real_T *in4,
                             int32_T in5);

static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             int32_T in3, const emxArray_real_T *in4,
                             int32_T in5, const emxArray_real_T *in6);

static void gpuEmxEnsureCapacity_int32_T(const emxArray_int32_T *cpu,
                                         emxArray_int32_T *gpu);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxFree_int32_T(emxArray_int32_T *gpu);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxMemcpyCpuToGpu_int32_T(emxArray_int32_T *gpu,
                                         const emxArray_int32_T *cpu);

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxReset_int32_T(emxArray_int32_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

static __device__ real_T shflDown2(real_T in1, uint32_T offset, uint32_T mask);

static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
                                              uint32_T mask);

static __device__ real_T workGroupReduction(real_T val, uint32_T mask,
                                            uint32_T numActiveWarps);

// Function Definitions
static __global__
    __launch_bounds__(512, 1) void BaumWelch_kernel1(const real_T X[58000],
                                                     int32_T idx[1000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 500) {
    int32_T j;
    boolean_T exitg1;
    boolean_T p;
    //  Find the initial means and covariance matrices for each of the states
    //  Split the observations into evenly size states from smallest to largest
    // [idx]=Divide(X,N,'sort');
    k = (k << 1) + 1;
    p = true;
    j = 1;
    exitg1 = false;
    while ((!static_cast<int32_T>(exitg1)) && (static_cast<int32_T>(j < 59))) {
      if ((static_cast<int32_T>(X[(k + 1000 * (j - 1)) - 1] ==
                                X[k + 1000 * (j - 1)])) ||
          (static_cast<int32_T>(
              (static_cast<int32_T>(isnan(X[(k + 1000 * (j - 1)) - 1]))) &&
              (static_cast<int32_T>(isnan(X[k + 1000 * (j - 1)])))))) {
        j++;
      } else {
        if ((!static_cast<int32_T>(X[(k + 1000 * (j - 1)) - 1] <=
                                   X[k + 1000 * (j - 1)])) &&
            (!static_cast<int32_T>(isnan(X[k + 1000 * (j - 1)])))) {
          p = false;
        }
        exitg1 = true;
      }
    }
    if (p) {
      idx[k - 1] = k;
      idx[k] = k + 1;
    } else {
      idx[k - 1] = k + 1;
      idx[k] = k;
    }
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel10(
    const int32_T x_size_dim0, real_T x_data[58000], real_T x[58])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 58) {
    x[xpageoffset] = x_data[x_size_dim0 * xpageoffset];
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel11(
    const int32_T x_size_dim0, real_T x_data[58000], real_T *muj)
{
  real_T tmpRed0;
  uint32_T blockStride;
  uint32_T mask;
  uint32_T thBlkId;
  uint32_T threadId;
  uint32_T threadStride;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  if (static_cast<uint32_T>(mwGetBlockIndex()) == 58U / blockStride) {
    int32_T m;
    m = static_cast<int32_T>(58U % blockStride);
    if (static_cast<uint32_T>(m) > 0U) {
      blockStride = static_cast<uint32_T>(m);
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (threadId <= 57U) {
    tmpRed0 = x_data[x_size_dim0 * static_cast<int32_T>(threadId)];
  }
  mask = __ballot_sync(MAX_uint32_T, threadId <= 57U);
  for (uint32_T idx{threadId + threadStride}; idx <= 57U; idx += threadStride) {
    tmpRed0 += x_data[x_size_dim0 * static_cast<int32_T>(idx)];
  }
  tmpRed0 = workGroupReduction(tmpRed0, mask, blockStride);
  if ((static_cast<int32_T>(threadId <= 57U)) &&
      (static_cast<int32_T>(thBlkId == 0U))) {
    atomicOpreal_T(&muj[0], tmpRed0);
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel12(real_T *muj,
                                                                   real_T x[58])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 58) {
    x[i] -= *muj;
  }
}

static __global__
    __launch_bounds__(32, 1) void BaumWelch_kernel13(real_T *muj,
                                                     real_T dv_data[3364])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    dv_data[0] = 0.017543859649122806 * *muj;
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel14(
    const int8_T dv_size_dim0, const int32_T i, real_T dv_data[3364],
    emxArray_real_T Cov)
{
  uint64_T threadId;
  int32_T k;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 58ULL);
  xpageoffset =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 58ULL);
  if ((static_cast<int32_T>(xpageoffset < 58)) &&
      (static_cast<int32_T>(k < 58))) {
    Cov.data[(k + 58 * xpageoffset) + 3364 * i] =
        dv_data[k + static_cast<int32_T>(dv_size_dim0) * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel15(
    const real_T X[58000], const cell_wrap_0 idx1[4], const int32_T i,
    const int32_T x_size_dim0, const int32_T b_idx1, real_T x_data[58000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_idx1) + 1ULL) * 58ULL - 1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_idx1) + 1ULL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                       (static_cast<uint64_T>(b_idx1) + 1ULL));
    x_data[k + x_size_dim0 * xpageoffset] =
        X[(static_cast<int32_T>(idx1[i].f1.data[k]) + 1000 * xpageoffset) - 1];
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel16(
    const int32_T vlen, const cell_wrap_0 idx1[4], const int32_T i,
    real_T x_data[58000], real_T y[58])
{
  uint64_T threadId;
  int32_T b_i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_i = static_cast<int32_T>(threadId);
  if (b_i < 58) {
    int32_T xpageoffset;
    xpageoffset = b_i * idx1[i].f1.size[0];
    y[b_i] = x_data[xpageoffset];
    for (int32_T k{0}; k <= vlen - 2; k++) {
      y[b_i] += x_data[(xpageoffset + k) + 1];
    }
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel17(real_T y[58])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 58) {
    y[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel18(
    const int32_T *m, const real_T y[58], const int32_T i,
    const int32_T Mu_dim0, emxArray_real_T Mu)
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 58) {
    Mu.data[i + Mu_dim0 * xpageoffset] =
        y[xpageoffset] / static_cast<real_T>(*m);
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel19(
    int32_T kEnd, real_T *muj, emxArray_real_T Pi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(kEnd - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    Pi.data[xpageoffset] /= *muj;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel2(
    const int32_T idx[1000], const real_T N, int32_T i, cell_wrap_0 idx1[4])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(i);
  for (uint64_T b_idx{threadId}; b_idx <= loopEnd; b_idx += threadStride) {
    real_T d;
    real_T muj;
    int32_T b_i;
    int32_T i36;
    int32_T j;
    int32_T k;
    int32_T xpageoffset;
    b_i = static_cast<int32_T>(b_idx);
    d = floor(((static_cast<real_T>(b_i) + 1.0) - 1.0) * 1000.0 / N + 1.0);
    muj = floor((static_cast<real_T>(b_i) + 1.0) * 1000.0 / N);
    if (d > muj) {
      i36 = 1;
      xpageoffset = 0;
    } else {
      i36 = static_cast<int32_T>(d);
      xpageoffset = static_cast<int32_T>(muj);
    }
    j = i36 - 1;
    k = xpageoffset - 1;
    idx1[b_i].f1.size[0] = (xpageoffset - i36) + 1;
    for (xpageoffset = 0; xpageoffset <= k - j; xpageoffset++) {
      idx1[b_i].f1.data[xpageoffset] =
          static_cast<real_T>(idx[(i36 + xpageoffset) - 1]);
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel20(
    const emxArray_real_T trans, const int32_T b_trans, int32_T c_trans,
    const int32_T x_dim0, const int32_T trans_dim0, emxArray_real_T x)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_trans) + 1ULL) *
                (static_cast<uint64_T>(c_trans) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_trans) + 1ULL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                       (static_cast<uint64_T>(b_trans) + 1ULL));
    x.data[k + x_dim0 * xpageoffset] = trans.data[xpageoffset + trans_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel21(
    const int32_T vlen, const emxArray_real_T x, int32_T *LDA,
    const int32_T x_dim0, emxArray_real_T y)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*LDA);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T xpageoffset;
    i = static_cast<int32_T>(idx);
    xpageoffset = i * x_dim0;
    y.data[i] = x.data[xpageoffset];
    for (int32_T k{0}; k <= vlen - 2; k++) {
      y.data[i] += x.data[(xpageoffset + k) + 1];
    }
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel22(int32_T sz,
                                                       emxArray_real_T y)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(sz);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    y.data[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel23(
    const emxArray_real_T y, int32_T b_y, emxArray_real_T c_y)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    c_y.data[xpageoffset] = y.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel24(
    const emxArray_real_T y, const emxArray_real_T trans, const int32_T b_trans,
    int32_T c_trans, const int32_T trans_dim0, const int32_T b_trans_dim0,
    emxArray_real_T d_trans)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_trans) + 1ULL) *
                (static_cast<uint64_T>(c_trans) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_trans) + 1ULL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                       (static_cast<uint64_T>(b_trans) + 1ULL));
    d_trans.data[k + trans_dim0 * xpageoffset] =
        trans.data[k + b_trans_dim0 * xpageoffset] / y.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel25(
    const emxArray_real_T trans, int32_T b_trans, emxArray_real_T c_trans)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_trans);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    c_trans.data[xpageoffset] = trans.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel26(
    const emxArray_real_T Cov, const int32_T i, real_T C[3364])
{
  uint64_T threadId;
  int32_T k;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 58ULL);
  xpageoffset =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 58ULL);
  if ((static_cast<int32_T>(xpageoffset < 58)) &&
      (static_cast<int32_T>(k < 58))) {
    C[k + 58 * xpageoffset] = Cov.data[(k + 58 * xpageoffset) + 3364 * i];
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel27(
    const emxArray_real_T Mu, const int32_T i, const real_T X[58000],
    const int32_T Mu_dim0, real_T X0[58000])
{
  uint64_T threadId;
  int32_T j;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int32_T>(threadId % 1000ULL);
  k = static_cast<int32_T>((threadId - static_cast<uint64_T>(j)) / 1000ULL);
  if ((static_cast<int32_T>(k < 58)) && (static_cast<int32_T>(j < 1000))) {
    X0[j + 1000 * k] = X[j + 1000 * k] - Mu.data[i + Mu_dim0 * k];
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel28(
    const emxArray_real_T Cov, const int32_T i, real_T x[58])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 58) {
    //  Assume identity covariance, data are already standardized
    //  Special case: if Sigma is supplied, then use it to try to interpret
    //  X and Mu as row vectors if they were both column vectors.
    // Check that sigma is the right size
    //  Make sure Sigma is a valid covariance matrix
    // CHOLCOV  Cholesky-like decomposition for covariance matrix.
    //    T = CHOLCOV(SIGMA) computes T such that SIGMA = T'*T.  SIGMA must be
    //    square, symmetric, and positive semi-definite.  If SIGMA is positive
    //    definite, then T is the square, upper triangular Cholesky factor.
    //
    //    If SIGMA is not positive definite, T is computed from an eigenvalue
    //    decomposition of SIGMA.  T is not necessarily triangular or square in
    //    this case.  Any eigenvectors whose corresponding eigenvalue is close
    //    to zero (within a small tolerance) are omitted.  If any remaining
    //    eigenvalues are negative, T is empty.
    //
    //    [T,P] = CHOLCOV(SIGMA) returns the number of negative eigenvalues of
    //    SIGMA, and T is empty if P>0.  If P==0, SIGMA is positive
    //    semi-definite.
    //
    //    If SIGMA is not square and symmetric, P is NaN and T is empty.
    //
    //    [T,P] = CHOLCOV(SIGMA,0) returns P==0 if SIGMA is positive definite,
    //    and T is the Cholesky factor.  If SIGMA is not positive definite, P is
    //    a positive integer and T is empty.  [...] = CHOLCOV(SIGMA,1) is
    //    equivalent to [...] = CHOLCOV(SIGMA).
    //
    //    Example:
    //    Factor a rank-deficient covariance matrix C.
    //        C = [2 1 1 2;1 2 1 2;1 1 2 2;2 2 2 3]
    //        T = cholcov(C)
    //        C2 = T'*T
    //    Generate data with this covariance (aside from random variation).
    //        C3 = cov(randn(10000,3)*T)
    //
    //    See also CHOL.
    //    Copyright 1993-2009 The MathWorks, Inc.
    //  Test for square, symmetric
    x[k] = fabs(Cov.data[(k + 58 * k) + 3364 * i]);
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel29(
    const emxArray_real_T Cov, const int32_T i, real_T x[3364])
{
  uint64_T threadId;
  int32_T k;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 58ULL);
  xpageoffset =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 58ULL);
  if ((static_cast<int32_T>(xpageoffset < 58)) &&
      (static_cast<int32_T>(k < 58))) {
    x[k + 58 * xpageoffset] = Cov.data[(k + 58 * xpageoffset) + 3364 * i] -
                              Cov.data[(xpageoffset + 58 * k) + 3364 * i];
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel3(real_T N,
                                                      emxArray_real_T Cov)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(3364 * static_cast<int32_T>(N) - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    Cov.data[xpageoffset] = 0.0;
  }
}

static __global__
    __launch_bounds__(512, 1) void BaumWelch_kernel30(const real_T x[3364],
                                                      real_T y[3364])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 3364) {
    y[k] = fabs(x[k]);
  }
}

static __global__ __launch_bounds__(64,
                                    1) void BaumWelch_kernel31(boolean_T y[58])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 58) {
    y[xpageoffset] = true;
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel32(
    const int32_T initAuxVar, real_T *muj, real_T y[3364], boolean_T b_y[58])
{
  uint64_T threadId;
  int32_T vlen;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  vlen = static_cast<int32_T>(threadId);
  if (vlen < 58) {
    int32_T i2;
    int32_T m;
    boolean_T exitg1;
    i2 = initAuxVar + vlen * 58;
    m = i2 + 57;
    exitg1 = false;
    while ((!static_cast<int32_T>(exitg1)) && (static_cast<int32_T>(i2 <= m))) {
      if (!static_cast<int32_T>(y[i2 - 1] < *muj)) {
        b_y[vlen] = false;
        exitg1 = true;
      } else {
        i2++;
      }
    }
  }
}

static __global__ __launch_bounds__(512,
                                    1) void BaumWelch_kernel33(real_T C[3364])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 3364) {
    C[xpageoffset] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel34(
    const int32_T T_size_dim0, const int32_T *LDA, int32_T *m, real_T C[3364],
    real_T T_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(*LDA) + 1ULL) *
                (static_cast<uint64_T>(*m) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(*LDA) + 1ULL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                       (static_cast<uint64_T>(*LDA) + 1ULL));
    T_data[k + T_size_dim0 * xpageoffset] = C[k + 58 * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel35(
    const real_T T_data[3364], int32_T T_size, real_T dv_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(T_size);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    dv_data[xpageoffset] = T_data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel36(
    const real_T T_data[3364], const int32_T T_size_dim0, const int32_T T_size,
    int32_T b_T_size, real_T dv_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(T_size) + 1ULL) *
                (static_cast<uint64_T>(b_T_size) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(T_size) + 1ULL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                       (static_cast<uint64_T>(T_size) + 1ULL));
    dv_data[k + 58 * xpageoffset] = T_data[xpageoffset + T_size_dim0 * k];
  }
}

static __global__ __launch_bounds__(512,
                                    1) void BaumWelch_kernel37(real_T X0[58000],
                                                               real_T B[58000])
{
  uint64_T threadId;
  int32_T k;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 58ULL);
  xpageoffset =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 58ULL);
  if ((static_cast<int32_T>(xpageoffset < 1000)) &&
      (static_cast<int32_T>(k < 58))) {
    B[k + 58 * xpageoffset] = X0[xpageoffset + 1000 * k];
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel38(int32_T *m,
                                                       int32_T ipiv_t[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    ipiv_t[xpageoffset] = 0;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel39(
    int32_T *m, ptrdiff_t jpvt_t_data[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    jpvt_t_data[xpageoffset] = (ptrdiff_t)0;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel4(
    const real_T X[58000], const cell_wrap_0 idx1[4], const int32_T i,
    const int32_T x_size_dim0, const int32_T b_idx1, real_T x_data[58000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_idx1) + 1ULL) * 58ULL - 1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_idx1) + 1ULL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                       (static_cast<uint64_T>(b_idx1) + 1ULL));
    x_data[k + x_size_dim0 * xpageoffset] =
        X[(static_cast<int32_T>(idx1[i].f1.data[k]) + 1000 * xpageoffset) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel40(
    const ptrdiff_t jpvt_t_data[58], int32_T *m, int32_T ipiv_t[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    ipiv_t[k] = (int32_T)jpvt_t_data[k];
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel41(int32_T *m,
                                                       real_T dv_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 58ULL * (static_cast<uint64_T>(*m - 1) + 1ULL) - 1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % 58ULL);
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) / 58ULL);
    dv_data[j * 58 + i] = HIP_NAN;
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel42(int32_T *m, real_T x[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x[k] = HIP_NAN;
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel43(int32_T *m,
                                                       int32_T ipiv_t[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    ipiv_t[k] = k + 1;
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel44(int16_T iv4_idx_0,
                                                       real_T Y_data[58000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(static_cast<int32_T>(iv4_idx_0) * 1000 - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    Y_data[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(512,
                                    1) void BaumWelch_kernel45(real_T B[58000])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 58000) {
    B[xpageoffset] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel46(
    const int32_T Y_size_dim0, const int32_T *LDA, real_T dv_data[3364],
    real_T B[58000], int32_T ipiv_t[58], real_T Y_data[58000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 1000) {
    for (int32_T i{0}; i < *LDA; i++) {
      Y_data[(ipiv_t[i] + Y_size_dim0 * k) - 1] = B[i + 58 * k];
    }
    for (int32_T xpageoffset{0}; xpageoffset < *LDA; xpageoffset++) {
      int32_T i36;
      int32_T j;
      j = *LDA - xpageoffset;
      Y_data[(ipiv_t[j - 1] + Y_size_dim0 * k) - 1] /=
          dv_data[(j + 58 * (j - 1)) - 1];
      i36 = j - 1;
      for (int32_T i{0}; i < i36; i++) {
        Y_data[(ipiv_t[i] + Y_size_dim0 * k) - 1] -=
            Y_data[(ipiv_t[j - 1] + Y_size_dim0 * k) - 1] *
            dv_data[i + 58 * (j - 1)];
      }
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel47(
    const real_T Y_data[58000], const int32_T Y_size_dim0, int32_T Y_size,
    real_T xRinv_data[58000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 1000ULL * (static_cast<uint64_T>(Y_size) + 1ULL) - 1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % 1000ULL);
    xpageoffset =
        static_cast<int32_T>((idx - static_cast<uint64_T>(k)) / 1000ULL);
    xRinv_data[k + 1000 * xpageoffset] = Y_data[xpageoffset + Y_size_dim0 * k];
  }
}

static __global__
    __launch_bounds__(64, 1) void BaumWelch_kernel48(int32_T ipiv_t[58],
                                                     int32_T ipiv_data[58])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 58) {
    ipiv_data[k] = ipiv_t[k];
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel49(int32_T x_size,
                                                       real_T dv_data[3364])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(x_size);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    dv_data[xpageoffset] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(512,
                                    1) void BaumWelch_kernel5(real_T C[3364])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 3364) {
    C[xpageoffset] = 0.0;
  }
}

static __global__
    __launch_bounds__(64, 1) void BaumWelch_kernel50(int32_T ipiv_data[58])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 58) {
    ipiv_data[k] = k + 1;
  }
}

static __global__
    __launch_bounds__(512, 1) void BaumWelch_kernel51(real_T X0[58000],
                                                      real_T xRinv_data[58000])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 58000) {
    xRinv_data[xpageoffset] = X0[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel52(
    const real_T T_data[3364], const int32_T T_size_dim0, int32_T *m,
    real_T x[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x[k] = T_data[k + T_size_dim0 * k];
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel53(int32_T *m, real_T x[58])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x[k] = log(x[k]);
  }
}

static __global__ __launch_bounds__(32, 1) void BaumWelch_kernel54(real_T x[58],
                                                                   real_T *muj)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *muj = x[0];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void BaumWelch_kernel55(int32_T vlen,
                                                               real_T x[58],
                                                               real_T *muj)
{
  int64_T loopEnd;
  real_T tmpRed0;
  uint32_T blockStride;
  uint32_T m;
  uint32_T thBlkId;
  uint32_T threadId;
  uint32_T threadStride;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  loopEnd = (static_cast<int64_T>(vlen - 2) + 1LL) - 1LL;
  if (static_cast<uint32_T>(mwGetBlockIndex()) ==
      static_cast<uint32_T>(static_cast<int64_T>(vlen - 2) + 1LL) /
          blockStride) {
    m = (static_cast<int64_T>(vlen - 2) + 1LL) %
        static_cast<int64_T>(blockStride);
    if (m > 0U) {
      blockStride = m;
    }
  }
  blockStride = ((blockStride + warpSize) - 1U) / warpSize;
  if (static_cast<int64_T>(threadId) <= loopEnd) {
    tmpRed0 = x[static_cast<int32_T>(threadId) + 1];
  }
  m = __ballot_sync(MAX_uint32_T, static_cast<int64_T>(threadId) <= loopEnd);
  for (uint32_T idx{threadId + threadStride};
       idx <= static_cast<uint32_T>(loopEnd); idx += threadStride) {
    tmpRed0 += x[static_cast<int32_T>(idx) + 1];
  }
  tmpRed0 = workGroupReduction(tmpRed0, m, blockStride);
  if ((static_cast<int32_T>(static_cast<int64_T>(threadId) <= loopEnd)) &&
      (static_cast<int32_T>(thBlkId == 0U))) {
    atomicOpreal_T(&muj[0], tmpRed0);
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel56(
    const real_T xRinv_data[58000], int32_T *m, real_T y_data[58000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    real_T muj;
    int32_T k;
    k = static_cast<int32_T>(idx);
    muj = xRinv_data[k];
    y_data[k] = muj * muj;
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel57(
    const real_T y_data[58000], real_T quadform[1000])
{
  uint64_T threadId;
  int32_T j;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int32_T>(threadId);
  if (j < 1000) {
    quadform[j] = y_data[j];
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel58(
    const real_T y_data[58000], const int32_T *LDA, real_T quadform[1000])
{
  uint64_T threadId;
  int32_T j;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int32_T>(threadId);
  if (j < 1000) {
    quadform[j] += y_data[*LDA + j];
  }
}

static __global__
    __launch_bounds__(512, 1) void BaumWelch_kernel59(real_T quadform[1000])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 1000) {
    quadform[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel6(
    const int32_T x_size_dim0, const int32_T *m, real_T x_data[58000])
{
  uint64_T threadId;
  int32_T j;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  j = static_cast<int32_T>(threadId);
  if (j < 58) {
    real_T muj;
    muj = 0.0;
    for (int32_T i{0}; i < *m; i++) {
      muj += x_data[i + x_size_dim0 * j];
    }
    muj /= static_cast<real_T>(*m);
    for (int32_T i{0}; i < *m; i++) {
      x_data[i + x_size_dim0 * j] -= muj;
    }
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel60(
    const real_T quadform[1000], const int32_T i, real_T *muj,
    emxArray_real_T B)
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 1000) {
    B.data[k + 1000 * i] =
        exp((-0.5 * quadform[k] - *muj) - 53.298434925871014);
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel61(
    const emxArray_int32_T iv1, int32_T kEnd, emxArray_real_T B)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(kEnd - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    B.data[iv1.data[xpageoffset] - 1] = 1.0E-200;
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel62(real_T N,
                                                       emxArray_real_T alpha)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(1000 * static_cast<int32_T>(N) - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    alpha.data[xpageoffset] = 0.0;
  }
}

static __global__
    __launch_bounds__(512, 1) void BaumWelch_kernel63(real_T scale[1000])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 1000) {
    scale[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel64(
    const emxArray_real_T B, const emxArray_real_T Pi, int32_T b_Pi,
    emxArray_real_T alpha)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_Pi);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    alpha.data[1000 * xpageoffset] =
        Pi.data[xpageoffset] * B.data[1000 * xpageoffset];
  }
}

static __global__
    __launch_bounds__(32, 1) void BaumWelch_kernel65(real_T *muj,
                                                     real_T scale[1000])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    scale[0] = *muj;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel66(
    const emxArray_real_T alpha, int32_T kEnd, real_T *muj,
    emxArray_real_T b_alpha)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(kEnd);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b_alpha.data[xpageoffset] = alpha.data[1000 * xpageoffset] / *muj;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel67(
    const emxArray_real_T alpha, int32_T b_alpha, emxArray_real_T c_alpha)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_alpha);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    c_alpha.data[1000 * xpageoffset] = alpha.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel68(
    const int32_T i, const emxArray_real_T trans, const emxArray_real_T alpha,
    int32_T kEnd, const int32_T trans_dim0, int32_T *info_t, emxArray_real_T x)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(kEnd);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    x.data[xpageoffset] = alpha.data[*info_t + 1000 * xpageoffset] *
                          trans.data[xpageoffset + trans_dim0 * i];
  }
}

static __global__ __launch_bounds__(32, 1) void BaumWelch_kernel69(
    const emxArray_real_T B, const int32_T i, real_T *muj, int32_T *info_t,
    emxArray_real_T alpha)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    alpha.data[(*info_t + 1000 * i) + 1] =
        *muj * B.data[(*info_t + 1000 * i) + 1];
  }
}

static __global__ __launch_bounds__(512,
                                    1) void BaumWelch_kernel7(real_T C[3364])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 3364) {
    C[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void BaumWelch_kernel70(
    real_T *muj, int32_T *info_t, real_T scale[1000])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    scale[*info_t + 1] = *muj;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel71(
    const emxArray_real_T alpha, const int32_T *m, int32_T kEnd, real_T *muj,
    emxArray_real_T b_alpha)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(kEnd);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b_alpha.data[xpageoffset] = alpha.data[*m + 1000 * xpageoffset] / *muj;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel72(
    const emxArray_real_T alpha, const int32_T *LDA, int32_T b_alpha,
    emxArray_real_T c_alpha)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_alpha);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    c_alpha.data[*LDA + 1000 * xpageoffset] = alpha.data[xpageoffset];
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel73(real_T N,
                                                       emxArray_real_T beta)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(1000 * static_cast<int32_T>(N) - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    beta.data[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel74(
    const real_T scale[1000], int32_T unnamed_idx_1, emxArray_real_T beta)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(unnamed_idx_1 - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    beta.data[1000 * xpageoffset + 999] = 1.0 / scale[999];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel75(
    const emxArray_real_T beta, const int32_T vlen, const emxArray_real_T B,
    const int32_T kEnd, const emxArray_real_T trans, const int32_T i,
    int32_T *LDA, const int32_T trans_dim0, emxArray_real_T x)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*LDA);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    x.data[xpageoffset] = trans.data[i + trans_dim0 * xpageoffset] *
                          B.data[kEnd + 1000 * xpageoffset] *
                          beta.data[vlen + 1000 * xpageoffset];
  }
}

static __global__ __launch_bounds__(32, 1) void BaumWelch_kernel76(
    const real_T scale[1000], const int32_T i, real_T *muj, int32_T *info_t,
    emxArray_real_T beta)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    beta.data[(1000 * i - *info_t) + 998] = *muj / scale[998 - *info_t];
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel77(real_T N,
                                                       emxArray_real_T Xi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(
      999 * static_cast<int32_T>(N) * static_cast<int32_T>(N) - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    Xi.data[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel78(
    const emxArray_real_T beta, const emxArray_real_T B,
    const emxArray_real_T trans, const emxArray_real_T alpha, const real_T N,
    int32_T i11, const int32_T Xi_dim1, const int32_T trans_dim0,
    int32_T *info_t, emxArray_real_T Xi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(static_cast<int32_T>(N) - 1) + 1ULL) *
                (static_cast<uint64_T>(i11) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T j;
    j = static_cast<int32_T>(
        idx % (static_cast<uint64_T>(static_cast<int32_T>(N) - 1) + 1ULL));
    i = static_cast<int32_T>(
        (idx - static_cast<uint64_T>(j)) /
        (static_cast<uint64_T>(static_cast<int32_T>(N) - 1) + 1ULL));
    Xi.data[(*info_t + 999 * i) + 999 * Xi_dim1 * j] =
        alpha.data[*info_t + 1000 * i] * trans.data[i + trans_dim0 * j] *
        B.data[(*info_t + 1000 * j) + 1] * beta.data[(*info_t + 1000 * j) + 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel79(
    const emxArray_real_T Xi, const int32_T kEnd, int32_T vlen,
    const int32_T Xi_dim1, const int32_T b_Xi_dim1, real_T *muj,
    int32_T *info_t, emxArray_real_T b_Xi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(kEnd) + 1ULL) *
                (static_cast<uint64_T>(vlen) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(kEnd) + 1ULL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                       (static_cast<uint64_T>(kEnd) + 1ULL));
    b_Xi.data[k + Xi_dim1 * xpageoffset] =
        Xi.data[(*info_t + 999 * k) + 999 * b_Xi_dim1 * xpageoffset] / *muj;
  }
}

static __global__ __launch_bounds__(512,
                                    1) void BaumWelch_kernel8(real_T C[3364])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 3364) {
    C[xpageoffset] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel80(
    const emxArray_real_T Xi, const int32_T b_Xi, int32_T c_Xi,
    const int32_T Xi_dim1, const int32_T b_Xi_dim1, int32_T *info_t,
    emxArray_real_T d_Xi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_Xi) + 1ULL) *
                (static_cast<uint64_T>(c_Xi) + 1ULL) -
            1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_Xi) + 1ULL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                       (static_cast<uint64_T>(b_Xi) + 1ULL));
    d_Xi.data[(*info_t + 999 * k) + 999 * Xi_dim1 * xpageoffset] =
        Xi.data[k + b_Xi_dim1 * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel81(
    const emxArray_real_T Xi, int32_T *m, emxArray_real_T Gamma)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    Gamma.data[j] = Xi.data[j];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel82(
    const emxArray_real_T Xi, const int32_T *LDA, int32_T *m,
    emxArray_real_T Gamma)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*m);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    Gamma.data[j] += Xi.data[*LDA + j];
  }
}

static __global__
    __launch_bounds__(1024, 1) void BaumWelch_kernel83(int32_T sz,
                                                       emxArray_real_T Gamma)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(sz);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    Gamma.data[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel84(
    const emxArray_real_T Gamma, int32_T kEnd, emxArray_real_T Pi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(kEnd);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    Pi.data[xpageoffset] = Gamma.data[999 * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel85(
    const emxArray_real_T Xi, const int32_T i, int32_T b_i,
    const int32_T Xi_dim1, emxArray_real_T a)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = 999ULL * (static_cast<uint64_T>(b_i) + 1ULL) - 1ULL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    int32_T xpageoffset;
    k = static_cast<int32_T>(idx % 999ULL);
    xpageoffset =
        static_cast<int32_T>((idx - static_cast<uint64_T>(k)) / 999ULL);
    a.data[k + 999 * xpageoffset] =
        Xi.data[(k + 999 * i) + 999 * Xi_dim1 * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel86(
    const emxArray_real_T a, int32_T *LDA, emxArray_real_T y)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(*LDA);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    real_T muj;
    int32_T i;
    int32_T xpageoffset;
    i = static_cast<int32_T>(idx);
    xpageoffset = i * 999;
    muj = a.data[xpageoffset];
    for (int32_T k{0}; k < 998; k++) {
      muj += a.data[(xpageoffset + k) + 1];
    }
    y.data[i] = muj;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel87(
    const emxArray_real_T y, const int32_T i, int32_T b_y,
    const int32_T trans_dim0, real_T *muj, emxArray_real_T trans)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    trans.data[i + trans_dim0 * xpageoffset] = y.data[xpageoffset] / *muj;
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel88(
    const emxArray_real_T Gamma, const int32_T i, const real_T X[58000],
    real_T x[57942])
{
  uint64_T threadId;
  int32_T k;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 999ULL);
  xpageoffset =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 999ULL);
  if ((static_cast<int32_T>(xpageoffset < 58)) &&
      (static_cast<int32_T>(k < 999))) {
    x[k + 999 * xpageoffset] =
        X[k + 1000 * xpageoffset] * Gamma.data[k + 999 * i];
  }
}

static __global__
    __launch_bounds__(64, 1) void BaumWelch_kernel89(const real_T x[57942],
                                                     real_T y[58])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 58) {
    real_T muj;
    int32_T xpageoffset;
    xpageoffset = i * 999;
    muj = x[xpageoffset];
    for (int32_T k{0}; k < 998; k++) {
      muj += x[(xpageoffset + k) + 1];
    }
    y[i] = muj;
  }
}

static __global__
    __launch_bounds__(512, 1) void BaumWelch_kernel9(real_T C[3364],
                                                     real_T dv_data[3364])
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 3364) {
    dv_data[xpageoffset] = C[xpageoffset];
  }
}

static __global__ __launch_bounds__(64, 1) void BaumWelch_kernel90(
    const real_T y[58], const int32_T i, const int32_T Mu_dim0, real_T *muj,
    emxArray_real_T Mu)
{
  uint64_T threadId;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId);
  if (xpageoffset < 58) {
    Mu.data[i + Mu_dim0 * xpageoffset] = y[xpageoffset] / *muj;
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel91(
    const emxArray_real_T Mu, const int32_T i, const real_T X[58000],
    const int32_T Mu_dim0, real_T d[57942])
{
  uint64_T threadId;
  int32_T k;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 999ULL);
  xpageoffset =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 999ULL);
  if ((static_cast<int32_T>(xpageoffset < 58)) &&
      (static_cast<int32_T>(k < 999))) {
    d[k + 999 * xpageoffset] =
        X[k + 1000 * xpageoffset] - Mu.data[i + Mu_dim0 * xpageoffset];
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel92(
    const emxArray_real_T Gamma, const int32_T i, real_T d[57942],
    real_T A[57942])
{
  uint64_T threadId;
  int32_T k;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 58ULL);
  xpageoffset =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 58ULL);
  if ((static_cast<int32_T>(xpageoffset < 999)) &&
      (static_cast<int32_T>(k < 58))) {
    A[k + 58 * xpageoffset] =
        d[xpageoffset + 999 * k] * Gamma.data[xpageoffset + 999 * i];
  }
}

static __global__ __launch_bounds__(512, 1) void BaumWelch_kernel93(
    const int32_T i, real_T *muj, real_T y[3364], emxArray_real_T Cov)
{
  uint64_T threadId;
  int32_T k;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 58ULL);
  xpageoffset =
      static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 58ULL);
  if ((static_cast<int32_T>(xpageoffset < 58)) &&
      (static_cast<int32_T>(k < 58))) {
    Cov.data[(k + 58 * xpageoffset) + 3364 * i] =
        y[k + 58 * xpageoffset] / *muj;
  }
}

static __global__ __launch_bounds__(1024, 1) void BaumWelch_kernel94(
    const int32_T iwork[1000], const int32_T j, int32_T kEnd, int32_T idx[1000])
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(kEnd - 1);
  for (uint64_T b_idx{threadId}; b_idx <= loopEnd; b_idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(b_idx);
    idx[(j + k) - 1] = iwork[k];
  }
}

static __device__ real_T atomicOpreal_T(real_T *address, real_T value)
{
  unsigned long long int old;
  unsigned long long int *address_as_up;
  address_as_up = (unsigned long long int *)address;
  old = *address_as_up;
  unsigned long long int assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_up, old,
                    __double_as_longlong(value + __longlong_as_double(old)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

static void b_error(const mxArray *m, const mxArray *m1, emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  pArrays[0] = m;
  pArrays[1] = m1;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, nullptr, 2, &pArrays[0],
                        (const char_T *)"error", true, location);
}

static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             int32_T in3, const emxArray_real_T *in4,
                             int32_T in5)
{
  int32_T i;
  int32_T i1;
  int32_T i2;
  int32_T stride_0_1;
  int32_T stride_1_1;
  i = in2->size[1];
  i1 = in4->size[0];
  i2 = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  if (i1 == 1) {
    in1->size[1] = i;
  } else {
    in1->size[1] = i1;
  }
  emxEnsureCapacity_real_T(in1, i2, &p_emlrtRTEI);
  stride_0_1 = (i != 1);
  stride_1_1 = (i1 != 1);
  if (i1 == 1) {
    i1 = i;
  }
  for (i2 = 0; i2 < i1; i2++) {
    in1->data[i2] = in2->data[in3 + 1000 * (i2 * stride_0_1)] *
                    in4->data[i2 * stride_1_1 + in4->size[0] * in5];
  }
}

static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             int32_T in3, const emxArray_real_T *in4,
                             int32_T in5, const emxArray_real_T *in6)
{
  int32_T i;
  int32_T i1;
  int32_T i2;
  int32_T i3;
  int32_T stride_0_1;
  int32_T stride_1_1;
  int32_T stride_2_1;
  i = in2->size[1];
  i1 = in4->size[1];
  i2 = in6->size[1];
  i3 = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  if (i2 == 1) {
    if (i1 == 1) {
      in1->size[1] = i;
    } else {
      in1->size[1] = i1;
    }
  } else {
    in1->size[1] = i2;
  }
  emxEnsureCapacity_real_T(in1, i3, &s_emlrtRTEI);
  stride_0_1 = (i != 1);
  stride_1_1 = (i1 != 1);
  stride_2_1 = (i2 != 1);
  if (i2 == 1) {
    if (i1 == 1) {
      i2 = i;
    } else {
      i2 = i1;
    }
  }
  for (i3 = 0; i3 < i2; i3++) {
    in1->data[i3] = in2->data[in3 + in2->size[0] * (i3 * stride_0_1)] *
                    in4->data[(1000 * (i3 * stride_1_1) - in5) + 999] *
                    in6->data[(1000 * (i3 * stride_2_1) - in5) + 999];
  }
}

static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in3,
                             const emxArray_real_T *in4)
{
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  i = in4->size[1];
  stride_0_1 = (in3->size[1] != 1);
  stride_1_1 = (i != 1);
  if (i == 1) {
    i = in3->size[1];
  }
  for (int32_T i1{0}; i1 < i; i1++) {
    in1->data[1000 * i1] =
        in3->data[i1 * stride_0_1] * in4->data[1000 * (i1 * stride_1_1)];
  }
}

static void gpuEmxEnsureCapacity_int32_T(const emxArray_int32_T *cpu,
                                         emxArray_int32_T *gpu)
{
  int32_T *newData;
  if (gpu->data == 0) {
    newData = 0ULL;
    hipMalloc(&newData, cpu->allocatedSize * sizeof(int32_T));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0ULL;
      hipMalloc(&newData, cpu->allocatedSize * sizeof(int32_T));
      hipMemcpy(newData, gpu->data, actualSizeGpu * sizeof(int32_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        hipFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  real_T *newData;
  if (gpu->data == 0) {
    newData = 0ULL;
    hipMalloc(&newData, cpu->allocatedSize * sizeof(real_T));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0ULL;
      hipMalloc(&newData, cpu->allocatedSize * sizeof(real_T));
      hipMemcpy(newData, gpu->data, actualSizeGpu * sizeof(real_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        hipFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxFree_int32_T(emxArray_int32_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    hipFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    hipFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_int32_T(emxArray_int32_T *gpu,
                                         const emxArray_int32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      hipFree(gpu->data);
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(int32_T));
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(int32_T),
             hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      hipFree(gpu->data);
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(real_T));
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(real_T),
             hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(real_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxReset_int32_T(emxArray_int32_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_int32_T));
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real_T));
}

static __device__ real_T shflDown2(real_T in1, uint32_T offset, uint32_T mask)
{
  int2 tmp;
  tmp.x = __shfl_down_sync(mask, ((int2 *)&in1)->x, offset);
  tmp.y = __shfl_down_sync(mask, ((int2 *)&in1)->y, offset);
  return *(real_T *)&tmp;
}

static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
                                              uint32_T mask)
{
  uint32_T activeSize;
  uint32_T offset;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    real_T other;
    other = shflDown2(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }
    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }
  return val;
}

static __device__ real_T workGroupReduction(real_T val, uint32_T mask,
                                            uint32_T numActiveWarps)
{
  __shared__ real_T shared[32];
  uint32_T lane;
  uint32_T thBlkId;
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  lane = thBlkId % warpSize;
  thBlkId /= warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[thBlkId] = val;
  }
  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (thBlkId == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }
  return val;
}

void BaumWelch(BaumWelchStackData *SD, const real_T X[58000], real_T N,
               emxArray_real_T *Mu, emxArray_real_T *Cov,
               emxArray_real_T *trans, emxArray_real_T *Pi, real_T *Time)
{
  static const int32_T iv2[2]{1, 26};
  static const int32_T iv3[2]{1, 46};
  static const char_T varargin_2[46]{
      'S', 'I', 'G', 'M', 'A', ' ', 'm', 'u', 's', 't', ' ', 'b',
      'e', ' ', 's', 'y', 'm', 'm', 'e', 't', 'r', 'i', 'c', ' ',
      'a', 'n', 'd', ' ', 'p', 'o', 's', 'i', 't', 'i', 'v', 'e',
      ' ', 'd', 'e', 'f', 'i', 'n', 'i', 't', 'e', '.'};
  static const char_T varargin_1[26]{
      's', 't', 'a', 't', 's', ':', 'm', 'v', 'n', 'p', 'd', 'f', ':',
      'B', 'a', 'd', 'C', 'o', 'v', 'a', 'r', 'i', 'a', 'n', 'c', 'e'};
  ptrdiff_t jpvt_t_data[58];
  ptrdiff_t(*gpu_jpvt_t_data)[58];
  cell_wrap_0 idx1[4];
  cell_wrap_0(*gpu_idx1)[4];
  dim3 block;
  dim3 grid;
  emlrtTimespec expl_temp;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T *iv1;
  emxArray_real_T b_gpu_Xi;
  emxArray_real_T b_gpu_alpha;
  emxArray_real_T b_gpu_trans;
  emxArray_real_T b_gpu_x;
  emxArray_real_T b_gpu_y;
  emxArray_real_T c_gpu_alpha;
  emxArray_real_T c_gpu_x;
  emxArray_real_T c_gpu_y;
  emxArray_real_T d_gpu_x;
  emxArray_real_T f_gpu_y;
  emxArray_real_T gpu_B;
  emxArray_real_T gpu_Cov;
  emxArray_real_T gpu_Gamma;
  emxArray_real_T gpu_Mu;
  emxArray_real_T gpu_Pi;
  emxArray_real_T gpu_Xi;
  emxArray_real_T gpu_a;
  emxArray_real_T gpu_alpha;
  emxArray_real_T gpu_beta;
  emxArray_real_T gpu_trans;
  emxArray_real_T *B;
  emxArray_real_T *Gamma;
  emxArray_real_T *Xi;
  emxArray_real_T *a;
  emxArray_real_T *alpha;
  emxArray_real_T *b_Xi;
  emxArray_real_T *b_alpha;
  emxArray_real_T *b_trans;
  emxArray_real_T *b_x;
  emxArray_real_T *b_y;
  emxArray_real_T *beta;
  emxArray_real_T *c_alpha;
  emxArray_real_T *c_x;
  emxArray_real_T *c_y;
  emxArray_real_T *d_x;
  emxArray_real_T *y;
  const mxArray *b_m;
  const mxArray *e_y;
  const mxArray *f_y;
  real_T(*b_gpu_B)[58000];
  real_T(*gpu_X)[58000];
  real_T(*gpu_X0)[58000];
  real_T(*gpu_Y_data)[58000];
  real_T(*gpu_xRinv_data)[58000];
  real_T(*gpu_x_data)[58000];
  real_T(*gpu_y_data)[58000];
  real_T(*f_gpu_x)[57942];
  real_T(*gpu_A)[57942];
  real_T(*gpu_d)[57942];
  real_T C[3364];
  real_T dv_data[3364];
  real_T(*d_gpu_y)[3364];
  real_T(*e_gpu_x)[3364];
  real_T(*gpu_C)[3364];
  real_T(*gpu_T_data)[3364];
  real_T(*gpu_dv_data)[3364];
  real_T(*gpu_quadform)[1000];
  real_T(*gpu_scale)[1000];
  real_T x[58];
  real_T(*gpu_x)[58];
  real_T(*gpu_y)[58];
  real_T b_muj;
  real_T beta1;
  real_T *gpu_muj;
  int32_T idx[1000];
  int32_T iwork[1000];
  int32_T(*gpu_idx)[1000];
  int32_T(*gpu_iwork)[1000];
  int32_T ipiv_data[58];
  int32_T(*gpu_ipiv_data)[58];
  int32_T(*gpu_ipiv_t)[58];
  int32_T T_size[2];
  int32_T b_x_size[2];
  int32_T x_size[2];
  int32_T LDA;
  int32_T b_i;
  int32_T b_vlen;
  int32_T exponent;
  int32_T i;
  int32_T i10;
  int32_T i13;
  int32_T i14;
  int32_T i8;
  int32_T i9;
  int32_T info_t;
  int32_T j;
  int32_T k;
  int32_T kEnd;
  int32_T m;
  int32_T vlen;
  int32_T xpageoffset;
  int32_T *gpu_LDA;
  int32_T *gpu_info_t;
  int32_T *gpu_m;
  int8_T dv_size[2];
  boolean_T d_y[58];
  boolean_T(*e_gpu_y)[58];
  boolean_T B_dirtyOnGpu;
  boolean_T C_dirtyOnCpu;
  boolean_T Cov_dirtyOnCpu;
  boolean_T Cov_dirtyOnGpu;
  boolean_T Gamma_dirtyOnGpu;
  boolean_T Mu_dirtyOnCpu;
  boolean_T Mu_dirtyOnGpu;
  boolean_T Pi_dirtyOnCpu;
  boolean_T Pi_dirtyOnGpu;
  boolean_T Xi_dirtyOnGpu;
  boolean_T alpha_dirtyOnCpu;
  boolean_T alpha_dirtyOnGpu;
  boolean_T b_x_dirtyOnCpu;
  boolean_T b_x_dirtyOnGpu;
  boolean_T beta_dirtyOnGpu;
  boolean_T dv_data_dirtyOnGpu;
  boolean_T exitg1;
  boolean_T idx1_dirtyOnGpu;
  boolean_T idx_dirtyOnGpu;
  boolean_T info_t_dirtyOnCpu;
  boolean_T iv1_dirtyOnCpu;
  boolean_T iwork_dirtyOnCpu;
  boolean_T jpvt_t_data_dirtyOnCpu;
  boolean_T jpvt_t_data_dirtyOnGpu;
  boolean_T p;
  boolean_T trans_dirtyOnCpu;
  boolean_T trans_dirtyOnGpu;
  boolean_T validLaunchParams;
  boolean_T x_dirtyOnCpu;
  boolean_T x_dirtyOnGpu;
  boolean_T y_dirtyOnGpu;
  hipMalloc(&gpu_Y_data, 464000ULL);
  hipMalloc(&gpu_quadform, 8000ULL);
  hipMalloc(&gpu_y_data, 464000ULL);
  hipMalloc(&gpu_xRinv_data, 464000ULL);
  hipMalloc(&gpu_jpvt_t_data, 464ULL);
  hipMalloc(&b_gpu_B, 464000ULL);
  hipMalloc(&gpu_ipiv_data, 232ULL);
  hipMalloc(&gpu_ipiv_t, 232ULL);
  gpuEmxReset_real_T(&f_gpu_y);
  hipMalloc(&gpu_T_data, 26912ULL);
  hipMalloc(&gpu_A, 463536ULL);
  hipMalloc(&gpu_d, 463536ULL);
  gpuEmxReset_real_T(&gpu_a);
  hipMalloc(&f_gpu_x, 463536ULL);
  gpuEmxReset_real_T(&b_gpu_Xi);
  hipMalloc(&e_gpu_y, 58ULL);
  hipMalloc(&d_gpu_y, 26912ULL);
  hipMalloc(&e_gpu_x, 26912ULL);
  gpuEmxReset_real_T(&gpu_Gamma);
  gpuEmxReset_real_T(&d_gpu_x);
  gpuEmxReset_real_T(&gpu_Xi);
  gpuEmxReset_real_T(&c_gpu_alpha);
  gpuEmxReset_real_T(&c_gpu_x);
  gpuEmxReset_real_T(&gpu_beta);
  hipMalloc(&gpu_info_t, 4ULL);
  hipMalloc(&gpu_X0, 464000ULL);
  gpuEmxReset_real_T(&b_gpu_alpha);
  hipMalloc(&gpu_scale, 8000ULL);
  gpuEmxReset_real_T(&gpu_alpha);
  gpuEmxReset_real_T(&gpu_B);
  gpuEmxReset_int32_T(&gpu_iv1);
  gpuEmxReset_real_T(&b_gpu_trans);
  gpuEmxReset_real_T(&c_gpu_y);
  gpuEmxReset_real_T(&b_gpu_y);
  hipMalloc(&gpu_LDA, 4ULL);
  gpuEmxReset_real_T(&b_gpu_x);
  gpuEmxReset_real_T(&gpu_trans);
  gpuEmxReset_real_T(&gpu_Mu);
  hipMalloc(&gpu_y, 464ULL);
  hipMalloc(&gpu_dv_data, 26912ULL);
  gpuEmxReset_real_T(&gpu_Pi);
  hipMalloc(&gpu_m, 4ULL);
  hipMalloc(&gpu_muj, 8ULL);
  hipMalloc(&gpu_x, 464ULL);
  hipMalloc(&gpu_C, 26912ULL);
  hipMalloc(&gpu_x_data, 464000ULL);
  hipMalloc(&gpu_iwork, 4000ULL);
  gpuEmxReset_real_T(&gpu_Cov);
  hipMalloc(&gpu_idx1, 32032ULL);
  hipMalloc(&gpu_idx, 4000ULL);
  hipMalloc(&gpu_X, 464000ULL);
  jpvt_t_data_dirtyOnGpu = false;
  Gamma_dirtyOnGpu = false;
  b_x_dirtyOnGpu = false;
  Xi_dirtyOnGpu = false;
  x_dirtyOnGpu = false;
  beta_dirtyOnGpu = false;
  alpha_dirtyOnGpu = false;
  B_dirtyOnGpu = false;
  y_dirtyOnGpu = false;
  dv_data_dirtyOnGpu = false;
  idx1_dirtyOnGpu = false;
  jpvt_t_data_dirtyOnCpu = false;
  b_x_dirtyOnCpu = false;
  x_dirtyOnCpu = false;
  alpha_dirtyOnCpu = false;
  iv1_dirtyOnCpu = false;
  C_dirtyOnCpu = false;
  iwork_dirtyOnCpu = false;
  trans_dirtyOnGpu = false;
  trans_dirtyOnCpu = true;
  Mu_dirtyOnGpu = false;
  Mu_dirtyOnCpu = true;
  Pi_dirtyOnGpu = false;
  Pi_dirtyOnCpu = true;
  Cov_dirtyOnGpu = false;
  Cov_dirtyOnCpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  Baum Welch algorithm to train Hidden Markov Model
  //
  //   Input:
  //   - X: Observations (MxT)
  //   - N: Number of states
  //   (optional inputs)
  //   - cyc: Number of iterations of the forward-backward algorithm
  //   - tol: Tolerance to stop iterations
  //
  //   Output:
  //   - Mu: Means of the normal distributions for each
  //         observation in each state (MxN)
  //   - Cov: Covariance of the normal distributions for eachs
  //          observation in each state (MxMxN)
  //   - Pi: Initial state probabilities (1xN)
  //   - trans: Transition matrix (NxN)
  expl_temp = coder::tic();
  //  Find the initial means and covariance matrices for each of the states
  //  Split the observations into evenly size states from smallest to largest
  // [idx]=Divide(X,N,'sort');
  hipMemcpy(*gpu_X, X, 464000ULL, hipMemcpyHostToDevice);
  BaumWelch_kernel1<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_X, *gpu_idx);
  idx_dirtyOnGpu = true;
  i = 2;
  while (i < 1000) {
    xpageoffset = i << 1;
    j = 1;
    for (m = i + 1; m < 1001; m = vlen + i) {
      LDA = j - 1;
      info_t = m - 1;
      vlen = j + xpageoffset;
      if (vlen > 1001) {
        vlen = 1001;
      }
      k = 0;
      kEnd = vlen - j;
      while (k + 1 <= kEnd) {
        p = true;
        b_vlen = 0;
        exitg1 = false;
        while ((!exitg1) && (b_vlen + 1 < 59)) {
          if (idx_dirtyOnGpu) {
            hipMemcpy(idx, *gpu_idx, 4000ULL, hipMemcpyDeviceToHost);
          }
          idx_dirtyOnGpu = false;
          if ((X[(idx[LDA] + 1000 * b_vlen) - 1] ==
               X[(idx[info_t] + 1000 * b_vlen) - 1]) ||
              (std::isnan(X[(idx[LDA] + 1000 * b_vlen) - 1]) &&
               std::isnan(X[(idx[info_t] + 1000 * b_vlen) - 1]))) {
            b_vlen++;
          } else {
            if ((!(X[(idx[LDA] + 1000 * b_vlen) - 1] <=
                   X[(idx[info_t] + 1000 * b_vlen) - 1])) &&
                (!std::isnan(X[(idx[info_t] + 1000 * b_vlen) - 1]))) {
              p = false;
            }
            exitg1 = true;
          }
        }
        if (p) {
          if (idx_dirtyOnGpu) {
            hipMemcpy(idx, *gpu_idx, 4000ULL, hipMemcpyDeviceToHost);
          }
          idx_dirtyOnGpu = false;
          iwork[k] = idx[LDA];
          iwork_dirtyOnCpu = true;
          LDA++;
          if (LDA + 1 == m) {
            while (info_t + 1 < vlen) {
              k++;
              iwork[k] = idx[info_t];
              info_t++;
            }
          }
        } else {
          if (idx_dirtyOnGpu) {
            hipMemcpy(idx, *gpu_idx, 4000ULL, hipMemcpyDeviceToHost);
          }
          idx_dirtyOnGpu = false;
          iwork[k] = idx[info_t];
          iwork_dirtyOnCpu = true;
          info_t++;
          if (info_t + 1 == vlen) {
            while (LDA + 1 < m) {
              k++;
              iwork[k] = idx[LDA];
              LDA++;
            }
          }
        }
        k++;
      }
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>((kEnd - 1) + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        if (iwork_dirtyOnCpu) {
          hipMemcpy(*gpu_iwork, iwork, 4000ULL, hipMemcpyHostToDevice);
        }
        iwork_dirtyOnCpu = false;
        BaumWelch_kernel94<<<grid, block>>>(*gpu_iwork, j, kEnd, *gpu_idx);
        idx_dirtyOnGpu = true;
      }
      j = vlen;
    }
    i = xpageoffset;
  }
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>((static_cast<int32_T>(N) - 1) + 1LL), &grid, &block,
      1024U, 65535U);
  if (validLaunchParams) {
    BaumWelch_kernel2<<<grid, block>>>(*gpu_idx, N, static_cast<int32_T>(N) - 1,
                                       *gpu_idx1);
    idx1_dirtyOnGpu = true;
  }
  kEnd = Mu->size[0] * Mu->size[1];
  Mu->size[0] = static_cast<int32_T>(N);
  Mu->size[1] = 58;
  emxEnsureCapacity_real_T(Mu, kEnd, &emlrtRTEI);
  kEnd = Cov->size[0] * Cov->size[1] * Cov->size[2];
  Cov->size[0] = 58;
  Cov->size[1] = 58;
  Cov->size[2] = static_cast<int32_T>(N);
  emxEnsureCapacity_real_T(Cov, kEnd, &b_emlrtRTEI);
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>((3364 * static_cast<int32_T>(N) - 1) + 1LL), &grid,
      &block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_Cov, Cov);
    BaumWelch_kernel3<<<grid, block>>>(N, gpu_Cov);
    Cov_dirtyOnCpu = false;
    Cov_dirtyOnGpu = true;
  }
  info_t = static_cast<int32_T>(N);
  info_t_dirtyOnCpu = true;
  for (i = 0; i < info_t; i++) {
    if (idx1_dirtyOnGpu) {
      hipMemcpy(idx1, *gpu_idx1, 32032ULL, hipMemcpyDeviceToHost);
    }
    kEnd = idx1[i].f1.size[0];
    x_size[0] = idx1[i].f1.size[0];
    xpageoffset = idx1[i].f1.size[0] - 1;
    validLaunchParams =
        mwGetLaunchParameters1D(static_cast<real_T>((xpageoffset + 1LL) * 58LL),
                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      BaumWelch_kernel4<<<grid, block>>>(*gpu_X, *gpu_idx1, i, x_size[0],
                                         xpageoffset, *gpu_x_data);
    }
    if (kEnd == 1) {
      BaumWelch_kernel10<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
          x_size[0], *gpu_x_data, *gpu_x);
      b_muj = 0.0;
      hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
      BaumWelch_kernel11<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
          x_size[0], *gpu_x_data, gpu_muj);
      hipMemcpy(&b_muj, gpu_muj, 8ULL, hipMemcpyDeviceToHost);
      b_muj /= 58.0;
      hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
      BaumWelch_kernel12<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(gpu_muj,
                                                                  *gpu_x);
      iwork_dirtyOnCpu = true;
      b_muj = 0.0;
      for (xpageoffset = 0; xpageoffset < 58; xpageoffset++) {
        real_T muj;
        if (iwork_dirtyOnCpu) {
          hipMemcpy(x, *gpu_x, 464ULL, hipMemcpyDeviceToHost);
        }
        iwork_dirtyOnCpu = false;
        muj = x[xpageoffset];
        b_muj += muj * muj;
      }
      dv_size[0] = 1;
      hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
      BaumWelch_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_muj,
                                                                  *gpu_dv_data);
    } else {
      BaumWelch_kernel5<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_C);
      if (x_size[0] == 0) {
        BaumWelch_kernel8<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_C);
      } else if (x_size[0] >= 2) {
        hipMemcpy(gpu_m, &x_size[0], 4ULL, hipMemcpyHostToDevice);
        BaumWelch_kernel6<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
            x_size[0], gpu_m, *gpu_x_data);
        b_muj = 1.0 / (static_cast<real_T>(x_size[0]) - 1.0);
        BaumWelch_kernel7<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_C);
        beta1 = 0.0;
        hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_C, HIPBLAS_OP_N, 58, 58,
                    x_size[0], (double *)&b_muj, (double *)&(*gpu_x_data)[0],
                    x_size[0], (double *)&(*gpu_x_data)[0], x_size[0],
                    (double *)&beta1, (double *)&(*gpu_C)[0], 58);
      }
      dv_size[0] = 58;
      BaumWelch_kernel9<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_C,
                                                                  *gpu_dv_data);
    }
    if (Cov_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_Cov, Cov);
    }
    BaumWelch_kernel14<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        dv_size[0], i, *gpu_dv_data, gpu_Cov);
    dv_data_dirtyOnGpu = true;
    Cov_dirtyOnCpu = false;
    Cov_dirtyOnGpu = true;
    b_x_size[0] = idx1[i].f1.size[0];
    xpageoffset = idx1[i].f1.size[0] - 1;
    validLaunchParams =
        mwGetLaunchParameters1D(static_cast<real_T>((xpageoffset + 1LL) * 58LL),
                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      BaumWelch_kernel15<<<grid, block>>>(*gpu_X, *gpu_idx1, i, b_x_size[0],
                                          xpageoffset, *gpu_x_data);
    }
    vlen = idx1[i].f1.size[0];
    if (vlen == 0) {
      BaumWelch_kernel17<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*gpu_y);
    } else {
      BaumWelch_kernel16<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
          vlen, *gpu_idx1, i, *gpu_x_data, *gpu_y);
    }
    idx1_dirtyOnGpu = false;
    xpageoffset = Mu->size[0];
    hipMemcpy(gpu_m, &idx1[i].f1.size[0], 4ULL, hipMemcpyHostToDevice);
    if (Mu_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_Mu, Mu);
    }
    BaumWelch_kernel18<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
        gpu_m, *gpu_y, i, xpageoffset, gpu_Mu);
    Mu_dirtyOnCpu = false;
    Mu_dirtyOnGpu = true;
  }
  //  Initialise Priors
  kEnd = Pi->size[0] * Pi->size[1];
  Pi->size[0] = 1;
  Pi->size[1] = static_cast<int32_T>(N);
  emxEnsureCapacity_real_T(Pi, kEnd, &c_emlrtRTEI);
  if (static_cast<int32_T>(N) != 0) {
    emlrtRandu(&Pi->data[0], static_cast<int32_T>(N));
  }
  vlen = Pi->size[1];
  if (Pi->size[1] == 0) {
    b_muj = 0.0;
  } else {
    b_muj = Pi->data[0];
    for (k = 0; k <= vlen - 2; k++) {
      b_muj += Pi->data[k + 1];
    }
  }
  kEnd = Pi->size[0] * Pi->size[1];
  Pi->size[0] = 1;
  emxEnsureCapacity_real_T(Pi, kEnd, &d_emlrtRTEI);
  kEnd = Pi->size[1];
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>((kEnd - 1) + 1LL), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_Pi, Pi);
    BaumWelch_kernel19<<<grid, block>>>(kEnd, gpu_muj, gpu_Pi);
    Pi_dirtyOnCpu = false;
    Pi_dirtyOnGpu = true;
  }
  //  Initialise Transition matrix
  kEnd = trans->size[0] * trans->size[1];
  trans->size[0] = static_cast<int32_T>(N);
  trans->size[1] = static_cast<int32_T>(N);
  emxEnsureCapacity_real_T(trans, kEnd, &c_emlrtRTEI);
  if (static_cast<int32_T>(N) != 0) {
    emlrtRandu(&trans->data[0],
               static_cast<int32_T>(N) * static_cast<int32_T>(N));
  }
  emxInit_real_T(&y, 2, &h_emlrtRTEI, true);
  emxInit_real_T(&b_x, 2, &e_emlrtRTEI, true);
  kEnd = b_x->size[0] * b_x->size[1];
  b_x->size[0] = trans->size[1];
  b_x->size[1] = trans->size[0];
  emxEnsureCapacity_real_T(b_x, kEnd, &e_emlrtRTEI);
  gpuEmxEnsureCapacity_real_T(b_x, &b_gpu_x);
  vlen = trans->size[0] - 1;
  m = trans->size[1] - 1;
  xpageoffset = b_x->size[0];
  k = trans->size[0];
  validLaunchParams =
      mwGetLaunchParameters1D(static_cast<real_T>((m + 1LL) * (vlen + 1LL)),
                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_trans, trans);
    trans_dirtyOnCpu = false;
    BaumWelch_kernel20<<<grid, block>>>(gpu_trans, m, vlen, xpageoffset, k,
                                        b_gpu_x);
  }
  vlen = b_x->size[0];
  if ((b_x->size[0] == 0) || (b_x->size[1] == 0)) {
    uint32_T sz[2];
    for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
      sz[xpageoffset] = static_cast<uint32_T>(b_x->size[xpageoffset]);
    }
    kEnd = y->size[0] * y->size[1];
    y->size[0] = 1;
    y->size[1] = static_cast<int32_T>(sz[1]);
    emxEnsureCapacity_real_T(y, kEnd, &g_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(y, &b_gpu_y);
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((static_cast<int32_T>(sz[1]) - 1) + 1LL), &grid,
        &block, 1024U, 65535U);
    if (validLaunchParams) {
      BaumWelch_kernel22<<<grid, block>>>(static_cast<int32_T>(sz[1]) - 1,
                                          b_gpu_y);
    }
  } else {
    LDA = b_x->size[1] - 1;
    kEnd = y->size[0] * y->size[1];
    y->size[0] = 1;
    y->size[1] = b_x->size[1];
    emxEnsureCapacity_real_T(y, kEnd, &f_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(y, &b_gpu_y);
    xpageoffset = b_x->size[0];
    validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(LDA + 1LL),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      hipMemcpy(gpu_LDA, &LDA, 4ULL, hipMemcpyHostToDevice);
      BaumWelch_kernel21<<<grid, block>>>(vlen, b_gpu_x, gpu_LDA, xpageoffset,
                                          b_gpu_y);
    }
  }
  emxFree_real_T(&b_x);
  emxInit_real_T(&b_y, 1, &h_emlrtRTEI, true);
  kEnd = b_y->size[0];
  b_y->size[0] = y->size[1];
  emxEnsureCapacity_real_T(b_y, kEnd, &h_emlrtRTEI);
  gpuEmxEnsureCapacity_real_T(b_y, &c_gpu_y);
  xpageoffset = y->size[1] - 1;
  validLaunchParams = mwGetLaunchParameters1D(
      static_cast<real_T>(xpageoffset + 1LL), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    BaumWelch_kernel23<<<grid, block>>>(b_gpu_y, xpageoffset, c_gpu_y);
    y_dirtyOnGpu = true;
  }
  emxFree_real_T(&y);
  if (trans->size[0] == b_y->size[0]) {
    emxInit_real_T(&b_trans, 2, &i_emlrtRTEI, true);
    kEnd = b_trans->size[0] * b_trans->size[1];
    b_trans->size[0] = trans->size[0];
    b_trans->size[1] = trans->size[1];
    emxEnsureCapacity_real_T(b_trans, kEnd, &i_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(b_trans, &b_gpu_trans);
    vlen = trans->size[1] - 1;
    m = trans->size[0] - 1;
    k = b_trans->size[0];
    xpageoffset = trans->size[0];
    validLaunchParams =
        mwGetLaunchParameters1D(static_cast<real_T>((m + 1LL) * (vlen + 1LL)),
                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (trans_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_trans, trans);
      }
      trans_dirtyOnCpu = false;
      BaumWelch_kernel24<<<grid, block>>>(c_gpu_y, gpu_trans, m, vlen, k,
                                          xpageoffset, b_gpu_trans);
    }
    kEnd = trans->size[0] * trans->size[1];
    trans->size[0] = b_trans->size[0];
    trans->size[1] = b_trans->size[1];
    emxEnsureCapacity_real_T(trans, kEnd, &j_emlrtRTEI);
    if (!trans_dirtyOnCpu) {
      gpuEmxEnsureCapacity_real_T(trans, &gpu_trans);
    }
    vlen = b_trans->size[0] * b_trans->size[1] - 1;
    validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(vlen + 1LL),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (trans_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_trans, trans);
      }
      BaumWelch_kernel25<<<grid, block>>>(b_gpu_trans, vlen, gpu_trans);
      trans_dirtyOnCpu = false;
      trans_dirtyOnGpu = true;
    }
    emxFree_real_T(&b_trans);
  } else {
    if (y_dirtyOnGpu) {
      gpuEmxMemcpyGpuToCpu_real_T(b_y, &c_gpu_y);
    }
    binary_expand_op(trans, b_y);
    trans_dirtyOnCpu = true;
  }
  emxFree_real_T(&b_y);
  i8 = static_cast<int32_T>(N);
  i9 = static_cast<int32_T>(N);
  i10 = static_cast<int32_T>(N);
  i13 = static_cast<int32_T>(N);
  i14 = static_cast<int32_T>(N);
  emxInit_real_T(&B, 2, &x_emlrtRTEI, true);
  emxInit_int32_T(&iv1, 1, &y_emlrtRTEI, true);
  emxInit_real_T(&alpha, 2, &m_emlrtRTEI, true);
  emxInit_real_T(&beta, 2, &o_emlrtRTEI, true);
  emxInit_real_T(&c_x, 2, &p_emlrtRTEI, true);
  emxInit_real_T(&Xi, 3, &r_emlrtRTEI, true);
  emxInit_real_T(&d_x, 2, &s_emlrtRTEI, true);
  emxInit_real_T(&Gamma, 2, &t_emlrtRTEI, true);
  emxInit_real_T(&a, 3, &w_emlrtRTEI, true);
  emxInit_real_T(&c_y, 2, &ab_emlrtRTEI, true);
  emxInit_real_T(&b_alpha, 2, &n_emlrtRTEI, true);
  emxInit_real_T(&c_alpha, 2, &q_emlrtRTEI, true);
  emxInit_real_T(&b_Xi, 3, &u_emlrtRTEI, true);
  for (int32_T cycle{0}; cycle < 100; cycle++) {
    //  Forward and backward algorithm to calculate the values of alpha and
    //  betha to determine the initial state probabilities
    kEnd = B->size[0] * B->size[1];
    B->size[0] = 1000;
    B->size[1] = static_cast<int32_T>(N);
    emxEnsureCapacity_real_T(B, kEnd, &k_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(B, &gpu_B);
    //  Find the probabilty of each observation being from each state
    for (i = 0; i < i8; i++) {
      if (Cov_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Cov, Cov);
      }
      if (C_dirtyOnCpu) {
        hipMemcpy(*gpu_C, C, 26912ULL, hipMemcpyHostToDevice);
      }
      BaumWelch_kernel26<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_Cov, i,
                                                                   *gpu_C);
      C_dirtyOnCpu = false;
      // MVNPDF Multivariate normal probability density function (pdf).
      //    Y = MVNPDF(X) returns the probability density of the multivariate
      //    normal distribution with zero mean and identity covariance matrix,
      //    evaluated at each row of X.  Rows of the N-by-D matrix X correspond
      //    to observations or points, and columns correspond to variables or
      //    coordinates.  Y is an N-by-1 vector.
      //
      //    Y = MVNPDF(X,MU) returns the density of the multivariate normal
      //    distribution with mean MU and identity covariance matrix, evaluated
      //    at each row of X.  MU is a 1-by-D vector, or an N-by-D matrix, in
      //    which case the density is evaluated for each row of X with the
      //    corresponding row of MU.  MU can also be a scalar value, which
      //    MVNPDF replicates to match the size of X.
      //
      //    Y = MVNPDF(X,MU,SIGMA) returns the density of the multivariate
      //    normal distribution with mean MU and covariance SIGMA, evaluated at
      //    each row of X.  SIGMA is a D-by-D matrix, or an D-by-D-by-N array,
      //    in which case the density is evaluated for each row of X with the
      //    corresponding page of SIGMA, i.e., MVNPDF computes Y(I) using X(I,:)
      //    and SIGMA(:,:,I). If the covariance matrix is diagonal, containing
      //    variances along the diagonal and zero covariances off the diagonal,
      //    SIGMA may also be specified as a 1-by-D matrix or a 1-by-D-by-N
      //    array, containing just the diagonal. Pass in the empty matrix for MU
      //    to use its default value when you want to only specify SIGMA.
      //
      //    If X is a 1-by-D vector, MVNPDF replicates it to match the leading
      //    dimension of MU or the trailing dimension of SIGMA.
      //
      //    Example:
      //
      //       mu = [1 -1]; Sigma = [.9 .4; .4 .3];
      //       [X1,X2] = meshgrid(linspace(-1,3,25)', linspace(-3,1,25)');
      //       X = [X1(:) X2(:)];
      //       p = mvnpdf(X, mu, Sigma);
      //       surf(X1,X2,reshape(p,25,25));
      //
      //    See also MVTPDF, MVNCDF, MVNRND, NORMPDF.
      //    Copyright 1993-2008 The MathWorks, Inc.
      //    $Revision: 1.2.4.8 $  $Date: 2008/02/29 13:12:08 $
      //  Get size of data.  Column vectors provisionally interpreted as
      //  multiple scalar data. Assume zero mean, data are already centered mean
      //  is a single row, rep it out to match data
      xpageoffset = Mu->size[0];
      if (Mu_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Mu, Mu);
      }
      Mu_dirtyOnCpu = false;
      BaumWelch_kernel27<<<dim3(114U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          gpu_Mu, i, *gpu_X, xpageoffset, *gpu_X0);
      //  Assume identity covariance, data are already standardized
      //  Special case: if Sigma is supplied, then use it to try to interpret
      //  X and Mu as row vectors if they were both column vectors.
      // Check that sigma is the right size
      //  Make sure Sigma is a valid covariance matrix
      // CHOLCOV  Cholesky-like decomposition for covariance matrix.
      //    T = CHOLCOV(SIGMA) computes T such that SIGMA = T'*T.  SIGMA must be
      //    square, symmetric, and positive semi-definite.  If SIGMA is positive
      //    definite, then T is the square, upper triangular Cholesky factor.
      //
      //    If SIGMA is not positive definite, T is computed from an eigenvalue
      //    decomposition of SIGMA.  T is not necessarily triangular or square
      //    in this case.  Any eigenvectors whose corresponding eigenvalue is
      //    close to zero (within a small tolerance) are omitted.  If any
      //    remaining eigenvalues are negative, T is empty.
      //
      //    [T,P] = CHOLCOV(SIGMA) returns the number of negative eigenvalues of
      //    SIGMA, and T is empty if P>0.  If P==0, SIGMA is positive
      //    semi-definite.
      //
      //    If SIGMA is not square and symmetric, P is NaN and T is empty.
      //
      //    [T,P] = CHOLCOV(SIGMA,0) returns P==0 if SIGMA is positive definite,
      //    and T is the Cholesky factor.  If SIGMA is not positive definite, P
      //    is a positive integer and T is empty.  [...] = CHOLCOV(SIGMA,1) is
      //    equivalent to [...] = CHOLCOV(SIGMA).
      //
      //    Example:
      //    Factor a rank-deficient covariance matrix C.
      //        C = [2 1 1 2;1 2 1 2;1 1 2 2;2 2 2 3]
      //        T = cholcov(C)
      //        C2 = T'*T
      //    Generate data with this covariance (aside from random variation).
      //        C3 = cov(randn(10000,3)*T)
      //
      //    See also CHOL.
      //    Copyright 1993-2009 The MathWorks, Inc.
      //  Test for square, symmetric
      BaumWelch_kernel28<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(gpu_Cov, i,
                                                                  *gpu_x);
      hipMemcpy(x, *gpu_x, 464ULL, hipMemcpyDeviceToHost);
      b_muj = x[0];
      for (b_vlen = 0; b_vlen < 57; b_vlen++) {
        beta1 = x[b_vlen + 1];
        if (std::isnan(beta1)) {
          p = false;
        } else if (std::isnan(b_muj)) {
          p = true;
        } else {
          p = (b_muj < beta1);
        }
        if (p) {
          b_muj = beta1;
        }
      }
      if ((!std::isinf(b_muj)) && (!std::isnan(b_muj))) {
        if (b_muj <= 2.2250738585072014E-308) {
          b_muj = 4.94065645841247E-324;
        } else {
          frexp(b_muj, &exponent);
          b_muj = std::ldexp(1.0, exponent - 53);
        }
      } else {
        b_muj = rtNaN;
      }
      b_muj *= 10.0;
      Cov_dirtyOnCpu = false;
      BaumWelch_kernel29<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_Cov, i,
                                                                   *e_gpu_x);
      BaumWelch_kernel30<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*e_gpu_x,
                                                                   *d_gpu_y);
      BaumWelch_kernel31<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*e_gpu_y);
      hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
      BaumWelch_kernel32<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
          1, gpu_muj, *d_gpu_y, *e_gpu_y);
      y_dirtyOnGpu = true;
      p = true;
      k = 0;
      exitg1 = false;
      while ((!exitg1) && (k < 58)) {
        if (y_dirtyOnGpu) {
          hipMemcpy(d_y, *e_gpu_y, 58ULL, hipMemcpyDeviceToHost);
        }
        y_dirtyOnGpu = false;
        if (!d_y[k]) {
          p = false;
          exitg1 = true;
        } else {
          k++;
        }
      }
      if (p) {
        hipsolverDnDpotrf_bufferSize(
            getCuSolverGlobalHandle(), HIPBLAS_FILL_MODE_UPPER, 58,
            (double *)&(*gpu_C)[0], 58, getCuSolverWorkspaceReq());
        setCuSolverWorkspaceTypeSize(8);
        cusolverInitWorkspace();
        if (info_t_dirtyOnCpu) {
          hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
        }
        hipsolverDnDpotrf(getCuSolverGlobalHandle(), HIPBLAS_FILL_MODE_UPPER, 58,
                         (double *)&(*gpu_C)[0], 58,
                         static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                         *getCuSolverWorkspaceReq(), gpu_info_t);
        info_t_dirtyOnCpu = false;
        iwork_dirtyOnCpu = true;
        hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
        if (info_t < 0) {
          BaumWelch_kernel33<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_C);
        }
        if (info_t == 0) {
          m = 56;
        } else {
          m = info_t - 3;
        }
        for (j = 0; j <= m; j++) {
          for (b_vlen = 0; b_vlen <= m - j; b_vlen++) {
            if (iwork_dirtyOnCpu) {
              hipMemcpy(C, *gpu_C, 26912ULL, hipMemcpyDeviceToHost);
            }
            C[((j + b_vlen) + 58 * j) + 1] = 0.0;
            iwork_dirtyOnCpu = false;
            C_dirtyOnCpu = true;
          }
        }
        if (m + 2 < 1) {
          LDA = -1;
          m = -1;
        } else {
          LDA = m + 1;
          m++;
        }
        T_size[0] = LDA + 1;
        T_size[1] = m + 1;
        validLaunchParams = mwGetLaunchParameters1D(
            static_cast<real_T>((LDA + 1LL) * (m + 1LL)), &grid, &block, 1024U,
            65535U);
        if (validLaunchParams) {
          hipMemcpy(gpu_LDA, &LDA, 4ULL, hipMemcpyHostToDevice);
          hipMemcpy(gpu_m, &m, 4ULL, hipMemcpyHostToDevice);
          if (C_dirtyOnCpu) {
            hipMemcpy(*gpu_C, C, 26912ULL, hipMemcpyHostToDevice);
          }
          BaumWelch_kernel34<<<grid, block>>>(LDA + 1, gpu_LDA, gpu_m, *gpu_C,
                                              *gpu_T_data);
          C_dirtyOnCpu = false;
        }
        b_muj = info_t;
        idx1_dirtyOnGpu = true;
        if (info_t > 0) {
          //  Test for positive definiteness
          T_size[0] = 0;
          T_size[1] = 0;
        }
      } else {
        T_size[0] = 0;
        T_size[1] = 0;
        b_muj = rtNaN;
        idx1_dirtyOnGpu = true;
      }
      if (b_muj != 0.0) {
        e_y = nullptr;
        b_m = emlrtCreateCharArray(2, &iv2[0]);
        emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 26, b_m, &varargin_1[0]);
        emlrtAssign(&e_y, b_m);
        f_y = nullptr;
        b_m = emlrtCreateCharArray(2, &iv3[0]);
        emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 46, b_m, &varargin_2[0]);
        emlrtAssign(&f_y, b_m);
        b_error(e_y, f_y, &emlrtMCI);
      }
      //  Create array of standardized data, and compute log(sqrt(det(Sigma)))
      xpageoffset = T_size[0] * T_size[1] - 1;
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>(xpageoffset + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        BaumWelch_kernel35<<<grid, block>>>(*gpu_T_data, xpageoffset,
                                            *gpu_dv_data);
        dv_data_dirtyOnGpu = true;
      }
      if (T_size[0] == 0) {
        b_vlen = 0;
      } else if (T_size[0] == 58) {
        hipsolverDnDgetrf_bufferSize(getCuSolverGlobalHandle(), 58, 58,
                                    (double *)&(*gpu_dv_data)[0], 58,
                                    getCuSolverWorkspaceReq());
        setCuSolverWorkspaceTypeSize(8);
        cusolverInitWorkspace();
        if (info_t_dirtyOnCpu) {
          hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
        }
        hipsolverDnDgetrf(getCuSolverGlobalHandle(), 58, 58,
                         (double *)&(*gpu_dv_data)[0], 58,
                         static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                         &(*gpu_ipiv_t)[0], gpu_info_t);
        dv_data_dirtyOnGpu = true;
        info_t_dirtyOnCpu = false;
        hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
        if (info_t < 0) {
          xpageoffset = 58 * T_size[1] - 1;
          validLaunchParams =
              mwGetLaunchParameters1D(static_cast<real_T>(xpageoffset + 1LL),
                                      &grid, &block, 1024U, 65535U);
          if (validLaunchParams) {
            BaumWelch_kernel49<<<grid, block>>>(xpageoffset, *gpu_dv_data);
          }
          BaumWelch_kernel50<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
              *gpu_ipiv_data);
          iwork_dirtyOnCpu = true;
        } else {
          BaumWelch_kernel48<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
              *gpu_ipiv_t, *gpu_ipiv_data);
          iwork_dirtyOnCpu = true;
        }
        b_muj = 1.0;
        hipblasDtrsm(getCublasGlobalHandle(), HIPBLAS_SIDE_RIGHT,
                    HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
                    1000, 58, (double *)&b_muj, (double *)&(*gpu_dv_data)[0],
                    58, (double *)&(*gpu_X0)[0], 1000);
        b_muj = 1.0;
        hipblasDtrsm(getCublasGlobalHandle(), HIPBLAS_SIDE_RIGHT,
                    HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, 1000,
                    58, (double *)&b_muj, (double *)&(*gpu_dv_data)[0], 58,
                    (double *)&(*gpu_X0)[0], 1000);
        p = false;
        idx_dirtyOnGpu = true;
        for (j = 0; j < 57; j++) {
          if (iwork_dirtyOnCpu) {
            hipMemcpy(ipiv_data, *gpu_ipiv_data, 232ULL,
                       hipMemcpyDeviceToHost);
          }
          iwork_dirtyOnCpu = false;
          kEnd = ipiv_data[56 - j];
          if (kEnd != 57 - j) {
            for (b_vlen = 0; b_vlen < 1000; b_vlen++) {
              if (idx_dirtyOnGpu) {
                hipMemcpy(SD->f0.X0, *gpu_X0, 464000ULL,
                           hipMemcpyDeviceToHost);
              }
              b_muj = SD->f0.X0[b_vlen + 1000 * (56 - j)];
              SD->f0.X0[b_vlen + 1000 * (56 - j)] =
                  SD->f0.X0[b_vlen + 1000 * (kEnd - 1)];
              SD->f0.X0[b_vlen + 1000 * (kEnd - 1)] = b_muj;
              idx_dirtyOnGpu = false;
              p = true;
            }
          }
        }
        b_vlen = 58;
        if (p) {
          hipMemcpy(*gpu_X0, SD->f0.X0, 464000ULL, hipMemcpyHostToDevice);
        }
        BaumWelch_kernel51<<<dim3(114U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
            *gpu_X0, *gpu_xRinv_data);
      } else {
        ptrdiff_t b_info_t;
        m = T_size[0];
        idx_dirtyOnGpu = true;
        validLaunchParams = mwGetLaunchParameters1D(
            static_cast<real_T>(((T_size[1] - 1) + 1LL) *
                                ((T_size[0] - 1) + 1LL)),
            &grid, &block, 1024U, 65535U);
        if (validLaunchParams) {
          BaumWelch_kernel36<<<grid, block>>>(*gpu_T_data, T_size[0],
                                              T_size[1] - 1, T_size[0] - 1,
                                              *gpu_dv_data);
          dv_data_dirtyOnGpu = true;
        }
        BaumWelch_kernel37<<<dim3(114U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
            *gpu_X0, *b_gpu_B);
        validLaunchParams =
            mwGetLaunchParameters1D(static_cast<real_T>((T_size[0] - 1) + 1LL),
                                    &grid, &block, 1024U, 65535U);
        if (validLaunchParams) {
          hipMemcpy(gpu_m, &T_size[0], 4ULL, hipMemcpyHostToDevice);
          idx_dirtyOnGpu = false;
          BaumWelch_kernel38<<<grid, block>>>(gpu_m, *gpu_ipiv_t);
        }
        validLaunchParams =
            mwGetLaunchParameters1D(static_cast<real_T>((T_size[0] - 1) + 1LL),
                                    &grid, &block, 1024U, 65535U);
        if (validLaunchParams) {
          if (idx_dirtyOnGpu) {
            hipMemcpy(gpu_m, &T_size[0], 4ULL, hipMemcpyHostToDevice);
          }
          idx_dirtyOnGpu = false;
          if (jpvt_t_data_dirtyOnCpu) {
            hipMemcpy(*gpu_jpvt_t_data, jpvt_t_data, 464ULL,
                       hipMemcpyHostToDevice);
          }
          BaumWelch_kernel39<<<grid, block>>>(gpu_m, *gpu_jpvt_t_data);
          jpvt_t_data_dirtyOnGpu = true;
        }
        if (dv_data_dirtyOnGpu) {
          hipMemcpy(dv_data, *gpu_dv_data, 26912ULL, hipMemcpyDeviceToHost);
        }
        if (jpvt_t_data_dirtyOnGpu) {
          hipMemcpy(jpvt_t_data, *gpu_jpvt_t_data, 464ULL,
                     hipMemcpyDeviceToHost);
        }
        b_info_t =
            LAPACKE_dgeqp3(102, (ptrdiff_t)58, (ptrdiff_t)T_size[0],
                           &dv_data[0], (ptrdiff_t)58, &jpvt_t_data[0], &x[0]);
        iwork_dirtyOnCpu = true;
        dv_data_dirtyOnGpu = false;
        p = true;
        jpvt_t_data_dirtyOnGpu = false;
        jpvt_t_data_dirtyOnCpu = true;
        if ((int32_T)b_info_t != 0) {
          validLaunchParams = mwGetLaunchParameters1D(
              static_cast<real_T>(58LL * ((T_size[0] - 1) + 1LL)), &grid,
              &block, 1024U, 65535U);
          if (validLaunchParams) {
            if (idx_dirtyOnGpu) {
              hipMemcpy(gpu_m, &T_size[0], 4ULL, hipMemcpyHostToDevice);
            }
            idx_dirtyOnGpu = false;
            hipMemcpy(*gpu_dv_data, dv_data, 26912ULL, hipMemcpyHostToDevice);
            BaumWelch_kernel41<<<grid, block>>>(gpu_m, *gpu_dv_data);
            p = false;
            dv_data_dirtyOnGpu = true;
          }
          validLaunchParams = mwGetLaunchParameters1D(
              static_cast<real_T>((T_size[0] - 1) + 1LL), &grid, &block, 1024U,
              65535U);
          if (validLaunchParams) {
            if (idx_dirtyOnGpu) {
              hipMemcpy(gpu_m, &T_size[0], 4ULL, hipMemcpyHostToDevice);
            }
            idx_dirtyOnGpu = false;
            hipMemcpy(*gpu_x, x, 464ULL, hipMemcpyHostToDevice);
            BaumWelch_kernel42<<<grid, block>>>(gpu_m, *gpu_x);
            iwork_dirtyOnCpu = false;
          }
          validLaunchParams = mwGetLaunchParameters1D(
              static_cast<real_T>((T_size[0] - 1) + 1LL), &grid, &block, 1024U,
              65535U);
          if (validLaunchParams) {
            if (idx_dirtyOnGpu) {
              hipMemcpy(gpu_m, &T_size[0], 4ULL, hipMemcpyHostToDevice);
            }
            BaumWelch_kernel43<<<grid, block>>>(gpu_m, *gpu_ipiv_t);
          }
        } else {
          validLaunchParams = mwGetLaunchParameters1D(
              static_cast<real_T>((T_size[0] - 1) + 1LL), &grid, &block, 1024U,
              65535U);
          if (validLaunchParams) {
            hipMemcpy(*gpu_jpvt_t_data, jpvt_t_data, 464ULL,
                       hipMemcpyHostToDevice);
            jpvt_t_data_dirtyOnCpu = false;
            if (idx_dirtyOnGpu) {
              hipMemcpy(gpu_m, &T_size[0], 4ULL, hipMemcpyHostToDevice);
            }
            BaumWelch_kernel40<<<grid, block>>>(*gpu_jpvt_t_data, gpu_m,
                                                *gpu_ipiv_t);
          }
        }
        LDA = 0;
        if (dv_data_dirtyOnGpu) {
          hipMemcpy(dv_data, *gpu_dv_data, 26912ULL, hipMemcpyDeviceToHost);
        }
        b_muj = 1.2878587085651816E-13 * std::abs(dv_data[0]);
        while ((LDA < m) && (!(std::abs(dv_data[LDA + 58 * LDA]) <= b_muj))) {
          LDA++;
        }
        validLaunchParams = mwGetLaunchParameters1D(
            static_cast<real_T>((static_cast<int16_T>(T_size[0]) * 1000 - 1) +
                                1LL),
            &grid, &block, 1024U, 65535U);
        if (validLaunchParams) {
          BaumWelch_kernel44<<<grid, block>>>(static_cast<int16_T>(T_size[0]),
                                              *gpu_Y_data);
        }
        info_t = 0;
        if (p) {
          hipMemcpy(*gpu_dv_data, dv_data, 26912ULL, hipMemcpyHostToDevice);
        }
        if (iwork_dirtyOnCpu) {
          hipMemcpy(*gpu_x, x, 464ULL, hipMemcpyHostToDevice);
        }
        hipsolverDnDormqr_bufferSize(
            getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, 58, 1000,
            T_size[0], (double *)&(*gpu_dv_data)[0], 58, (double *)&(*gpu_x)[0],
            (double *)&(*b_gpu_B)[0], 58, getCuSolverWorkspaceReq());
        setCuSolverWorkspaceTypeSize(8);
        cusolverInitWorkspace();
        hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
        hipsolverDnDormqr(getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT,
                         HIPBLAS_OP_T, 58, 1000, T_size[0],
                         (double *)&(*gpu_dv_data)[0], 58,
                         (double *)&(*gpu_x)[0], (double *)&(*b_gpu_B)[0], 58,
                         static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                         *getCuSolverWorkspaceReq(), gpu_info_t);
        info_t_dirtyOnCpu = false;
        hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
        if (info_t < 0) {
          BaumWelch_kernel45<<<dim3(114U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
              *b_gpu_B);
        }
        hipMemcpy(gpu_LDA, &LDA, 4ULL, hipMemcpyHostToDevice);
        BaumWelch_kernel46<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
            T_size[0], gpu_LDA, *gpu_dv_data, *b_gpu_B, *gpu_ipiv_t,
            *gpu_Y_data);
        dv_data_dirtyOnGpu = true;
        b_vlen = T_size[0];
        validLaunchParams = mwGetLaunchParameters1D(
            static_cast<real_T>(1000LL * ((T_size[0] - 1) + 1LL)), &grid,
            &block, 1024U, 65535U);
        if (validLaunchParams) {
          BaumWelch_kernel47<<<grid, block>>>(*gpu_Y_data, T_size[0],
                                              T_size[0] - 1, *gpu_xRinv_data);
        }
      }
      validLaunchParams =
          mwGetLaunchParameters1D(static_cast<real_T>((T_size[0] - 1) + 1LL),
                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        hipMemcpy(gpu_m, &T_size[0], 4ULL, hipMemcpyHostToDevice);
        BaumWelch_kernel52<<<grid, block>>>(*gpu_T_data, T_size[0], gpu_m,
                                            *gpu_x);
      }
      m = T_size[0] - 1;
      validLaunchParams =
          mwGetLaunchParameters1D(static_cast<real_T>((T_size[0] - 1) + 1LL),
                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        hipMemcpy(gpu_m, &m, 4ULL, hipMemcpyHostToDevice);
        BaumWelch_kernel53<<<grid, block>>>(gpu_m, *gpu_x);
      }
      if (T_size[0] == 0) {
        b_muj = 0.0;
      } else {
        hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
        BaumWelch_kernel54<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_x,
                                                                    gpu_muj);
        idx1_dirtyOnGpu = false;
        validLaunchParams =
            mwGetLaunchParameters(static_cast<real_T>((T_size[0] - 2) + 1LL),
                                  &grid, &block, 1024U, 65535U);
        if (validLaunchParams) {
          BaumWelch_kernel55<<<grid, block>>>(T_size[0], *gpu_x, gpu_muj);
        }
      }
      //  The quadratic form is the inner products of the standardized data
      m = 1000 * b_vlen - 1;
      validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(m + 1LL),
                                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        hipMemcpy(gpu_m, &m, 4ULL, hipMemcpyHostToDevice);
        BaumWelch_kernel56<<<grid, block>>>(*gpu_xRinv_data, gpu_m,
                                            *gpu_y_data);
      }
      if (b_vlen == 0) {
        BaumWelch_kernel59<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
            *gpu_quadform);
      } else {
        BaumWelch_kernel57<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
            *gpu_y_data, *gpu_quadform);
        for (k = 0; k <= b_vlen - 2; k++) {
          LDA = (k + 1) * 1000;
          hipMemcpy(gpu_LDA, &LDA, 4ULL, hipMemcpyHostToDevice);
          BaumWelch_kernel58<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
              *gpu_y_data, gpu_LDA, *gpu_quadform);
        }
      }
      if (idx1_dirtyOnGpu) {
        hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
      }
      BaumWelch_kernel60<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          *gpu_quadform, i, gpu_muj, gpu_B);
      B_dirtyOnGpu = true;
    }
    LDA = 1000 * B->size[1] - 1;
    m = 0;
    for (i = 0; i <= LDA; i++) {
      if (B_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(B, &gpu_B);
      }
      B_dirtyOnGpu = false;
      if (B->data[i] == 0.0) {
        m++;
      }
    }
    kEnd = iv1->size[0];
    iv1->size[0] = m;
    emxEnsureCapacity_int32_T(iv1, kEnd, &l_emlrtRTEI);
    if (!iv1_dirtyOnCpu) {
      gpuEmxEnsureCapacity_int32_T(iv1, &gpu_iv1);
    }
    m = 0;
    for (i = 0; i <= LDA; i++) {
      if (B_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(B, &gpu_B);
      }
      B_dirtyOnGpu = false;
      if (B->data[i] == 0.0) {
        iv1->data[m] = i + 1;
        iv1_dirtyOnCpu = true;
        m++;
      }
    }
    kEnd = iv1->size[0];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((kEnd - 1) + 1LL), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (iv1_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_iv1, iv1);
      }
      iv1_dirtyOnCpu = false;
      BaumWelch_kernel61<<<grid, block>>>(gpu_iv1, kEnd, gpu_B);
      B_dirtyOnGpu = true;
    }
    //  Initial alpha step
    kEnd = alpha->size[0] * alpha->size[1];
    alpha->size[0] = 1000;
    alpha->size[1] = static_cast<int32_T>(N);
    emxEnsureCapacity_real_T(alpha, kEnd, &m_emlrtRTEI);
    if (!alpha_dirtyOnCpu) {
      gpuEmxEnsureCapacity_real_T(alpha, &gpu_alpha);
    }
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((1000 * static_cast<int32_T>(N) - 1) + 1LL), &grid,
        &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (alpha_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
      }
      BaumWelch_kernel62<<<grid, block>>>(N, gpu_alpha);
      alpha_dirtyOnCpu = false;
      alpha_dirtyOnGpu = true;
    }
    BaumWelch_kernel63<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_scale);
    if (Pi->size[1] == B->size[1]) {
      xpageoffset = Pi->size[1] - 1;
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>(xpageoffset + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        if (Pi_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_Pi, Pi);
        }
        Pi_dirtyOnCpu = false;
        if (alpha_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
        }
        BaumWelch_kernel64<<<grid, block>>>(gpu_B, gpu_Pi, xpageoffset,
                                            gpu_alpha);
        alpha_dirtyOnCpu = false;
        alpha_dirtyOnGpu = true;
      }
    } else {
      if (alpha_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(alpha, &gpu_alpha);
      }
      if (Pi_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(Pi, &gpu_Pi);
      }
      Pi_dirtyOnGpu = false;
      if (B_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(B, &gpu_B);
      }
      B_dirtyOnGpu = false;
      binary_expand_op(alpha, Pi, B);
      alpha_dirtyOnGpu = false;
      alpha_dirtyOnCpu = true;
    }
    vlen = alpha->size[1];
    if (alpha->size[1] == 0) {
      b_muj = 0.0;
    } else {
      if (alpha_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(alpha, &gpu_alpha);
      }
      alpha_dirtyOnGpu = false;
      b_muj = alpha->data[0];
      for (k = 0; k <= vlen - 2; k++) {
        b_muj += alpha->data[1000 * (k + 1)];
      }
    }
    hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
    BaumWelch_kernel65<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_muj,
                                                                *gpu_scale);
    kEnd = alpha->size[1] - 1;
    b_vlen = b_alpha->size[0] * b_alpha->size[1];
    b_alpha->size[0] = 1;
    b_alpha->size[1] = alpha->size[1];
    emxEnsureCapacity_real_T(b_alpha, b_vlen, &n_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(b_alpha, &b_gpu_alpha);
    validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(kEnd + 1LL),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (alpha_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
      }
      alpha_dirtyOnCpu = false;
      BaumWelch_kernel66<<<grid, block>>>(gpu_alpha, kEnd, gpu_muj,
                                          b_gpu_alpha);
    }
    xpageoffset = b_alpha->size[1] - 1;
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>(xpageoffset + 1LL), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (alpha_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
      }
      BaumWelch_kernel67<<<grid, block>>>(b_gpu_alpha, xpageoffset, gpu_alpha);
      alpha_dirtyOnCpu = false;
      alpha_dirtyOnGpu = true;
    }
    //  Alpha recursion
    for (info_t = 0; info_t < 999; info_t++) {
      iwork_dirtyOnCpu = false;
      info_t_dirtyOnCpu = true;
      for (i = 0; i < i9; i++) {
        if (trans->size[0] == alpha->size[1]) {
          kEnd = alpha->size[1] - 1;
          b_vlen = c_x->size[0] * c_x->size[1];
          c_x->size[0] = 1;
          c_x->size[1] = alpha->size[1];
          emxEnsureCapacity_real_T(c_x, b_vlen, &p_emlrtRTEI);
          if (!x_dirtyOnCpu) {
            gpuEmxEnsureCapacity_real_T(c_x, &c_gpu_x);
          }
          k = trans->size[0];
          validLaunchParams = mwGetLaunchParameters1D(
              static_cast<real_T>(kEnd + 1LL), &grid, &block, 1024U, 65535U);
          if (validLaunchParams) {
            if (trans_dirtyOnCpu) {
              gpuEmxMemcpyCpuToGpu_real_T(&gpu_trans, trans);
            }
            trans_dirtyOnCpu = false;
            if (alpha_dirtyOnCpu) {
              gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
            }
            alpha_dirtyOnCpu = false;
            if (info_t_dirtyOnCpu) {
              hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
            }
            if (x_dirtyOnCpu) {
              gpuEmxMemcpyCpuToGpu_real_T(&c_gpu_x, c_x);
            }
            BaumWelch_kernel68<<<grid, block>>>(i, gpu_trans, gpu_alpha, kEnd,
                                                k, gpu_info_t, c_gpu_x);
            info_t_dirtyOnCpu = false;
            x_dirtyOnCpu = false;
            x_dirtyOnGpu = true;
          }
        } else {
          if (x_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(c_x, &c_gpu_x);
          }
          if (alpha_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(alpha, &gpu_alpha);
          }
          if (iwork_dirtyOnCpu) {
            hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
          }
          if (trans_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(trans, &gpu_trans);
          }
          trans_dirtyOnGpu = false;
          binary_expand_op(c_x, alpha, info_t, trans, i);
          x_dirtyOnGpu = false;
          x_dirtyOnCpu = true;
        }
        vlen = c_x->size[1];
        if (c_x->size[1] == 0) {
          b_muj = 0.0;
        } else {
          if (x_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(c_x, &c_gpu_x);
          }
          x_dirtyOnGpu = false;
          b_muj = c_x->data[0];
          for (k = 0; k <= vlen - 2; k++) {
            b_muj += c_x->data[k + 1];
          }
        }
        hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
        if (info_t_dirtyOnCpu) {
          hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
        }
        if (alpha_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
        }
        BaumWelch_kernel69<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            gpu_B, i, gpu_muj, gpu_info_t, gpu_alpha);
        info_t_dirtyOnCpu = false;
        iwork_dirtyOnCpu = true;
        alpha_dirtyOnCpu = false;
        alpha_dirtyOnGpu = true;
      }
      vlen = alpha->size[1];
      if (alpha->size[1] == 0) {
        b_muj = 0.0;
      } else {
        if (alpha_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(alpha, &gpu_alpha);
        }
        alpha_dirtyOnGpu = false;
        if (iwork_dirtyOnCpu) {
          hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
        }
        b_muj = alpha->data[info_t + 1];
        for (k = 0; k <= vlen - 2; k++) {
          b_muj += alpha->data[(info_t + 1000 * (k + 1)) + 1];
        }
      }
      hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
      if (info_t_dirtyOnCpu) {
        hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
      }
      BaumWelch_kernel70<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          gpu_muj, gpu_info_t, *gpu_scale);
      info_t_dirtyOnCpu = false;
      hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
      m = info_t + 1;
      LDA = info_t + 1;
      kEnd = alpha->size[1] - 1;
      b_vlen = c_alpha->size[0] * c_alpha->size[1];
      c_alpha->size[0] = 1;
      c_alpha->size[1] = alpha->size[1];
      emxEnsureCapacity_real_T(c_alpha, b_vlen, &q_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(c_alpha, &c_gpu_alpha);
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>(kEnd + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        if (alpha_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
        }
        alpha_dirtyOnCpu = false;
        hipMemcpy(gpu_m, &m, 4ULL, hipMemcpyHostToDevice);
        BaumWelch_kernel71<<<grid, block>>>(gpu_alpha, gpu_m, kEnd, gpu_muj,
                                            c_gpu_alpha);
      }
      xpageoffset = c_alpha->size[1] - 1;
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>(xpageoffset + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        hipMemcpy(gpu_LDA, &LDA, 4ULL, hipMemcpyHostToDevice);
        if (alpha_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
        }
        BaumWelch_kernel72<<<grid, block>>>(c_gpu_alpha, gpu_LDA, xpageoffset,
                                            gpu_alpha);
        alpha_dirtyOnCpu = false;
        alpha_dirtyOnGpu = true;
      }
    }
    //  Beta recursion
    kEnd = beta->size[0] * beta->size[1];
    beta->size[0] = 1000;
    beta->size[1] = static_cast<int32_T>(N);
    emxEnsureCapacity_real_T(beta, kEnd, &o_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(beta, &gpu_beta);
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((1000 * static_cast<int32_T>(N) - 1) + 1LL), &grid,
        &block, 1024U, 65535U);
    if (validLaunchParams) {
      BaumWelch_kernel73<<<grid, block>>>(N, gpu_beta);
      beta_dirtyOnGpu = true;
    }
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((static_cast<int32_T>(N) - 1) + 1LL), &grid, &block,
        1024U, 65535U);
    if (validLaunchParams) {
      BaumWelch_kernel74<<<grid, block>>>(*gpu_scale, static_cast<int32_T>(N),
                                          gpu_beta);
      beta_dirtyOnGpu = true;
    }
    for (info_t = 0; info_t < 999; info_t++) {
      iwork_dirtyOnCpu = false;
      info_t_dirtyOnCpu = true;
      for (i = 0; i < i10; i++) {
        if (trans->size[1] == 1) {
          vlen = B->size[1];
        } else {
          vlen = trans->size[1];
        }
        if ((trans->size[1] == B->size[1]) && (vlen == beta->size[1])) {
          if (iwork_dirtyOnCpu) {
            hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
          }
          kEnd = 999 - info_t;
          LDA = trans->size[1] - 1;
          m = d_x->size[0] * d_x->size[1];
          d_x->size[0] = 1;
          d_x->size[1] = trans->size[1];
          emxEnsureCapacity_real_T(d_x, m, &s_emlrtRTEI);
          if (!b_x_dirtyOnCpu) {
            gpuEmxEnsureCapacity_real_T(d_x, &d_gpu_x);
          }
          k = trans->size[0];
          validLaunchParams = mwGetLaunchParameters1D(
              static_cast<real_T>(LDA + 1LL), &grid, &block, 1024U, 65535U);
          if (validLaunchParams) {
            if (trans_dirtyOnCpu) {
              gpuEmxMemcpyCpuToGpu_real_T(&gpu_trans, trans);
            }
            trans_dirtyOnCpu = false;
            hipMemcpy(gpu_LDA, &LDA, 4ULL, hipMemcpyHostToDevice);
            if (b_x_dirtyOnCpu) {
              gpuEmxMemcpyCpuToGpu_real_T(&d_gpu_x, d_x);
            }
            BaumWelch_kernel75<<<grid, block>>>(gpu_beta, 999 - info_t, gpu_B,
                                                kEnd, gpu_trans, i, gpu_LDA, k,
                                                d_gpu_x);
            b_x_dirtyOnCpu = false;
            b_x_dirtyOnGpu = true;
          }
        } else {
          if (b_x_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(d_x, &d_gpu_x);
          }
          if (trans_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(trans, &gpu_trans);
          }
          trans_dirtyOnGpu = false;
          if (B_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(B, &gpu_B);
          }
          B_dirtyOnGpu = false;
          if (iwork_dirtyOnCpu) {
            hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
          }
          if (beta_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(beta, &gpu_beta);
          }
          binary_expand_op(d_x, trans, i, B, info_t, beta);
          b_x_dirtyOnGpu = false;
          b_x_dirtyOnCpu = true;
        }
        vlen = d_x->size[1];
        if (d_x->size[1] == 0) {
          b_muj = 0.0;
        } else {
          if (b_x_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(d_x, &d_gpu_x);
          }
          b_x_dirtyOnGpu = false;
          b_muj = d_x->data[0];
          for (k = 0; k <= vlen - 2; k++) {
            b_muj += d_x->data[k + 1];
          }
        }
        hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
        if (info_t_dirtyOnCpu) {
          hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
        }
        BaumWelch_kernel76<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            *gpu_scale, i, gpu_muj, gpu_info_t, gpu_beta);
        beta_dirtyOnGpu = true;
        info_t_dirtyOnCpu = false;
        iwork_dirtyOnCpu = true;
      }
    }
    //  E (Expectation step) Estimate the state occupation probabilities
    //  which is necessary to obtain the training step of the HMM model
    kEnd = Xi->size[0] * Xi->size[1] * Xi->size[2];
    Xi->size[0] = 999;
    Xi->size[1] = static_cast<int32_T>(N);
    Xi->size[2] = static_cast<int32_T>(N);
    emxEnsureCapacity_real_T(Xi, kEnd, &r_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(Xi, &gpu_Xi);
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>(
            (999 * static_cast<int32_T>(N) * static_cast<int32_T>(N) - 1) +
            1LL),
        &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      BaumWelch_kernel77<<<grid, block>>>(N, gpu_Xi);
      Xi_dirtyOnGpu = true;
    }
    for (info_t = 0; info_t < 999; info_t++) {
      iwork_dirtyOnCpu = false;
      info_t_dirtyOnCpu = true;
      LDA = Xi->size[1];
      k = trans->size[0];
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>(((static_cast<int32_T>(N) - 1) + 1LL) *
                              ((static_cast<int32_T>(N) - 1) + 1LL)),
          &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        if (trans_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_trans, trans);
        }
        trans_dirtyOnCpu = false;
        if (alpha_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_alpha, alpha);
        }
        alpha_dirtyOnCpu = false;
        hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
        BaumWelch_kernel78<<<grid, block>>>(
            gpu_beta, gpu_B, gpu_trans, gpu_alpha, N,
            static_cast<int32_T>(N) - 1, LDA, k, gpu_info_t, gpu_Xi);
        info_t_dirtyOnCpu = false;
        iwork_dirtyOnCpu = true;
        Xi_dirtyOnGpu = true;
      }
      vlen = Xi->size[1] * Xi->size[2];
      if (Xi->size[1] * Xi->size[2] == 0) {
        b_muj = 0.0;
      } else {
        if (Xi_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(Xi, &gpu_Xi);
        }
        Xi_dirtyOnGpu = false;
        if (iwork_dirtyOnCpu) {
          hipMemcpy(&info_t, gpu_info_t, 4ULL, hipMemcpyDeviceToHost);
        }
        b_muj = Xi->data[info_t];
        for (k = 0; k <= vlen - 2; k++) {
          b_muj += Xi->data[(info_t + 999 * ((k + 1) % Xi->size[1])) +
                            999 * Xi->size[1] * ((k + 1) / Xi->size[1])];
        }
      }
      kEnd = Xi->size[1] - 1;
      b_vlen = Xi->size[2] - 1;
      LDA = b_Xi->size[0] * b_Xi->size[1] * b_Xi->size[2];
      b_Xi->size[0] = 1;
      b_Xi->size[1] = Xi->size[1];
      b_Xi->size[2] = Xi->size[2];
      emxEnsureCapacity_real_T(b_Xi, LDA, &u_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(b_Xi, &b_gpu_Xi);
      LDA = b_Xi->size[1];
      vlen = Xi->size[1];
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>((kEnd + 1LL) * (b_vlen + 1LL)), &grid, &block,
          1024U, 65535U);
      if (validLaunchParams) {
        hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
        if (info_t_dirtyOnCpu) {
          hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
        }
        BaumWelch_kernel79<<<grid, block>>>(gpu_Xi, kEnd, b_vlen, LDA, vlen,
                                            gpu_muj, gpu_info_t, b_gpu_Xi);
        info_t_dirtyOnCpu = false;
      }
      m = b_Xi->size[2] - 1;
      xpageoffset = b_Xi->size[1] - 1;
      LDA = Xi->size[1];
      vlen = b_Xi->size[1];
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>((xpageoffset + 1LL) * (m + 1LL)), &grid, &block,
          1024U, 65535U);
      if (validLaunchParams) {
        if (info_t_dirtyOnCpu) {
          hipMemcpy(gpu_info_t, &info_t, 4ULL, hipMemcpyHostToDevice);
        }
        info_t_dirtyOnCpu = false;
        BaumWelch_kernel80<<<grid, block>>>(b_gpu_Xi, xpageoffset, m, LDA, vlen,
                                            gpu_info_t, gpu_Xi);
        Xi_dirtyOnGpu = true;
      }
    }
    vlen = Xi->size[2];
    if ((Xi->size[1] == 0) || (Xi->size[2] == 0)) {
      uint32_T b_sz[3];
      for (xpageoffset = 0; xpageoffset < 3; xpageoffset++) {
        b_sz[xpageoffset] = static_cast<uint32_T>(Xi->size[xpageoffset]);
      }
      kEnd = Gamma->size[0] * Gamma->size[1];
      Gamma->size[0] = 999;
      Gamma->size[1] = static_cast<int32_T>(b_sz[1]);
      emxEnsureCapacity_real_T(Gamma, kEnd, &t_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(Gamma, &gpu_Gamma);
      xpageoffset = 999 * static_cast<int32_T>(b_sz[1]) - 1;
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>(xpageoffset + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        BaumWelch_kernel83<<<grid, block>>>(xpageoffset, gpu_Gamma);
        Gamma_dirtyOnGpu = true;
      }
    } else {
      m = 999 * Xi->size[1] - 1;
      idx_dirtyOnGpu = true;
      kEnd = Gamma->size[0] * Gamma->size[1];
      Gamma->size[0] = 999;
      Gamma->size[1] = Xi->size[1];
      emxEnsureCapacity_real_T(Gamma, kEnd, &f_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(Gamma, &gpu_Gamma);
      validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(m + 1LL),
                                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        hipMemcpy(gpu_m, &m, 4ULL, hipMemcpyHostToDevice);
        idx_dirtyOnGpu = false;
        BaumWelch_kernel81<<<grid, block>>>(gpu_Xi, gpu_m, gpu_Gamma);
        Gamma_dirtyOnGpu = true;
      }
      for (k = 0; k <= vlen - 2; k++) {
        LDA = (k + 1) * (m + 1);
        validLaunchParams = mwGetLaunchParameters1D(
            static_cast<real_T>(m + 1LL), &grid, &block, 1024U, 65535U);
        if (validLaunchParams) {
          hipMemcpy(gpu_LDA, &LDA, 4ULL, hipMemcpyHostToDevice);
          if (idx_dirtyOnGpu) {
            hipMemcpy(gpu_m, &m, 4ULL, hipMemcpyHostToDevice);
          }
          idx_dirtyOnGpu = false;
          BaumWelch_kernel82<<<grid, block>>>(gpu_Xi, gpu_LDA, gpu_m,
                                              gpu_Gamma);
          Gamma_dirtyOnGpu = true;
        }
      }
    }
    //  M (Maximization step) Re-estimate the HMM parameters to obtain the
    //  maximum likelihood of the calculated parameters
    kEnd = Gamma->size[1] - 1;
    b_vlen = Pi->size[0] * Pi->size[1];
    Pi->size[0] = 1;
    Pi->size[1] = Gamma->size[1];
    emxEnsureCapacity_real_T(Pi, b_vlen, &v_emlrtRTEI);
    if (!Pi_dirtyOnCpu) {
      gpuEmxEnsureCapacity_real_T(Pi, &gpu_Pi);
    }
    validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(kEnd + 1LL),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      if (Pi_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Pi, Pi);
      }
      BaumWelch_kernel84<<<grid, block>>>(gpu_Gamma, kEnd, gpu_Pi);
      Pi_dirtyOnCpu = false;
      Pi_dirtyOnGpu = true;
    }
    if (static_cast<int32_T>(N) - 1 >= 0) {
      b_i = Xi->size[2] - 1;
    }
    for (i = 0; i < i13; i++) {
      kEnd = a->size[0] * a->size[1] * a->size[2];
      a->size[0] = 999;
      a->size[1] = 1;
      a->size[2] = b_i + 1;
      emxEnsureCapacity_real_T(a, kEnd, &w_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(a, &gpu_a);
      LDA = Xi->size[1];
      validLaunchParams =
          mwGetLaunchParameters1D(static_cast<real_T>(999LL * (b_i + 1LL)),
                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        BaumWelch_kernel85<<<grid, block>>>(gpu_Xi, i, b_i, LDA, gpu_a);
      }
      m = 1;
      if (Xi->size[2] != 1) {
        m = Xi->size[2];
      }
      if (m == 0) {
        c_y->size[0] = 1;
        c_y->size[1] = 0;
      } else {
        LDA = m - 1;
        kEnd = c_y->size[0] * c_y->size[1];
        c_y->size[0] = 1;
        c_y->size[1] = m;
        emxEnsureCapacity_real_T(c_y, kEnd, &f_emlrtRTEI);
        gpuEmxEnsureCapacity_real_T(c_y, &f_gpu_y);
        validLaunchParams = mwGetLaunchParameters1D(
            static_cast<real_T>((m - 1) + 1LL), &grid, &block, 1024U, 65535U);
        if (validLaunchParams) {
          hipMemcpy(gpu_LDA, &LDA, 4ULL, hipMemcpyHostToDevice);
          BaumWelch_kernel86<<<grid, block>>>(gpu_a, gpu_LDA, f_gpu_y);
        }
      }
      if (Gamma_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(Gamma, &gpu_Gamma);
      }
      Gamma_dirtyOnGpu = false;
      b_muj = Gamma->data[999 * i];
      for (k = 0; k < 998; k++) {
        b_muj += Gamma->data[(k + 999 * i) + 1];
      }
      xpageoffset = c_y->size[1] - 1;
      k = trans->size[0];
      validLaunchParams = mwGetLaunchParameters1D(
          static_cast<real_T>(xpageoffset + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
        if (trans_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_trans, trans);
        }
        BaumWelch_kernel87<<<grid, block>>>(f_gpu_y, i, xpageoffset, k, gpu_muj,
                                            gpu_trans);
        trans_dirtyOnCpu = false;
        trans_dirtyOnGpu = true;
      }
    }
    for (i = 0; i < i14; i++) {
      BaumWelch_kernel88<<<dim3(114U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          gpu_Gamma, i, *gpu_X, *f_gpu_x);
      BaumWelch_kernel89<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*f_gpu_x,
                                                                  *gpu_y);
      if (Gamma_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(Gamma, &gpu_Gamma);
      }
      Gamma_dirtyOnGpu = false;
      b_muj = Gamma->data[999 * i];
      for (k = 0; k < 998; k++) {
        b_muj += Gamma->data[(k + 999 * i) + 1];
      }
      xpageoffset = Mu->size[0];
      hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
      if (Mu_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Mu, Mu);
      }
      BaumWelch_kernel90<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(
          *gpu_y, i, xpageoffset, gpu_muj, gpu_Mu);
      Mu_dirtyOnCpu = false;
      Mu_dirtyOnGpu = true;
      xpageoffset = Mu->size[0];
      BaumWelch_kernel91<<<dim3(114U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          gpu_Mu, i, *gpu_X, xpageoffset, *gpu_d);
      BaumWelch_kernel92<<<dim3(114U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          gpu_Gamma, i, *gpu_d, *gpu_A);
      b_muj = 1.0;
      beta1 = 0.0;
      hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 58, 58,
                  999, (double *)&b_muj, (double *)&(*gpu_A)[0], 58,
                  (double *)&(*gpu_d)[0], 999, (double *)&beta1,
                  (double *)&(*d_gpu_y)[0], 58);
      b_muj = Gamma->data[999 * i];
      for (k = 0; k < 998; k++) {
        b_muj += Gamma->data[(k + 999 * i) + 1];
      }
      hipMemcpy(gpu_muj, &b_muj, 8ULL, hipMemcpyHostToDevice);
      if (Cov_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_Cov, Cov);
      }
      BaumWelch_kernel93<<<dim3(7U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          i, gpu_muj, *d_gpu_y, gpu_Cov);
      Cov_dirtyOnCpu = false;
      Cov_dirtyOnGpu = true;
    }
  }
  emxFree_real_T(&b_Xi);
  emxFree_real_T(&c_alpha);
  emxFree_real_T(&b_alpha);
  emxFree_real_T(&c_y);
  emxFree_real_T(&a);
  emxFree_real_T(&Gamma);
  emxFree_real_T(&d_x);
  emxFree_real_T(&Xi);
  emxFree_real_T(&c_x);
  emxFree_real_T(&beta);
  emxFree_real_T(&alpha);
  emxFree_int32_T(&iv1);
  emxFree_real_T(&B);
  *Time = coder::toc(expl_temp.tv_sec, expl_temp.tv_nsec);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (Mu_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(Mu, &gpu_Mu);
  }
  if (Cov_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(Cov, &gpu_Cov);
  }
  if (trans_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(trans, &gpu_trans);
  }
  if (Pi_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(Pi, &gpu_Pi);
  }
  hipFree(*gpu_X);
  hipFree(*gpu_idx);
  hipFree(*gpu_idx1);
  gpuEmxFree_real_T(&gpu_Cov);
  hipFree(*gpu_iwork);
  hipFree(*gpu_x_data);
  hipFree(*gpu_C);
  hipFree(*gpu_x);
  hipFree(gpu_muj);
  hipFree(gpu_m);
  gpuEmxFree_real_T(&gpu_Pi);
  hipFree(*gpu_dv_data);
  hipFree(*gpu_y);
  gpuEmxFree_real_T(&gpu_Mu);
  gpuEmxFree_real_T(&gpu_trans);
  gpuEmxFree_real_T(&b_gpu_x);
  hipFree(gpu_LDA);
  gpuEmxFree_real_T(&b_gpu_y);
  gpuEmxFree_real_T(&c_gpu_y);
  gpuEmxFree_real_T(&b_gpu_trans);
  gpuEmxFree_int32_T(&gpu_iv1);
  gpuEmxFree_real_T(&gpu_B);
  gpuEmxFree_real_T(&gpu_alpha);
  hipFree(*gpu_scale);
  gpuEmxFree_real_T(&b_gpu_alpha);
  hipFree(*gpu_X0);
  hipFree(gpu_info_t);
  gpuEmxFree_real_T(&gpu_beta);
  gpuEmxFree_real_T(&c_gpu_x);
  gpuEmxFree_real_T(&c_gpu_alpha);
  gpuEmxFree_real_T(&gpu_Xi);
  gpuEmxFree_real_T(&d_gpu_x);
  gpuEmxFree_real_T(&gpu_Gamma);
  hipFree(*e_gpu_x);
  hipFree(*d_gpu_y);
  hipFree(*e_gpu_y);
  gpuEmxFree_real_T(&b_gpu_Xi);
  hipFree(*f_gpu_x);
  gpuEmxFree_real_T(&gpu_a);
  hipFree(*gpu_d);
  hipFree(*gpu_A);
  hipFree(*gpu_T_data);
  gpuEmxFree_real_T(&f_gpu_y);
  hipFree(*gpu_ipiv_t);
  hipFree(*gpu_ipiv_data);
  hipFree(*b_gpu_B);
  hipFree(*gpu_jpvt_t_data);
  hipFree(*gpu_xRinv_data);
  hipFree(*gpu_y_data);
  hipFree(*gpu_quadform);
  hipFree(*gpu_Y_data);
}

// End of code generation (BaumWelch.cu)
