//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// div.cu
//
// Code generation for function 'div'
//

// Include files
#include "div.h"
#include "BaumWelch_data.h"
#include "BaumWelch_emxutil.h"
#include "BaumWelch_types.h"
#include "rt_nonfinite.h"

// Function Definitions
void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2)
{
  emxArray_real_T *b_in1;
  int32_T i;
  int32_T in2_idx_0;
  int32_T stride_0_0;
  int32_T stride_1_0;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&b_in1, 2, &i_emlrtRTEI, true);
  in2_idx_0 = in2->size[0];
  i = b_in1->size[0] * b_in1->size[1];
  if (in2_idx_0 == 1) {
    b_in1->size[0] = in1->size[0];
  } else {
    b_in1->size[0] = in2_idx_0;
  }
  b_in1->size[1] = in1->size[1];
  emxEnsureCapacity_real_T(b_in1, i, &i_emlrtRTEI);
  stride_0_0 = (in1->size[0] != 1);
  stride_1_0 = (in2_idx_0 != 1);
  for (i = 0; i < in1->size[1]; i++) {
    int32_T b_in2_idx_0;
    if (in2_idx_0 == 1) {
      b_in2_idx_0 = in1->size[0];
    } else {
      b_in2_idx_0 = in2_idx_0;
    }
    for (int32_T i1{0}; i1 < b_in2_idx_0; i1++) {
      b_in1->data[i1 + b_in1->size[0] * i] =
          in1->data[i1 * stride_0_0 + in1->size[0] * i] /
          in2->data[i1 * stride_1_0];
    }
  }
  i = in1->size[0] * in1->size[1];
  in1->size[0] = b_in1->size[0];
  in1->size[1] = b_in1->size[1];
  emxEnsureCapacity_real_T(in1, i, &j_emlrtRTEI);
  for (i = 0; i < b_in1->size[1]; i++) {
    for (int32_T i1{0}; i1 < b_in1->size[0]; i1++) {
      in1->data[i1 + in1->size[0] * i] = b_in1->data[i1 + b_in1->size[0] * i];
    }
  }
  emxFree_real_T(&b_in1);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

// End of code generation (div.cu)
