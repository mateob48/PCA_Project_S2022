//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_BaumWelch_api.cu
//
// Code generation for function '_coder_BaumWelch_api'
//

// Include files
#include "_coder_BaumWelch_api.h"
#include "BaumWelch.h"
#include "BaumWelch_data.h"
#include "BaumWelch_emxutil.h"
#include "BaumWelch_types.h"
#include "rt_nonfinite.h"

// Variable Definitions
static emlrtRTEInfo bb_emlrtRTEI{
    1,                      // lineNo
    1,                      // colNo
    "_coder_BaumWelch_api", // fName
    ""                      // pName
};

// Function Declarations
static real_T b_emlrt_marshallIn(const mxArray *N, const char_T *identifier);

static real_T b_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId);

static const mxArray *b_emlrt_marshallOut(const emxArray_real_T *u);

static real_T (*c_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[58000];

static real_T d_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId);

static real_T (*emlrt_marshallIn(const mxArray *X,
                                 const char_T *identifier))[58000];

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[58000];

static const mxArray *emlrt_marshallOut(const emxArray_real_T *u);

static const mxArray *emlrt_marshallOut(const real_T u);

// Function Definitions
static real_T b_emlrt_marshallIn(const mxArray *N, const char_T *identifier)
{
  emlrtMsgIdentifier thisId;
  real_T y;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(N), &thisId);
  emlrtDestroyArray(&N);
  return y;
}

static real_T b_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId)
{
  real_T y;
  y = d_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static const mxArray *b_emlrt_marshallOut(const emxArray_real_T *u)
{
  static const int32_T iv[3]{0, 0, 0};
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateNumericArray(3, (const void *)&iv[0], mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m, &u->data[0]);
  emlrtSetDimensions((mxArray *)m, &u->size[0], 3);
  emlrtAssign(&y, m);
  return y;
}

static real_T (*c_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[58000]
{
  static const int32_T dims[2]{1000, 58};
  real_T(*ret)[58000];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  ret = (real_T(*)[58000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T d_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 0U, (void *)&dims);
  ret = *static_cast<real_T *>(emlrtMxGetData(src));
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*emlrt_marshallIn(const mxArray *X,
                                 const char_T *identifier))[58000]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[58000];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = emlrt_marshallIn(emlrtAlias(X), &thisId);
  emlrtDestroyArray(&X);
  return y;
}

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[58000]
{
  real_T(*y)[58000];
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static const mxArray *emlrt_marshallOut(const emxArray_real_T *u)
{
  static const int32_T iv[2]{0, 0};
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateNumericArray(2, (const void *)&iv[0], mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m, &u->data[0]);
  emlrtSetDimensions((mxArray *)m, &u->size[0], 2);
  emlrtAssign(&y, m);
  return y;
}

static const mxArray *emlrt_marshallOut(const real_T u)
{
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateDoubleScalar(u);
  emlrtAssign(&y, m);
  return y;
}

void BaumWelch_api(BaumWelchStackData *SD, const mxArray *const prhs[2],
                   int32_T nlhs, const mxArray *plhs[5])
{
  emxArray_real_T *Cov;
  emxArray_real_T *Mu;
  emxArray_real_T *Pi;
  emxArray_real_T *trans;
  real_T(*X)[58000];
  real_T N;
  real_T Time;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&Mu, 2, &bb_emlrtRTEI, true);
  emxInit_real_T(&Cov, 3, &bb_emlrtRTEI, true);
  emxInit_real_T(&trans, 2, &bb_emlrtRTEI, true);
  emxInit_real_T(&Pi, 2, &bb_emlrtRTEI, true);
  // Marshall function inputs
  X = emlrt_marshallIn(emlrtAlias(prhs[0]), "X");
  N = b_emlrt_marshallIn(emlrtAliasP(prhs[1]), "N");
  // Invoke the target function
  BaumWelch(SD, *X, N, Mu, Cov, trans, Pi, &Time);
  // Marshall function outputs
  Mu->canFreeData = false;
  plhs[0] = emlrt_marshallOut(Mu);
  emxFree_real_T(&Mu);
  if (nlhs > 1) {
    Cov->canFreeData = false;
    plhs[1] = b_emlrt_marshallOut(Cov);
  }
  emxFree_real_T(&Cov);
  if (nlhs > 2) {
    trans->canFreeData = false;
    plhs[2] = emlrt_marshallOut(trans);
  }
  emxFree_real_T(&trans);
  if (nlhs > 3) {
    Pi->canFreeData = false;
    plhs[3] = emlrt_marshallOut(Pi);
  }
  emxFree_real_T(&Pi);
  if (nlhs > 4) {
    plhs[4] = emlrt_marshallOut(Time);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

// End of code generation (_coder_BaumWelch_api.cu)
