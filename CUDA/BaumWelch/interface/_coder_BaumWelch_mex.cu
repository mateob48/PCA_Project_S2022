//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_BaumWelch_mex.cu
//
// Code generation for function '_coder_BaumWelch_mex'
//

// Include files
#include "_coder_BaumWelch_mex.h"
#include "BaumWelch_data.h"
#include "BaumWelch_initialize.h"
#include "BaumWelch_terminate.h"
#include "BaumWelch_types.h"
#include "_coder_BaumWelch_api.h"
#include "rt_nonfinite.h"
#include <stdexcept>

void emlrtExceptionBridge();
void emlrtExceptionBridge()
{
  throw std::runtime_error("");
}
// Function Definitions
void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs,
                 const mxArray *prhs[])
{
  BaumWelchStackData *BaumWelchStackDataGlobal{nullptr};
  BaumWelchStackDataGlobal =
      static_cast<BaumWelchStackData *>(new BaumWelchStackData);
  mexAtExit(&BaumWelch_atexit);
  // Module initialization.
  BaumWelch_initialize();
  try {
    emlrtShouldCleanupOnError((emlrtCTX *)emlrtRootTLSGlobal, false);
    // Dispatch the entry-point.
    unsafe_BaumWelch_mexFunction(BaumWelchStackDataGlobal, nlhs, plhs, nrhs,
                                 prhs);
    // Module termination.
    BaumWelch_terminate();
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
  delete BaumWelchStackDataGlobal;
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLSR2022a(&emlrtRootTLSGlobal, &emlrtContextGlobal, nullptr, 1,
                           (void *)&emlrtExceptionBridge,
                           (const char_T *)"windows-1252", true);
  return emlrtRootTLSGlobal;
}

void unsafe_BaumWelch_mexFunction(BaumWelchStackData *SD, int32_T nlhs,
                                  mxArray *plhs[5], int32_T nrhs,
                                  const mxArray *prhs[2])
{
  const mxArray *outputs[5];
  int32_T b_nlhs;
  // Check for proper number of arguments.
  if (nrhs != 2) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 2, 4, 9, "BaumWelch");
  }
  if (nlhs > 5) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 9,
                        "BaumWelch");
  }
  // Call the function.
  BaumWelch_api(SD, prhs, nlhs, outputs);
  // Copy over outputs to the caller.
  if (nlhs < 1) {
    b_nlhs = 1;
  } else {
    b_nlhs = nlhs;
  }
  emlrtReturnArrays(b_nlhs, &plhs[0], &outputs[0]);
}

// End of code generation (_coder_BaumWelch_mex.cu)
