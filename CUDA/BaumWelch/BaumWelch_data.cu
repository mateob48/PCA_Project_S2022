//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// BaumWelch_data.cu
//
// Code generation for function 'BaumWelch_data'
//

// Include files
#include "BaumWelch_data.h"
#include "rt_nonfinite.h"

// Variable Definitions
emlrtCTX emlrtRootTLSGlobal{nullptr};

emlrtContext emlrtContextGlobal{
    true,                                               // bFirstTime
    false,                                              // bInitialized
    131626U,                                            // fVersionInfo
    nullptr,                                            // fErrorFunction
    "BaumWelch",                                        // fFunctionName
    nullptr,                                            // fRTCallStack
    false,                                              // bDebugMode
    {3130694236U, 2616137409U, 972914731U, 129233577U}, // fSigWrd
    nullptr                                             // fSigMem
};

emlrtRTEInfo i_emlrtRTEI{
    52,    // lineNo
    9,     // colNo
    "div", // fName
    "C:\\Program "
    "Files\\MATLAB\\R2022a\\toolbox\\eml\\eml\\+coder\\+internal\\div.m" // pName
};

emlrtRTEInfo j_emlrtRTEI{
    50,          // lineNo
    1,           // colNo
    "BaumWelch", // fName
    "C:\\Users\\mateo\\OneDrive\\Documents\\Mateo\\Project "
    "PAC\\Code\\CUDA\\BaumWelch.m" // pName
};

// End of code generation (BaumWelch_data.cu)
