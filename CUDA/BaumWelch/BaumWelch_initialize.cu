//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// BaumWelch_initialize.cu
//
// Code generation for function 'BaumWelch_initialize'
//

// Include files
#include "BaumWelch_initialize.h"
#include "BaumWelch_data.h"
#include "_coder_BaumWelch_mex.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"

// Function Definitions
void BaumWelch_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(
      emlrtRootTLSGlobal,
      (const char_T *)"EMLRT:runTime:MexFunctionNeedsLicense",
      (const char_T *)"distrib_computing_toolbox", 2);
  cublasEnsureInitialization(HIPBLAS_POINTER_MODE_HOST);
  cusolverEnsureInitialization();
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (BaumWelch_initialize.cu)
