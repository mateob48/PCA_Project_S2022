//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// toc.cu
//
// Code generation for function 'toc'
//

// Include files
#include "toc.h"
#include "rt_nonfinite.h"
#include "emlrt.h"

// Function Definitions
namespace coder {
real_T toc(real_T tstart_tv_sec, real_T tstart_tv_nsec)
{
  emlrtTimespec tnow;
  emlrtClockGettimeMonotonic(&tnow);
  return (tnow.tv_sec - tstart_tv_sec) +
         (tnow.tv_nsec - tstart_tv_nsec) / 1.0E+9;
}

} // namespace coder

// End of code generation (toc.cu)
