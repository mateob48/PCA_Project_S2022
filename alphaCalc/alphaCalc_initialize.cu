//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// alphaCalc_initialize.cu
//
// Code generation for function 'alphaCalc_initialize'
//

// Include files
#include "alphaCalc_initialize.h"
#include "_coder_alphaCalc_mex.h"
#include "alphaCalc_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void alphaCalc_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(
      emlrtRootTLSGlobal,
      (const char_T *)"EMLRT:runTime:MexFunctionNeedsLicense",
      (const char_T *)"distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (alphaCalc_initialize.cu)
